#include "BondList_interface.h"

BondList::BondList ()
{
  bondType = NULL;
  paramPosi = NULL;
  NBondForce = 0;
  param = NULL;
  paramLength = 0;
  initDeviceBondList (dbdlist);
}

BondList::~BondList()
{
  freeAPointer ((void**)&bondType);
  freeAPointer ((void**)&paramPosi);
  freeAPointer ((void**)&param);
  destroyDeviceBondList (dbdlist);
}


void BondList::init (const DeviceMDData & ddata,
		     const IndexType & listLength)
{
  hbdlist.init (ddata.numAtom, listLength);

  NBondForce_mem = 1024;
  paramLength_mem = NBondForce_mem * 3;
  bondType = (mdBondInteraction_t *) realloc (
      bondType, sizeof(mdBondInteraction_t) * NBondForce_mem);
  if (bondType == NULL) {
    throw MDExcptFailedReallocOnHost ("BondList::init", "bondType",
				      sizeof(mdBondInteraction_t) * NBondForce_mem);
  }
  paramPosi = (IndexType *) realloc (
      paramPosi, sizeof(mdBondInteraction_t) * NBondForce_mem);
  if (paramPosi == NULL){
    throw MDExcptFailedReallocOnHost ("BondList::init", "paramPosi",
				      sizeof(mdBondInteraction_t) * NBondForce_mem);
  }				      
  param = (ScalorType *) realloc (
      param, sizeof(ScalorType) * paramLength_mem);
  if (param == NULL){
    throw MDExcptFailedReallocOnHost ("BondList::init", "param",
				      sizeof(ScalorType) * paramLength_mem);
  }
}

  
void BondList::addBond (const IndexType & ii, const IndexType & jj,
			const mdBondInteraction_t & type,
			const ScalorType * thisparam)
{
  bool exist = false;
  ForceIndexType looking;;
  IndexType NParam = calNumBondParameter (type);
  for (looking = 0; looking < NBondForce; ++looking){
    if (type == bondType[looking]){
      bool same = true;
      for (IndexType i = 0; i < NParam; ++i){
	if (thisparam[i] != param[paramPosi[looking] + i]){
	  same = false;
	  break;
	}
      }
      if (same){
	exist = true;
	break;
      }
    }
  }
  if (!exist){
    if (NBondForce == NBondForce_mem){
      NBondForce_mem *= 2;
      bondType = (mdBondInteraction_t *) realloc (
	  bondType, sizeof(mdBondInteraction_t) * NBondForce_mem);
      if (bondType == NULL) {
	throw MDExcptFailedReallocOnHost ("BondList::init", "bondType",
					  sizeof(mdBondInteraction_t) * NBondForce_mem);
      }
      paramPosi = (IndexType *) realloc (
	  paramPosi, sizeof(mdBondInteraction_t) * NBondForce_mem);
      if (paramPosi == NULL){
	throw MDExcptFailedReallocOnHost ("BondList::init", "paramPosi",
					  sizeof(mdBondInteraction_t) * NBondForce_mem);
      }				      
    }
    if (paramLength == paramLength_mem){
      paramLength_mem *= 2;
      param = (ScalorType *) realloc (
	  param, sizeof(ScalorType) * paramLength_mem);
      if (param == NULL){
	throw MDExcptFailedReallocOnHost ("BondList::init", "param",
					  sizeof(ScalorType) * paramLength_mem);
      }
    } 
    bondType [NBondForce] = type;
    paramPosi[NBondForce] = paramLength;
    for (IndexType i = 0; i < NParam; ++i){
      param[paramPosi[NBondForce] + i] = thisparam[i];
    }
    NBondForce ++;
    paramLength += NParam;
  }
  hbdlist.addBond (ii, jj, looking);
}


// bubble sorting
// void BondList::sortBond()
// {
//   IndexType *indexMap = (IndexType *)malloc (sizeof(IndexType) * hbdlist.listLength);
//   if (indexMap == NULL){
//     MDExcptFailedMallocOnHost ("BondList::sortBond", "indexMap",
// 			       sizeof(IndexType) * hbdlist.listLength);
//   }
//   TypeType  *typeBuff = (TypeType *) malloc (sizeof(TypeType)  * hbdlist.listLength);
//   if (typeBuff == NULL){
//     MDExcptFailedMallocOnHost ("BondList::sortBond", "typeBuff",
// 			       sizeof(TypeType)  * hbdlist.listLength);
//   }
//   ForceIndexType * bkForceIndex = (ForceIndexType *) malloc (
//       sizeof (ForceIndexType) * hbdlist.listLength);
//   if (bkForceIndex == NULL){
//     MDExcptFailedMallocOnHost ("BondList::sortBond", "bkForceIndex",
// 			       sizeof (ForceIndexType) * hbdlist.listLength);
//   }			     
//   IndexType * bkData = (IndexType *) malloc (
//       sizeof (IndexType) * hbdlist.listLength);
//   if (bkData == NULL){
//     MDExcptFailedMallocOnHost ("BondList::sortBond", "bkData",
// 			       sizeof (IndexType) * hbdlist.listLength);
//   }
//   for (IndexType i = 0; i < hbdlist.stride; ++i){
//     for (IndexType j = 0; j < hbdlist.Nbond[i]; ++j){
//       indexMap[j] = j;
//       typeBuff[j] = bondType[hbdlist.bondIndex[j * hbdlist.stride + i]];
//       bkForceIndex[j] = hbdlist.bondIndex[j * hbdlist.stride + i];
//       bkData[j]       = hbdlist.data     [j * hbdlist.stride + i];
//     }
//     sortBuff (typeBuff, indexMap, hbdlist.Nbond[i]);
//     for (IndexType j = 0; j < hbdlist.Nbond[i]; ++j){
//       hbdlist.bondIndex[j * hbdlist.stride + i] = bkForceIndex[indexMap[j]];
//       hbdlist.data     [j * hbdlist.stride + i] = bkData      [indexMap[j]];
//     }
//   }
//   freeAPointer ((void**)&indexMap);
//   freeAPointer ((void**)&typeBuff);
//   freeAPointer ((void**)&bkForceIndex);
//   freeAPointer ((void**)&bkData);
// }    


void BondList::build()
{
  hbdlist.sort(bondType);
  buildDeviceBondList (hbdlist, dbdlist);
}

void initDeviceBondList (DeviceBondList & dbdlist)
{
  dbdlist.malloced = false;
  dbdlist.stride = 0;
  dbdlist.listLength = 0;
}

void destroyDeviceBondList(DeviceBondList &dbdlist )
{
  if (dbdlist.malloced) {
    hipFree (dbdlist.data);
    hipFree (dbdlist.bondIndex);
    hipFree (dbdlist.Nbond);
    checkCUDAError ("destroyDeviceBondList");
  }
}

void buildDeviceBondList (const HostBondList & hbdlist,
			  DeviceBondList & dbdlist)
{
  dbdlist.stride = hbdlist.stride;
  dbdlist.listLength = hbdlist.listLength;
  
  hipMalloc (&(dbdlist.data), 
	      sizeof(IndexType) * hbdlist.stride * hbdlist.listLength);
  checkCUDAError ("buildDeviceBondList malloc data");
  hipMalloc (&(dbdlist.bondIndex),
	      sizeof(TypeType) * hbdlist.stride * hbdlist.listLength);
  checkCUDAError ("buildDeviceBondList malloc bondIndex");
  hipMalloc (&(dbdlist.Nbond),
	      sizeof(IndexType) * hbdlist.stride);
  checkCUDAError ("buildDeviceBondList malloc Nbond");
  
  dbdlist.malloced = true;

  hipMemcpy (dbdlist.data, hbdlist.data,
	      sizeof(IndexType) * hbdlist.stride * hbdlist.listLength,
	      hipMemcpyHostToDevice);
  checkCUDAError ("buildDeviceBondList cpy host data to device");
  hipMemcpy (dbdlist.bondIndex, hbdlist.bondIndex,
	      sizeof(ForceIndexType) * hbdlist.stride * hbdlist.listLength,
	      hipMemcpyHostToDevice);
  checkCUDAError ("buildDeviceBondList cpy host bondIndex to device");
  hipMemcpy (dbdlist.Nbond, hbdlist.Nbond,
	      sizeof(IndexType) * hbdlist.stride,
	      hipMemcpyHostToDevice);
  checkCUDAError ("buildDeviceBondList cpy host Nbond to device");
}



HostBondList::HostBondList ()
{
  stride = 0;
  listLength = 0;
  data = NULL;
  bondIndex = NULL;
  Nbond = NULL;
}

HostBondList::~HostBondList()
{
  freeAPointer ((void**)&data);
  freeAPointer ((void**)&bondIndex);
  freeAPointer ((void**)&Nbond);
}

void HostBondList::init (const IndexType & stride_,
				const IndexType & listLength_)
{
  stride = stride_;
  listLength = listLength_;
  data = (IndexType *) malloc (sizeof(IndexType) * stride * listLength);
  if (data == NULL){
    throw MDExcptFailedMallocOnHost ("HostBondList::init", "data",
				     sizeof(IndexType) * stride * listLength);
  }
  bondIndex = (ForceIndexType *) malloc (sizeof(ForceIndexType *) * stride * listLength);
  if (bondIndex == NULL){
    throw MDExcptFailedMallocOnHost ("HostBondList::init", "bondIndex",
				     sizeof(ForceIndexType *) * stride * listLength);
  }
  Nbond = (IndexType *) malloc (sizeof(IndexType) * stride);
  if (Nbond == NULL){
    throw MDExcptFailedMallocOnHost ("HostBondList::init", "Nbond",
				     sizeof(IndexType) * stride);
  }
  
  for (IndexType i = 0; i < stride; ++i){
    Nbond[i] = 0;
  }
  for (IndexType i = 0; i < stride * listLength; ++i){
    data[i] = MaxForceIndexValue;
    bondIndex[i] = 0;
  }
}


void HostBondList::addBond (const IndexType & ii,
			    const IndexType & jj,
			    const ForceIndexType &looking)
{  
  data[Nbond[ii] * stride + ii] = jj;
  data[Nbond[jj] * stride + jj] = ii;
  bondIndex[Nbond[ii] * stride + ii] = looking;
  bondIndex[Nbond[jj] * stride + jj] = looking;
  Nbond[ii] ++;
  Nbond[jj] ++;
}


static void sortBuff (TypeType * ref, IndexType * indexMap, IndexType N)
{
  if (N == 0){
    return ;
  }
  for (IndexType i = 0; i < N - 1; ++i){
    IndexType j = i;
    while (j + 1 < N && ref[j] > ref[j+1]){
      TypeType tmptype = ref[j];
      ref[j] = ref[j+1];
      ref[j+1] = tmptype;
      IndexType tmpindex = indexMap[j];
      indexMap[j] = indexMap[j+1];
      indexMap[j+1] = tmpindex;
      j++;
    }
  }
}

void HostBondList::sort(mdBondInteraction_t * bondType)
{
  IndexType *indexMap = (IndexType *)malloc (sizeof(IndexType) * listLength);
  if (indexMap == NULL){
    MDExcptFailedMallocOnHost ("BondList::sortBond", "indexMap",
			       sizeof(IndexType) * listLength);
  }
  TypeType  *typeBuff = (TypeType *) malloc (sizeof(TypeType)  * listLength);
  if (typeBuff == NULL){
    MDExcptFailedMallocOnHost ("BondList::sortBond", "typeBuff",
			       sizeof(TypeType)  * listLength);
  }
  ForceIndexType * bkForceIndex = (ForceIndexType *) malloc (
      sizeof (ForceIndexType) * listLength);
  if (bkForceIndex == NULL){
    MDExcptFailedMallocOnHost ("BondList::sortBond", "bkForceIndex",
			       sizeof (ForceIndexType) * listLength);
  }			     
  IndexType * bkData = (IndexType *) malloc (
      sizeof (IndexType) * listLength);
  if (bkData == NULL){
    MDExcptFailedMallocOnHost ("BondList::sortBond", "bkData",
			       sizeof (IndexType) * listLength);
  }
  for (IndexType i = 0; i < stride; ++i){
    for (IndexType j = 0; j < Nbond[i]; ++j){
      indexMap[j] = j;
      typeBuff[j] = bondType[bondIndex[j * stride + i]];
      bkForceIndex[j] = bondIndex[j * stride + i];
      bkData[j]       = data     [j * stride + i];
    }
    sortBuff (typeBuff, indexMap, Nbond[i]);
    for (IndexType j = 0; j < Nbond[i]; ++j){
      bondIndex[j * stride + i] = bkForceIndex[indexMap[j]];
      data     [j * stride + i] = bkData      [indexMap[j]];
    }
  }
  freeAPointer ((void**)&indexMap);
  freeAPointer ((void**)&typeBuff);
  freeAPointer ((void**)&bkForceIndex);
  freeAPointer ((void**)&bkData);
}    
