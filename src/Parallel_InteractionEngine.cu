#include "hip/hip_runtime.h"
#define DEVICE_CODE

#include "common.h"
#include "Parallel_InteractionEngine.h"
#include "Parallel_Interface.h"
#include "NonBondedInteraction.h"
#include "compile_error_mixcode.h"

__constant__
InteractionType nonBondedInteractionType [MaxNumberNonBondedInteraction];
__constant__
ScalorType nonBondedInteractionParameter [MaxNumberNonBondedInteractionParameter];
__constant__
IndexType nonBondedInteractionParameterPosition [MaxNumberNonBondedInteraction];
__constant__
IndexType const_nonBondedInteractionTableLength[1];
__constant__
IndexType const_numAtomType[1];
__constant__
IndexType const_nonBondedInteractionTable [MaxLengthNonBondedInteractionTable];


Parallel::InteractionEngine::
InteractionEngine (const DeviceCellListedMDData & ddata)
    : hasBond (false), hasAngle(false)
{
  init (ddata);
}

void Parallel::InteractionEngine::
init (const DeviceCellListedMDData & ddata)
{
  totalNumCell = ddata.getNumCell().x *
      ddata.getNumCell().y * ddata.getNumCell().z;
  gridDim = toGridDim (totalNumCell);
  IndexType numThreadsInCell = Parallel::Interface::numThreadsInCell();
  
  sum_nb_p.init (totalNumCell*numThreadsInCell, NThreadForSum);
  sum_nb_vxx.init (totalNumCell*numThreadsInCell, NThreadForSum);
  sum_nb_vyy.init (totalNumCell*numThreadsInCell, NThreadForSum);
  sum_nb_vzz.init (totalNumCell*numThreadsInCell, NThreadForSum);
  sum_b_p.init (totalNumCell, NThreadForSum);
  sum_b_vxx.init (totalNumCell, NThreadForSum);
  sum_b_vyy.init (totalNumCell, NThreadForSum);
  sum_b_vzz.init (totalNumCell, NThreadForSum);
  sum_angle_p.init (totalNumCell, NThreadForSum);
}


void Parallel::InteractionEngine::
registNonBondedInteraction (const SystemNonBondedInteraction & sysNbInter)
{
  if (! sysNbInter.beBuilt()) {
    throw MDExcptUnbuiltNonBondedInteraction ("InteractionEngine_interface");
  }
  if (sysNbInter.numberOfInteraction() > MaxNumberBondedInteraction ){
    throw MDExcptExceedConstantMemLimit (
	"InteractionEngine::registNonBondedInteraction",
	"nonBonedInteractionType",
	MaxNumberNonBondedInteraction * sizeof(InteractionType));
  }
  if (sysNbInter.numberOfParameter() > MaxNumberNonBondedInteractionParameter ){
    throw MDExcptExceedConstantMemLimit (
	"InteractionEngine::registNonBondedInteraction",
	"nonBondedInteractionParameter",
	MaxNumberNonBondedInteractionParameter * sizeof(ScalorType));
  }

  hipMemcpyToSymbol(HIP_SYMBOL(nonBondedInteractionType),
		      sysNbInter.interactionType(), 
  		      sizeof(InteractionType) * sysNbInter.numberOfInteraction());
  hipMemcpyToSymbol(HIP_SYMBOL(nonBondedInteractionParameterPosition),
		      sysNbInter.interactionParameterPosition(),
  		      sizeof(ScalorType) * sysNbInter.numberOfInteraction());
  hipMemcpyToSymbol(HIP_SYMBOL(nonBondedInteractionParameter),
		      sysNbInter.interactionParameter(),
		      sizeof(IndexType) * sysNbInter.numberOfParameter());
  checkCUDAError ("InteractionEngine::init, init NB force setting");

  IndexType tableSize = sysNbInter.interactionTableSize();
  IndexType tmpNumAtomType = sysNbInter.numberOfAtomTypes();
  if (tableSize > MaxLengthNonBondedInteractionTable){
    throw MDExcptExceedConstantMemLimit(
	"InteractionEngine::registNonBondedInteraction",
	"nonBondedInteractionTable",
	MaxLengthNonBondedInteractionTable * sizeof (ScalorType));
  }
  hipMemcpyToSymbol(HIP_SYMBOL(const_nonBondedInteractionTableLength),
  		      &tableSize,
  		      sizeof (IndexType));
  checkCUDAError ("InteractionEngine::init, const_nonBondedInteractionTableLength");
  hipMemcpyToSymbol(HIP_SYMBOL(const_numAtomType),
		      &tmpNumAtomType,
		      sizeof (IndexType));
  checkCUDAError ("InteractionEngine::init, const_numAtomType");
  hipMemcpyToSymbol(HIP_SYMBOL(const_nonBondedInteractionTable),
  		      sysNbInter.interactionTable(),
  		      sizeof (IndexType) * tableSize);
  checkCUDAError ("InteractionEngine::init, const_nonBondedInteractionTable");

  IndexType numThreadsInCell = Parallel::Interface::numThreadsInCell();
  applyNonBondedInteraction_CellList_sbuffSize =
      sizeof(CoordType) * numThreadsInCell +
      sizeof(TypeType)  * numThreadsInCell;
  checkCUDAError ("InteractionEngine::init, init nonBondedInteractionTable");
}

void Parallel::InteractionEngine::
applyNonBondedInteraction (DeviceCellListedMDData & ddata,
			   const DeviceCellRelation & relation)
{
  Parallel::CudaGlobal::calNonBondedInteraction
      <<<gridDim, Parallel::Interface::numThreadsInCell()>>> (
	  ddata.dptr_coordinate(),
	  ddata.dptr_type(),
	  ddata.getGlobalBox().size,
	  ddata.getGlobalBox().sizei,
	  ddata.getRlist(),
	  ddata.dptr_numAtomInCell(),
	  relation.dptr_numNeighborCell(),
	  relation.dptr_neighborCellIndex(),
	  totalNumCell,
	  ddata.dptr_forceX(),
	  ddata.dptr_forceY(),
	  ddata.dptr_forceZ(),
	  sum_nb_p.getBuff(),
	  sum_nb_vxx.getBuff(),
	  sum_nb_vyy.getBuff(),
	  sum_nb_vzz.getBuff(),
	  err.ptr_de);
  checkCUDAError ("InteractionEngine::applyNonBondedInteraction");
}

void Parallel::InteractionEngine::
applyNonBondedInteraction (DeviceCellListedMDData & ddata,
			   const DeviceCellRelation & relation,
			   DeviceStatistic & st)
{
  Parallel::CudaGlobal::calNonBondedInteraction
      <<<gridDim, Parallel::Interface::numThreadsInCell()>>> (
	  ddata.dptr_coordinate(),
	  ddata.dptr_type(),
	  ddata.getGlobalBox().size,
	  ddata.getGlobalBox().sizei,
	  ddata.getRlist(),
	  ddata.dptr_numAtomInCell(),
	  relation.dptr_numNeighborCell(),
	  relation.dptr_neighborCellIndex(),
	  totalNumCell,
	  ddata.dptr_forceX(),
	  ddata.dptr_forceY(),
	  ddata.dptr_forceZ(),
	  sum_nb_p.getBuff(),
	  sum_nb_vxx.getBuff(),
	  sum_nb_vyy.getBuff(),
	  sum_nb_vzz.getBuff(),
	  err.ptr_de);
  checkCUDAError ("InteractionEngine::applyNonBondedInteraction");
  sum_nb_p.sumBuffAdd   (st.dptr_statisticData(), mdStatisticNonBondedPotential, 0);
  sum_nb_vxx.sumBuffAdd (st.dptr_statisticData(), mdStatisticVirialXX, 0);
  sum_nb_vyy.sumBuffAdd (st.dptr_statisticData(), mdStatisticVirialYY, 0);
  sum_nb_vzz.sumBuffAdd (st.dptr_statisticData(), mdStatisticVirialZZ, 0);
}
	  
__global__ void Parallel::CudaGlobal::
calNonBondedInteraction (const CoordType * coord,
			 const TypeType  * type,
			 const HostVectorType boxSize,
			 const HostVectorType boxSizei,
			 const ScalorType  rlist,
			 const IndexType * numAtomInCell,
			 const IndexType * numNeighborCell,
			 const IndexType * neighborCellIndex,
			 const IndexType   stride,
			 ScalorType * forcx,
			 ScalorType * forcy,
			 ScalorType * forcz,
			 ScalorType * statistic_nb_buff0,
			 ScalorType * statistic_nb_buff1,
			 ScalorType * statistic_nb_buff2,
			 ScalorType * statistic_nb_buff3,
			 mdError_t * ptr_de)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;

  IndexType this_numAtomInCell;
  IndexType this_numNeighborCell;  
  IndexType target_cellIndex;
  IndexType target_numAtomInCell;

  this_numAtomInCell = numAtomInCell[bid];
  if (this_numAtomInCell == 0) return;
  this_numNeighborCell = numNeighborCell[bid];

  IndexType ii = bid * blockDim.x + tid;
  CoordType refCoord;
  TypeType refType;
  ScalorType fsumx (0.f), fsumy(0.f), fsumz(0.f);
  ScalorType myPoten (0.0f), myVxx (0.0f), myVyy (0.0f), myVzz (0.0f);

  if (tid < this_numAtomInCell){
    refCoord = coord[ii];
    refType = type[ii];
  }  
  ScalorType rlist2 = rlist * rlist;
  
  extern __shared__ volatile char pub_sbuff[];
  CoordType * targetCoord =
      (CoordType *) & pub_sbuff;
  TypeType * targetType =
      (TypeType *) & targetCoord[blockDim.x];
  
  for (IndexType kk = 0; kk < this_numNeighborCell; ++kk){
    target_cellIndex = neighborCellIndex[bid * stride + kk];
    target_numAtomInCell = numAtomInCell[target_cellIndex];
    if (target_numAtomInCell == 0) continue;
    IndexType tmpLower = target_cellIndex * blockDim.x;
    IndexType jj = tmpLower + tid;
    if (tid < target_numAtomInCell) {
      targetCoord[tid] = coord[jj];
      targetType[tid] = type[jj];
    }
    if (tid < this_numAtomInCell){
      IndexType tmpUpper = tmpLower + target_numAtomInCell;
      for (IndexType ll = tmpLower; ll < tmpUpper; ++ll){
	if (ll != ii) {
	  ScalorType diffx = targetCoord[ll].x - refCoord.x;
	  ScalorType diffy = targetCoord[ll].y - refCoord.y;
	  ScalorType diffz = targetCoord[ll].z - refCoord.z;
	  shortestImage (boxSize.x, boxSizei.x, &diffx);
	  shortestImage (boxSize.y, boxSizei.y, &diffy);
	  shortestImage (boxSize.z, boxSizei.z, &diffz);
	  if (diffx*diffx+diffy*diffy+diffz*diffz < rlist2) {
	    IndexType fidx(0);
	    fidx = Parallel::CudaDevice::calNonBondedForceIndex (
		const_nonBondedInteractionTable,
		const_numAtomType[0],
		refType,
		targetType[ll]);
	    ScalorType fx, fy, fz, dp;
	    nbForcePoten (nonBondedInteractionType[fidx],
			  &nonBondedInteractionParameter
			  [nonBondedInteractionParameterPosition[fidx]],
			  diffx, diffy, diffz,
			  &fx, &fy, &fz, &dp);
	    myPoten += dp;
	    myVxx += fx * diffx;
	    myVyy += fy * diffy;
	    myVzz += fz * diffz;
	    fsumx += fx;
	    fsumy += fy;
	    fsumz += fz;
	  }
	  // __syncthreads();
	}
      }
    }
  }

  if (tid < this_numAtomInCell){
    forcx[ii] += fsumx;
    forcy[ii] += fsumy;
    forcz[ii] += fsumz;
  }
  
  statistic_nb_buff0[ii] = myPoten * 0.5f;
  statistic_nb_buff1[ii] = myVxx * 0.5f;
  statistic_nb_buff2[ii] = myVyy * 0.5f;
  statistic_nb_buff3[ii] = myVzz * 0.5f;
}

	  
    



    
