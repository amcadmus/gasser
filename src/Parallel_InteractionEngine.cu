#include "hip/hip_runtime.h"
#define DEVICE_CODE

#include "common.h"
#include "Parallel_InteractionEngine.h"
#include "Parallel_Interface.h"
#include "NonBondedInteraction.h"
#include "Parallel_Auxiliary.h"
#include "compile_error_mixcode.h"


__constant__
InteractionType nonBondedInteractionType [MaxNumberNonBondedInteraction];
__constant__
ScalorType nonBondedInteractionParameter [MaxNumberNonBondedInteractionParameter];
__constant__
IndexType nonBondedInteractionParameterPosition [MaxNumberNonBondedInteraction];
__constant__
IndexType const_nonBondedInteractionTableLength[1];
__constant__
IndexType const_numAtomType[1];
__constant__
IndexType const_nonBondedInteractionTable [MaxLengthNonBondedInteractionTable];


Parallel::InteractionEngine::
InteractionEngine ()
    : hasBond (false), hasAngle(false)
{
}

Parallel::InteractionEngine::
InteractionEngine (const DeviceCellListedMDData & ddata)
    : hasBond (false), hasAngle(false)
{
  reinit (ddata);
}

void Parallel::InteractionEngine::
reinit (const DeviceCellListedMDData & ddata)
{
  totalNumCell = ddata.getNumCell().x *
      ddata.getNumCell().y * ddata.getNumCell().z;
  gridDim = toGridDim (totalNumCell);
  IndexType numThreadsInCell = Parallel::Interface::numThreadsInCell();
  
  sum_nb_p.reinit (totalNumCell*numThreadsInCell, NThreadForSum);
  sum_nb_vxx.reinit (totalNumCell*numThreadsInCell, NThreadForSum);
  sum_nb_vyy.reinit (totalNumCell*numThreadsInCell, NThreadForSum);
  sum_nb_vzz.reinit (totalNumCell*numThreadsInCell, NThreadForSum);
  sum_b_p.reinit (totalNumCell, NThreadForSum);
  sum_b_vxx.reinit (totalNumCell, NThreadForSum);
  sum_b_vyy.reinit (totalNumCell, NThreadForSum);
  sum_b_vzz.reinit (totalNumCell, NThreadForSum);
  sum_angle_p.reinit (totalNumCell, NThreadForSum);
}


void Parallel::InteractionEngine::
registNonBondedInteraction (const SystemNonBondedInteraction & sysNbInter)
{
  if (! sysNbInter.beBuilt()) {
    throw MDExcptUnbuiltNonBondedInteraction ("InteractionEngine_interface");
  }
  if (sysNbInter.numberOfInteraction() > MaxNumberBondedInteraction ){
    throw MDExcptExceedConstantMemLimit (
	"InteractionEngine::registNonBondedInteraction",
	"nonBonedInteractionType",
	MaxNumberNonBondedInteraction * sizeof(InteractionType));
  }
  if (sysNbInter.numberOfParameter() > MaxNumberNonBondedInteractionParameter ){
    throw MDExcptExceedConstantMemLimit (
	"InteractionEngine::registNonBondedInteraction",
	"nonBondedInteractionParameter",
	MaxNumberNonBondedInteractionParameter * sizeof(ScalorType));
  }

  hipMemcpyToSymbol(HIP_SYMBOL(nonBondedInteractionType),
		      sysNbInter.interactionType(), 
  		      sizeof(InteractionType) * sysNbInter.numberOfInteraction());
  hipMemcpyToSymbol(HIP_SYMBOL(nonBondedInteractionParameterPosition),
		      sysNbInter.interactionParameterPosition(),
  		      sizeof(ScalorType) * sysNbInter.numberOfInteraction());
  hipMemcpyToSymbol(HIP_SYMBOL(nonBondedInteractionParameter),
		      sysNbInter.interactionParameter(),
		      sizeof(IndexType) * sysNbInter.numberOfParameter());
  checkCUDAError ("InteractionEngine::init, init NB force setting");

  IndexType tableSize = sysNbInter.interactionTableSize();
  IndexType tmpNumAtomType = sysNbInter.numberOfAtomTypes();
  if (tableSize > MaxLengthNonBondedInteractionTable){
    throw MDExcptExceedConstantMemLimit(
	"InteractionEngine::registNonBondedInteraction",
	"nonBondedInteractionTable",
	MaxLengthNonBondedInteractionTable * sizeof (ScalorType));
  }
  hipMemcpyToSymbol(HIP_SYMBOL(const_nonBondedInteractionTableLength),
  		      &tableSize,
  		      sizeof (IndexType));
  checkCUDAError ("InteractionEngine::init, const_nonBondedInteractionTableLength");
  hipMemcpyToSymbol(HIP_SYMBOL(const_numAtomType),
		      &tmpNumAtomType,
		      sizeof (IndexType));
  checkCUDAError ("InteractionEngine::init, const_numAtomType");
  hipMemcpyToSymbol(HIP_SYMBOL(const_nonBondedInteractionTable),
  		      sysNbInter.interactionTable(),
  		      sizeof (IndexType) * tableSize);
  checkCUDAError ("InteractionEngine::init, const_nonBondedInteractionTable");

  IndexType numThreadsInCell = Parallel::Interface::numThreadsInCell();
  applyNonBondedInteraction_CellList_sbuffSize =
      sizeof(CoordType) * numThreadsInCell +
      sizeof(TypeType)  * numThreadsInCell;
  checkCUDAError ("InteractionEngine::init, init nonBondedInteractionTable");
}

void Parallel::InteractionEngine::
applyNonBondedInteraction (DeviceCellListedMDData & ddata,
			   const DeviceCellRelation & relation,
			   DeviceStatistic & st)
{
  Parallel::CudaGlobal::calNonBondedInteraction
      <<<gridDim, Parallel::Interface::numThreadsInCell(),
      applyNonBondedInteraction_CellList_sbuffSize>>> (
	  ddata.dptr_coordinate(),
	  ddata.dptr_type(),
	  ddata.getGlobalBox().size,
	  ddata.getGlobalBox().sizei,
	  ddata.getRlist(),
	  ddata.dptr_numAtomInCell(),
	  relation.dptr_numNeighborCell(),
	  relation.dptr_neighborCellIndex(),
	  relation.stride_neighborCellIndex(),
	  ddata.dptr_forceX(),
	  ddata.dptr_forceY(),
	  ddata.dptr_forceZ(),
	  sum_nb_p.getBuff(),
	  sum_nb_vxx.getBuff(),
	  sum_nb_vyy.getBuff(),
	  sum_nb_vzz.getBuff(),
	  err.ptr_de);
  checkCUDAError ("InteractionEngine::applyNonBondedInteraction");
  sum_nb_p.sumBuffAdd   (st.dptr_statisticData(), mdStatistic_NonBondedPotential, 0);
  sum_nb_vxx.sumBuffAdd (st.dptr_statisticData(), mdStatistic_VirialXX, 0);
  sum_nb_vyy.sumBuffAdd (st.dptr_statisticData(), mdStatistic_VirialYY, 0);
  sum_nb_vzz.sumBuffAdd (st.dptr_statisticData(), mdStatistic_VirialZZ, 0);
}

void Parallel::InteractionEngine::
applyNonBondedInteraction (DeviceCellListedMDData & ddata,
			   const DeviceCellRelation & relation)
{
  Parallel::CudaGlobal::calNonBondedInteraction
      <<<gridDim, Parallel::Interface::numThreadsInCell(),
      applyNonBondedInteraction_CellList_sbuffSize>>> (
	  ddata.dptr_coordinate(),
	  ddata.dptr_type(),
	  ddata.getGlobalBox().size,
	  ddata.getGlobalBox().sizei,
	  ddata.getRlist(),
	  ddata.dptr_numAtomInCell(),
	  relation.dptr_numNeighborCell(),
	  relation.dptr_neighborCellIndex(),
	  relation.stride_neighborCellIndex(),
	  ddata.dptr_forceX(),
	  ddata.dptr_forceY(),
	  ddata.dptr_forceZ(),
	  err.ptr_de);
  checkCUDAError ("InteractionEngine::applyNonBondedInteraction");
}


__global__ void Parallel::CudaGlobal::
calNonBondedInteraction (const CoordType * coord,
			 const TypeType  * type,
			 const HostVectorType boxSize,
			 const HostVectorType boxSizei,
			 const ScalorType  rlist,
			 const IndexType * numAtomInCell,
			 const IndexType * numNeighborCell,
			 const IndexType * neighborCellIndex,
			 const IndexType   stride,
			 ScalorType * forcx,
			 ScalorType * forcy,
			 ScalorType * forcz,
			 ScalorType * statistic_nb_buff0,
			 ScalorType * statistic_nb_buff1,
			 ScalorType * statistic_nb_buff2,
			 ScalorType * statistic_nb_buff3,
			 mdError_t * ptr_de)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;

  IndexType this_numAtomInCell;
  IndexType this_numNeighborCell;  
  IndexType target_cellIndex;
  IndexType target_numAtomInCell;
  IndexType ii = bid * blockDim.x + tid;

  this_numNeighborCell = numNeighborCell[bid];
  if (this_numNeighborCell == 0) {
    statistic_nb_buff0[ii] = 0.f;
    statistic_nb_buff1[ii] = 0.f;
    statistic_nb_buff2[ii] = 0.f;
    statistic_nb_buff3[ii] = 0.f;
    return;
  }  
  this_numAtomInCell = numAtomInCell[bid];
  if (this_numAtomInCell == 0) {
    statistic_nb_buff0[ii] = 0.f;
    statistic_nb_buff1[ii] = 0.f;
    statistic_nb_buff2[ii] = 0.f;
    statistic_nb_buff3[ii] = 0.f;
    return;
  }  
    
  // if (tid == 0){
  //   printf ("bid: %d, numNei: %d\n", bid, this_numNeighborCell);
  // }    

  CoordType refCoord;
  TypeType refType;
  ScalorType fsumx (0.f), fsumy(0.f), fsumz(0.f);
  ScalorType myPoten (0.0f), myVxx (0.0f), myVyy (0.0f), myVzz (0.0f);

  if (tid < this_numAtomInCell){
    refCoord = coord[ii];
    refType = type[ii];
  }  
  ScalorType rlist2 = rlist * rlist;
  
  extern __shared__ volatile char pub_sbuff[];
  CoordType * targetCoord =
      (CoordType *) & pub_sbuff;
  TypeType * targetType =
      (TypeType *) & targetCoord[blockDim.x];
  
  // IndexType count = 0;
  for (IndexType kk = 0; kk < this_numNeighborCell; ++kk){
    __syncthreads();
    target_cellIndex = neighborCellIndex[bid * stride + kk];
    target_numAtomInCell = numAtomInCell[target_cellIndex];
    if (target_numAtomInCell == 0) continue;
    IndexType indexShift = target_cellIndex * blockDim.x;
    IndexType jj = indexShift + tid;
    if (tid < target_numAtomInCell) {
      targetCoord[tid] = coord[jj];
      targetType[tid] = type[jj];
    }
    __syncthreads();
    if (tid < this_numAtomInCell){
      for (IndexType ll = 0; ll < target_numAtomInCell; ++ll){
	if (ll + indexShift != ii) {	  
	  ScalorType diffx = targetCoord[ll].x - refCoord.x;
	  ScalorType diffy = targetCoord[ll].y - refCoord.y;
	  ScalorType diffz = targetCoord[ll].z - refCoord.z;
	  shortestImage (boxSize.x, boxSizei.x, &diffx);
	  shortestImage (boxSize.y, boxSizei.y, &diffy);
	  shortestImage (boxSize.z, boxSizei.z, &diffz);
	  // shortestImage (boxSize.x, diffx);
	  // shortestImage (boxSize.y, diffy);
	  // shortestImage (boxSize.z, diffz);
	  if (diffx*diffx+diffy*diffy+diffz*diffz < rlist2) {
	    // count ++;
	    IndexType fidx(0);
	    fidx = Parallel::CudaDevice::calNonBondedForceIndex (
		const_nonBondedInteractionTable,
		const_numAtomType[0],
		refType,
		targetType[ll]);
	    ScalorType fx, fy, fz, dp;
	    nbForcePoten (nonBondedInteractionType[fidx],
			  &nonBondedInteractionParameter
			  [nonBondedInteractionParameterPosition[fidx]],
			  diffx, diffy, diffz,
			  &fx, &fy, &fz, &dp);
	    myPoten += dp;
	    myVxx += fx * diffx;
	    myVyy += fy * diffy;
	    myVzz += fz * diffz;
	    fsumx += fx;
	    fsumy += fy;
	    fsumz += fz;
	  }
	  // __syncthreads();
	}
      }
    }
  }
  // printf ("bid: %d, tid: %d, num eff: %d. fsum %f\n", bid, tid, count, fsumx);

  if (tid < this_numAtomInCell){
    forcx[ii] += fsumx;
    forcy[ii] += fsumy;
    forcz[ii] += fsumz;
  }
  
  statistic_nb_buff0[ii] = myPoten * 0.5f;
  statistic_nb_buff1[ii] = myVxx * 0.5f;
  statistic_nb_buff2[ii] = myVyy * 0.5f;
  statistic_nb_buff3[ii] = myVzz * 0.5f;
}

__global__ void Parallel::CudaGlobal::
calNonBondedInteraction (const CoordType * coord,
			 const TypeType  * type,
			 const HostVectorType boxSize,
			 const HostVectorType boxSizei,
			 const ScalorType  rlist,
			 const IndexType * numAtomInCell,
			 const IndexType * numNeighborCell,
			 const IndexType * neighborCellIndex,
			 const IndexType   stride,
			 ScalorType * forcx,
			 ScalorType * forcy,
			 ScalorType * forcz,
			 mdError_t * ptr_de)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;

  IndexType this_numAtomInCell;
  IndexType this_numNeighborCell;  
  IndexType target_cellIndex;
  IndexType target_numAtomInCell;
  IndexType ii = bid * blockDim.x + tid;

  this_numNeighborCell = numNeighborCell[bid];
  if (this_numNeighborCell == 0) {
    return;
  }  
  this_numAtomInCell = numAtomInCell[bid];
  if (this_numAtomInCell == 0) {
    return;
  }  
    
  // if (tid == 0){
  //   printf ("bid: %d, numNei: %d\n", bid, this_numNeighborCell);
  // }    

  CoordType refCoord;
  TypeType refType;
  ScalorType fsumx (0.f), fsumy(0.f), fsumz(0.f);

  if (tid < this_numAtomInCell){
    refCoord = coord[ii];
    refType = type[ii];
  }  
  ScalorType rlist2 = rlist * rlist;
  
  extern __shared__ volatile char pub_sbuff[];
  CoordType * targetCoord =
      (CoordType *) & pub_sbuff;
  TypeType * targetType =
      (TypeType *) & targetCoord[blockDim.x];
  
  // IndexType count = 0;
  for (IndexType kk = 0; kk < this_numNeighborCell; ++kk){
    __syncthreads();
    target_cellIndex = neighborCellIndex[bid * stride + kk];
    target_numAtomInCell = numAtomInCell[target_cellIndex];
    if (target_numAtomInCell == 0) continue;
    IndexType indexShift = target_cellIndex * blockDim.x;
    IndexType jj = indexShift + tid;
    if (tid < target_numAtomInCell) {
      targetCoord[tid] = coord[jj];
      targetType[tid] = type[jj];
    }
    __syncthreads();
    if (tid < this_numAtomInCell){
      for (IndexType ll = 0; ll < target_numAtomInCell; ++ll){
	if (ll + indexShift != ii) {	  
	  ScalorType diffx = targetCoord[ll].x - refCoord.x;
	  ScalorType diffy = targetCoord[ll].y - refCoord.y;
	  ScalorType diffz = targetCoord[ll].z - refCoord.z;
	  shortestImage (boxSize.x, boxSizei.x, &diffx);
	  shortestImage (boxSize.y, boxSizei.y, &diffy);
	  shortestImage (boxSize.z, boxSizei.z, &diffz);
	  // shortestImage (boxSize.x, diffx);
	  // shortestImage (boxSize.y, diffy);
	  // shortestImage (boxSize.z, diffz);
	  if (diffx*diffx+diffy*diffy+diffz*diffz < rlist2) {
	    // count ++;
	    // IndexType fidx(0);
	    IndexType fidx;
	    fidx = Parallel::CudaDevice::calNonBondedForceIndex (
		const_nonBondedInteractionTable,
		const_numAtomType[0],
		refType,
		targetType[ll]);
	    ScalorType fx, fy, fz, dp;
	    nbForcePoten (nonBondedInteractionType[fidx],
			  &nonBondedInteractionParameter
			  [nonBondedInteractionParameterPosition[fidx]],
			  diffx, diffy, diffz,
			  &fx, &fy, &fz, &dp);
	    fsumx += fx;
	    fsumy += fy;
	    fsumz += fz;
	  }
	  // __syncthreads();
	}
      }
    }
  }
  // printf ("bid: %d, tid: %d, num eff: %d. fsum %f\n", bid, tid, count, fsumx);

  if (tid < this_numAtomInCell){
    forcx[ii] += fsumx;
    forcy[ii] += fsumy;
    forcz[ii] += fsumz;
  }  
}


void Parallel::InteractionEngine::
clearInteraction (DeviceCellListedMDData & data)
{
  Parallel::Auxiliary::setValue
      <<<gridDim, Parallel::Interface::numThreadsInCell()>>> (
	  data.dptr_forceX(), 0.f);
  Parallel::Auxiliary::setValue
      <<<gridDim, Parallel::Interface::numThreadsInCell()>>> (
	  data.dptr_forceY(), 0.f);
  Parallel::Auxiliary::setValue
      <<<gridDim, Parallel::Interface::numThreadsInCell()>>> (
	  data.dptr_forceZ(), 0.f);
  checkCUDAError ("InteractionEngine::clearInteraction");
}

    



    
