#define DEVICE_CODE

#include "MDError_interface.h"

#define NErrorIndex	2
#define NErrorScalor	13

MDError::MDError ()
{
  he = mdSuccess;
  hindex = 0;
  hscalor = 0;
  hindex = (IndexType *) malloc (sizeof(IndexType) * NErrorIndex);
  hscalor = (ScalorType *) malloc (sizeof(ScalorType) * NErrorScalor);
  for (IndexType i = 0; i < NErrorIndex; ++i){
    hindex[i] = 0;
  }
  for (IndexType i = 0; i < NErrorScalor; ++i){
    hscalor[i] = 0.f;
  }
  hipMalloc ((void **)&ptr_de, sizeof(mdError_t));
  hipMemcpy (ptr_de, &he, sizeof(mdError_t), hipMemcpyHostToDevice);
  hipMalloc ((void **)&ptr_dindex, NErrorIndex * sizeof(IndexType));
  hipMalloc ((void **)&ptr_dscalor, NErrorScalor * sizeof(ScalorType));
  hipMemcpy (ptr_dindex, &hindex, NErrorIndex * sizeof(IndexType),
	      hipMemcpyHostToDevice);
  hipMemcpy (ptr_dscalor, &hscalor, NErrorScalor * sizeof(ScalorType),
	      hipMemcpyHostToDevice);
  // hipMemset (ptr_dindex, 0, NErrorIndex * sizeof(IndexType));
  // hipMemset (ptr_dscalor, 0, NErrorScalor * sizeof(ScalorType));
  checkCUDAError ("MDError::MDError");
}

MDError::~MDError()
{
  freeAPointer((void**)&hindex);
  freeAPointer((void**)&hscalor);
  hipFree (ptr_de);
  hipFree (ptr_dindex);
  hipFree (ptr_dscalor);
  checkCUDAError ("MDError::~MDError");
}

inline char * MDError::getErrorString (mdError_t err) {
  switch (err){
  case mdSuccess:
      return "Success";
  case mdErrorShortCellList:
      return "The cell list is too short, increase the number of thread per block";
  case mdErrorShortNeighborList:
      return "The neighbor list is too shor, increase the DeviceNeighborListExpansion";
  case mdErrorOverFlowCellIdx:
      return "Detect an over flown cell index";
  case mdErrorBreakFENEBond:
      return "Detect a broken FENE bond";
  default:
      return "Unknow error status";
  }
}

void MDError::updateHost ()
{
  hipMemcpy (hindex, ptr_dindex, NErrorIndex * sizeof(IndexType),
	      hipMemcpyDeviceToHost);
  hipMemcpy (hscalor, ptr_dscalor, NErrorScalor * sizeof(ScalorType),
	      hipMemcpyDeviceToHost);
  checkCUDAError ("MDError::updateHost");
}


// #include "Parallel_Interface.h"
void MDError::check (const char * msg)
{
  hipMemcpy (&he, ptr_de, sizeof(mdError_t), hipMemcpyDeviceToHost);
  updateHost();
  if (mdSuccess != he){
//    fprintf (stderr, "myrank: %d, Md error: %s: %s.\n", Parallel::Interface::myRank(), msg, getErrorString(he));
    fprintf (stderr, "recorded indexes are");
    for (IndexType i = 0; i < NErrorIndex; ++i){
      printf ("%d  ", hindex[i]);
    }
    printf ("\n");
    for (IndexType i = 0; i < NErrorScalor; ++i){
      printf ("%f  ", hscalor[i]);
    }
    printf ("\n");
    // exit (EXIT_FAILURE);
    throw MDExcptCuda ();
  }
}

  
