#include "hip/hip_runtime.h"
#include "InteractionEngine_interface.h"
#include "NonBondedInteraction.h"
#include "BondInteraction.h"
#include "AngleInteraction.h"

texture<CoordType,  1, hipReadModeElementType> global_texRef_interaction_coord;
__constant__
InteractionType nonBondedInteractionType [MaxNumberNonBondedInteraction];
__constant__
ScalorType nonBondedInteractionParameter [MaxNumberNonBondedInteractionParameter];
__constant__
IndexType nonBondedInteractionParameterPosition [MaxNumberNonBondedInteraction];
__constant__
InteractionType bondedInteractionType [MaxNumberBondedInteraction];
__constant__
IndexType bondedInteractionParameterPosition [MaxNumberBondedInteraction];
__constant__
ScalorType bondedInteractionParameter [MaxNumberBondedInteractionParamemter];

void InteractionEngine_interface::init (const MDSystem  & sys,
					const IndexType & NTread)
{
  hasBond = false;
  hasAngle = false;
  myBlockDim.y = 1;
  myBlockDim.z = 1;
  myBlockDim.x = NTread;
  IndexType nob;
  if (sys.ddata.numAtom % myBlockDim.x == 0){
    nob = sys.ddata.numAtom / myBlockDim.x;
  } else {
    nob = sys.ddata.numAtom / myBlockDim.x + 1;
  }
  atomGridDim = toGridDim (nob);

  // size_t sizetype = sizeof(TypeType)*sys.ddata.numMem;
  hipBindTexture(0, global_texRef_interaction_coord, sys.ddata.coord,
		  sizeof(CoordType) * sys.ddata.numMem);
  checkCUDAError ("InteractionEngine::init, bind texture");
  
  // init sum vectors
  sum_nb_p.init (sys.ddata.numAtom, NThreadForSum);
  sum_nb_vxx.init (sys.ddata.numAtom, NThreadForSum);
  sum_nb_vyy.init (sys.ddata.numAtom, NThreadForSum);
  sum_nb_vzz.init (sys.ddata.numAtom, NThreadForSum);
  sum_b_p.init (nob, NThreadForSum);
  sum_b_vxx.init (nob, NThreadForSum);
  sum_b_vyy.init (nob, NThreadForSum);
  sum_b_vzz.init (nob, NThreadForSum);
  sum_angle_p.init (nob, NThreadForSum);
  for (IndexType i = 0; i < 8; ++i){
    hipStreamCreate(&sum_stream[i]);
  }
  checkCUDAError ("InteractionEngine::init init sum statistic");
  
  // init nb force param
}

void InteractionEngine_interface::
registNonBondedInteraction (const SystemNonBondedInteraction & sysNbInter)
{
  if (! sysNbInter.beBuilt()) {
    throw MDExcptUnbuiltNonBondedInteraction ("InteractionEngine_interface");
  }
  if (sysNbInter.numberOfInteraction() > MaxNumberBondedInteraction ){
    throw MDExcptExceedConstantMemLimit (
	"InteractionEngine::registNonBondedInteraction",
	"nonBonedInteractionType",
	MaxNumberNonBondedInteraction * sizeof(InteractionType));
  }
  if (sysNbInter.numberOfParameter() > MaxNumberNonBondedInteractionParameter ){
    throw MDExcptExceedConstantMemLimit (
	"InteractionEngine::registNonBondedInteraction",
	"nonBondedInteractionParameter",
	MaxNumberNonBondedInteractionParameter * sizeof(ScalorType));
  }

  hipMemcpyToSymbol(HIP_SYMBOL(nonBondedInteractionType),
		      sysNbInter.interactionType(), 
  		      sizeof(InteractionType) * sysNbInter.numberOfInteraction());
  hipMemcpyToSymbol(HIP_SYMBOL(nonBondedInteractionParameterPosition),
		      sysNbInter.interactionParameterPosition(),
  		      sizeof(ScalorType) * sysNbInter.numberOfInteraction());
  hipMemcpyToSymbol(HIP_SYMBOL(nonBondedInteractionParameter),
		      sysNbInter.interactionParameter(),
		      sizeof(IndexType) * sysNbInter.numberOfParameter());
  checkCUDAError ("InteractionEngine::init, init NB force setting");
}


void InteractionEngine_interface::
registBondedInteraction (const SystemBondedInteraction & sysBdInter)
{
  if (sysBdInter.hasBond() ){
    hasBond = true;
  }
  if (sysBdInter.hasAngle()){
    hasAngle = true;
  }

  if (sysBdInter.numberOfInteraction() > MaxNumberBondedInteraction ){
    throw MDExcptExceedConstantMemLimit (
	"InteractionEngine::registBondedInteraction",
	"bondedInteractionType",
	MaxNumberBondedInteraction * sizeof(InteractionType));
  }
  if (sysBdInter.numberOfParameter() > MaxNumberBondedInteractionParamemter ){
    throw MDExcptExceedConstantMemLimit (
	"InteractionEngine::registBondedInteraction",
	"bondedInteractionParameter",
	MaxNumberBondedInteractionParamemter * sizeof(ScalorType));
  }

  if (hasBond || hasAngle){
    hipMemcpyToSymbol(HIP_SYMBOL(bondedInteractionType),
			sysBdInter.interactionType(),
			sizeof(InteractionType) * sysBdInter.numberOfInteraction());
    hipMemcpyToSymbol(HIP_SYMBOL(bondedInteractionParameterPosition),
			sysBdInter.interactionParameterPosition(),
			sizeof(ScalorType) * sysBdInter.numberOfInteraction());
    hipMemcpyToSymbol(HIP_SYMBOL(bondedInteractionParameter),
			sysBdInter.interactionParameter(),
			sizeof(IndexType) * sysBdInter.numberOfParameter());
    checkCUDAError ("InteractionEngine::init, init bond force setting");
    // cal shared buff size
    calBondInteraction_sbuffSize  = myBlockDim.x * sizeof(ScalorType);
    calAngleInteraction_sbuffSize = myBlockDim.x * sizeof(ScalorType);
  }
}

InteractionEngine_interface::~InteractionEngine_interface()
{
  hipUnbindTexture(global_texRef_interaction_coord);
  for (IndexType i = 0; i < 8; ++i){
    hipStreamDestroy(sum_stream[i]);
  }
}

void InteractionEngine_interface::clearInteraction (MDSystem & sys)
{
  clearForce
      <<<atomGridDim, myBlockDim>>>(
	  sys.ddata.numAtom,
	  sys.ddata.forcx, sys.ddata.forcy, sys.ddata.forcz);
  checkCUDAError ("InteractionEngine::clearInteraction");
}


void InteractionEngine_interface::
applyNonBondedInteraction  (MDSystem & sys,
			    const NeighborList & nlist,
			    MDTimer *timer )
{
  if (timer != NULL) timer->tic(mdTimeNonBondedInteraction);
  calNonBondedInteraction
      <<<atomGridDim, myBlockDim>>> (
	  sys.ddata.numAtom,
	  sys.ddata.coord,
	  sys.ddata.forcx,  sys.ddata.forcy,  sys.ddata.forcz,
	  sys.ddata.type, 
	  sys.box, nlist.dnlist,
	  err.ptr_de,
	  err.ptr_dindex,
	  err.ptr_dscalor);
  checkCUDAError ("InteractionEngine::applyInteraction nb");
  err.check ("interaction engine nb");	
  if (timer != NULL) timer->toc(mdTimeNonBondedInteraction);
}

void InteractionEngine_interface::
applyBondedInteraction (MDSystem & sys,
			const BondedInteractionList & bdlist,
			MDTimer *timer )
{
  if (hasBond) {
    if (timer != NULL) timer->tic(mdTimeBondedInteraction);
    calBondInteraction
	<<<atomGridDim, myBlockDim>>> (
	    sys.ddata.numAtom,
	    sys.ddata.coord,
	    sys.ddata.forcx,  sys.ddata.forcy,  sys.ddata.forcz,
	    sys.box,
	    bdlist.deviceBondList());
    checkCUDAError ("InteractionEngine::applyInteraction bonded");
    err.check ("interaction engine b");	
    if (timer != NULL) timer->toc(mdTimeBondedInteraction);
  }
  if (hasAngle){
    if (timer != NULL) timer->tic(mdTimeAngleInteraction);
    calAngleInteraction
	<<<atomGridDim, myBlockDim>>> (
	    sys.ddata.numAtom,
	    sys.ddata.coord,
	    sys.ddata.forcx,  sys.ddata.forcy,  sys.ddata.forcz,
	    sys.box,
	    bdlist.deviceAngleList());
    checkCUDAError ("InteractionEngine::applyInteraction angle");
    err.check ("interaction engine angle");	
    if (timer != NULL) timer->toc(mdTimeAngleInteraction);
  }
}
  
void InteractionEngine_interface::
applyNonBondedInteraction (MDSystem & sys,
			   const NeighborList & nlist,
			   MDStatistic & st,
			   MDTimer *timer )
{
  if (timer != NULL) timer->tic(mdTimeNBInterStatistic);
  calNonBondedInteraction
      <<<atomGridDim, myBlockDim>>> (
	  sys.ddata.numAtom,
	  sys.ddata.coord,
	  sys.ddata.forcx,  sys.ddata.forcy,  sys.ddata.forcz,
	  sys.ddata.type, 
	  sys.box, nlist.dnlist
	  ,
	  sum_nb_p.getBuff(),
	  sum_nb_vxx.getBuff(),
	  sum_nb_vyy.getBuff(),
	  sum_nb_vzz.getBuff(),
	  err.ptr_de,
	  err.ptr_dindex,
	  err.ptr_dscalor
	  );
  checkCUDAError ("InteractionEngine::applyInteraction nb (with statistic)");
  err.check ("interaction engine nb");	
  hipDeviceSynchronize();
  sum_nb_p.sumBuffAdd(st.ddata, mdStatisticNonBondedPotential, 0);
  sum_nb_vxx.sumBuffAdd(st.ddata, mdStatisticVirialXX, 1);
  sum_nb_vyy.sumBuffAdd(st.ddata, mdStatisticVirialYY, 2);
  sum_nb_vzz.sumBuffAdd(st.ddata, mdStatisticVirialZZ, 3);
  hipDeviceSynchronize();
  if (timer != NULL) timer->toc(mdTimeNBInterStatistic);
}


void InteractionEngine_interface::
applyBondedInteraction (MDSystem & sys,
			const BondedInteractionList & bdlist,
			MDStatistic & st,
			MDTimer *timer)
{
  if (hasBond) {
    if (timer != NULL) timer->tic(mdTimeBInterStatistic);
    calBondInteraction
	<<<atomGridDim, myBlockDim,
	calBondInteraction_sbuffSize>>> (
	    sys.ddata.numAtom,
	    sys.ddata.coord,
	    sys.ddata.forcx,  sys.ddata.forcy,  sys.ddata.forcz,
	    sys.box,
	    bdlist.deviceBondList()
	    ,
	    sum_b_p.getBuff(),
	    sum_b_vxx.getBuff(),
	    sum_b_vyy.getBuff(),
	    sum_b_vzz.getBuff(),
	    err.ptr_de
	    );
    checkCUDAError ("InteractionEngine::applyInteraction bonded (with statistic)");
    err.check ("interaction engine");	
    if (timer != NULL) timer->toc(mdTimeBInterStatistic);
  }
  if (hasAngle){
    if (timer != NULL) timer->tic(mdTimeAngleInterStatistic);
    calAngleInteraction
	<<<atomGridDim, myBlockDim,
	calAngleInteraction_sbuffSize>>> (
	    sys.ddata.numAtom,
	    sys.ddata.coord,
	    sys.ddata.forcx,  sys.ddata.forcy,  sys.ddata.forcz,
	    sys.box,
	    bdlist.deviceAngleList(),
	    sum_angle_p.getBuff(),
	    err.ptr_de);
    checkCUDAError ("InteractionEngine::applyInteraction angle");
    err.check ("interaction engine angle");	
    if (timer != NULL) timer->toc(mdTimeAngleInterStatistic);
  }
  if (hasBond) {
    if (timer != NULL) timer->tic(mdTimeBInterStatistic);
    hipDeviceSynchronize();
    sum_b_p.sumBuffAdd(st.ddata, mdStatisticBondedPotential, 4);
    sum_b_vxx.sumBuffAdd(st.ddata, mdStatisticVirialXX, 5);
    sum_b_vyy.sumBuffAdd(st.ddata, mdStatisticVirialYY, 6);
    sum_b_vzz.sumBuffAdd(st.ddata, mdStatisticVirialZZ, 7);
    hipDeviceSynchronize();
    if (timer != NULL) timer->toc(mdTimeBInterStatistic);
  }
  if (hasAngle){
    if (timer != NULL) timer->tic(mdTimeAngleInterStatistic);
    sum_angle_p.sumBuffAdd(st.ddata, mdStatisticBondedPotential, 4);
    if (timer != NULL) timer->toc(mdTimeAngleInterStatistic);
  }
  checkCUDAError ("InteractionEngine::applyInteraction sum statistic (with statistic)");
}



__global__ void clearForce (const IndexType numAtom,
			    ScalorType * forcx,
			    ScalorType * forcy, 
			    ScalorType * forcz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  if (ii < numAtom) {
    forcx[ii] = 0.0f;
    forcy[ii] = 0.0f;
    forcz[ii] = 0.0f;
  }
}

__global__ void calNonBondedInteraction (const IndexType numAtom,
					 const CoordType * coord,
					 ScalorType * forcx,
					 ScalorType * forcy, 
					 ScalorType * forcz,
					 const TypeType * type,
					 const RectangularBox box,
					 const DeviceNeighborList nlist,
					 mdError_t * ptr_de,
					 IndexType * errorIndex,
					 ScalorType * errorScalor)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  ScalorType fsumx = 0.0f;
  ScalorType fsumy = 0.0f;
  ScalorType fsumz = 0.0f;
  IndexType ii = tid + bid * blockDim.x;

  // IndexType num = 0;
  // IndexType start = bid * blockDim.x;
  // if (start < numAtom){
  //   if (start + blockDim.x <= numAtom){
  //     num = blockDim.x;
  //   }
  //   else {
  //     num = start + blockDim.x - numAtom;
  //   }
  // }
  // IndexType maxNei = maxVectorBlock (nlist.Nneighbor, start, num);
  // IndexType myNei = nlist.Nneighbor[ii];
  
  if (ii < numAtom) {
#ifdef COMPILE_NO_TEX
    CoordType ref (coord[ii]);
#else
    CoordType ref (tex1Dfetch(global_texRef_interaction_coord, ii));
#endif    
    ScalorType fx(0.f), fy(0.f), fz(0.f);
    for (IndexType jj = 0, nlistPosi = ii;
    	 jj < nlist.Nneighbor[ii];
    	 ++jj, nlistPosi += nlist.stride){

    // for (IndexType jj = 0, nlistPosi = ii;
    // 	 jj < maxNei;
    // 	 ++jj, nlistPosi += nlist.stride){
    //   __syncthreads();
    //   if (jj >= myNei) continue;
      
      IndexType targetIdx ( nlist.data [nlistPosi] );
      ForceIndexType nbForceIndex ( nlist.forceIndex [nlistPosi] );
#ifdef COMPILE_NO_TEX
      CoordType target ( coord[targetIdx] );
#else
      CoordType target ( tex1Dfetch(global_texRef_interaction_coord, targetIdx) );
#endif
      ScalorType diffx ( target.x - ref.x );
      ScalorType diffy ( target.y - ref.y );
      ScalorType diffz ( target.z - ref.z );
      shortestImage (box, &diffx, &diffy, &diffz);
      // ScalorType * forceParam;
      // NBForceSetting::getParam (nbForceIndex, nbForceParam, nbForceParamPosi,
      // 				&forceParam);
      // nbForce (nbForceType[nbForceIndex], forceParam,
      // 	       diffx, diffy, diffz, 
      // 	       &fx, &fy, &fz);
      nbForce (nonBondedInteractionType[nbForceIndex],
	       &nonBondedInteractionParameter
	       [nonBondedInteractionParameterPosition[nbForceIndex]],
      	       diffx, diffy, diffz, 
      	       &fx, &fy, &fz);
      fsumx += fx;
      fsumy += fy;
      fsumz += fz;
    }
    forcx[ii] += fsumx;
    forcy[ii] += fsumy;
    forcz[ii] += fsumz;
  }
}


__global__ void calNonBondedInteraction (const IndexType numAtom,
					 const CoordType * coord,
					 ScalorType * forcx,
					 ScalorType * forcy, 
					 ScalorType * forcz,
					 const TypeType * type,
					 const RectangularBox box,
					 const DeviceNeighborList nlist,
					 ScalorType * statistic_nb_buff0,
					 ScalorType * statistic_nb_buff1,
					 ScalorType * statistic_nb_buff2,
					 ScalorType * statistic_nb_buff3,
					 mdError_t * ptr_de,
					 IndexType * errorIndex,
					 ScalorType * errorScalor)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;

  ScalorType fsumx = 0.0f;
  ScalorType fsumy = 0.0f;
  ScalorType fsumz = 0.0f;
  IndexType ii = tid + bid * blockDim.x;
  ScalorType myPoten = 0.0f, myVxx = 0.0f, myVyy = 0.0f, myVzz = 0.0f;
  if (ii < numAtom) {
    CoordType ref;
#ifdef COMPILE_NO_TEX    
    ref = coord[ii];
#else
    ref = tex1Dfetch(global_texRef_interaction_coord, ii);
#endif
    ScalorType fx(0.f), fy(0.f), fz(0.f);
    ScalorType dp;
    for (IndexType jj = 0, nlistPosi = ii;
	 jj < nlist.Nneighbor[ii];
	 ++jj, nlistPosi += nlist.stride){
      IndexType targetIdx ( nlist.data[nlistPosi] );
      ForceIndexType nbForceIndex ( nlist.forceIndex [nlistPosi] );
#ifdef COMPILE_NO_TEX    
      CoordType target ( coord[targetIdx] );
#else
      CoordType target ( tex1Dfetch(global_texRef_interaction_coord, targetIdx) );
#endif
      ScalorType diffx ( target.x - ref.x );
      ScalorType diffy ( target.y - ref.y );
      ScalorType diffz ( target.z - ref.z );
      shortestImage (box, &diffx, &diffy, &diffz);
      // ScalorType * forceParam;
      // NBForceSetting::getParam (nbForceIndex, nbForceParam, nbForceParamPosi,
      // 				&forceParam);
      // nbForcePoten (nbForceType[nbForceIndex],
      // 		    forceParam,
      // 		    diffx, diffy, diffz, 
      // 		    &fx, &fy, &fz, &dp);
      nbForcePoten (nonBondedInteractionType[nbForceIndex],
		    &nonBondedInteractionParameter
		    [nonBondedInteractionParameterPosition[nbForceIndex]],
      		    diffx, diffy, diffz, 
      		    &fx, &fy, &fz, &dp);
      myPoten += dp;
      myVxx += fx * diffx;
      myVyy += fy * diffy;
      myVzz += fz * diffz;
      fsumx += fx;
      fsumy += fy;
      fsumz += fz;
    }
    forcx[ii] += fsumx;
    forcy[ii] += fsumy;
    forcz[ii] += fsumz;
  }
  
  if (ii < numAtom){
    statistic_nb_buff0[ii] = myPoten * 0.5f;
    statistic_nb_buff1[ii] = myVxx * 0.5f;
    statistic_nb_buff2[ii] = myVyy * 0.5f;
    statistic_nb_buff3[ii] = myVzz * 0.5f;
  }  
}


__global__ void calBondInteraction (const IndexType numAtom,
				    const CoordType * coord,
				    ScalorType * forcx,
				    ScalorType * forcy, 
				    ScalorType * forcz,
				    const RectangularBox box,
				    const DeviceBondList bdlist)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  ScalorType fsumx = 0.0f;
  ScalorType fsumy = 0.0f;
  ScalorType fsumz = 0.0f;
  IndexType ii = tid + bid * blockDim.x;
  
  if (ii >= numAtom) return;
  CoordType ref;
#ifdef COMPILE_NO_TEX
  ref = coord[ii];
#else
  ref = tex1Dfetch(global_texRef_interaction_coord, ii);
#endif  
      
  IndexType myNumBond = bdlist.numBond[ii];
  
  for (IndexType jj = 0; jj < bdlist.maxNumBond; ++jj){
    if (jj == myNumBond) break;
    IndexType targetIdx = bdlist.bondNeighborIndex[jj * bdlist.stride + ii];
    CoordType target;
#ifdef COMPILE_NO_TEX
    target = coord[targetIdx];
#else
    target = tex1Dfetch(global_texRef_interaction_coord, targetIdx);
#endif 
    ScalorType diffx, diffy, diffz;
    diffx = target.x - ref.x;
    diffy = target.y - ref.y;
    diffz = target.z - ref.z;
    shortestImage (box, &diffx, &diffy, &diffz);
    ScalorType fx, fy, fz;
    ForceIndexType bondFindex = bdlist.bondIndex[jj * bdlist.stride + ii];
    bondForce (bondedInteractionType[bondFindex],
	       &bondedInteractionParameter
	       [bondedInteractionParameterPosition[bondFindex]],
	       diffx, diffy, diffz, &fx, &fy, &fz);
    fsumx += fx;
    fsumy += fy;
    fsumz += fz;
  }
  forcx[ii] += fsumx;
  forcy[ii] += fsumy;
  forcz[ii] += fsumz;
}


__global__ void calBondInteraction (const IndexType numAtom,
				    const CoordType * coord,
				    ScalorType * forcx,
				    ScalorType * forcy, 
				    ScalorType * forcz,
				    const RectangularBox box,
				    const DeviceBondList bdlist,
				    ScalorType * statistic_b_buff0,
				    ScalorType * statistic_b_buff1,
				    ScalorType * statistic_b_buff2,
				    ScalorType * statistic_b_buff3,
				    mdError_t * ptr_de)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;

  extern __shared__ volatile ScalorType buff[];
  buff[tid] = 0.f;
  __syncthreads();
  
  ScalorType fsumx = 0.0f;
  ScalorType fsumy = 0.0f;
  ScalorType fsumz = 0.0f;
  IndexType ii = tid + bid * blockDim.x;
  ScalorType myPoten = 0.0f, myVxx = 0.0f, myVyy = 0.0f, myVzz = 0.0f;
  if (ii < numAtom) {
    CoordType ref;
#ifdef COMPILE_NO_TEX
    ref = coord[ii];
#else 
    ref = tex1Dfetch(global_texRef_interaction_coord, ii);
#endif
    IndexType myNumBond = bdlist.numBond[ii];
    for (IndexType jj = 0; jj < bdlist.maxNumBond; ++jj){
      if (jj == myNumBond) break;
      IndexType targetIdx = bdlist.bondNeighborIndex[jj * bdlist.stride + ii];
      CoordType target;
#ifdef COMPILE_NO_TEX
      target = coord[targetIdx];
#else
      target = tex1Dfetch(global_texRef_interaction_coord, targetIdx);
#endif
      ScalorType diffx, diffy, diffz;
      diffx = target.x - ref.x;
      diffy = target.y - ref.y;
      diffz = target.z - ref.z;
      shortestImage (box, &diffx, &diffy, &diffz);
      ScalorType fx, fy, fz;
      ForceIndexType bondFindex = bdlist.bondIndex[jj * bdlist.stride + ii];
      ScalorType dp;
      bondForcePoten (bondedInteractionType[bondFindex],
		      &bondedInteractionParameter
		      [bondedInteractionParameterPosition[bondFindex]],
		      diffx, diffy, diffz, &fx, &fy, &fz, &dp);
      myPoten += dp;
      myVxx += fx * diffx;
      myVyy += fy * diffy;
      myVzz += fz * diffz;
      fsumx += fx;
      fsumy += fy;
      fsumz += fz;
    }
    forcx[ii] += fsumx;
    forcy[ii] += fsumy;
    forcz[ii] += fsumz;
  }

  buff[tid] = myPoten * 0.5f;
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_b_buff0[bid] = buff[0];
  __syncthreads();
  buff[tid] = myVxx * 0.5f;
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_b_buff1[bid] = buff[0];
  __syncthreads();
  buff[tid] = myVyy * 0.5f;
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_b_buff2[bid] = buff[0];
  __syncthreads();
  buff[tid] = myVzz * 0.5f;
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_b_buff3[bid] = buff[0];
  __syncthreads();
}



__global__ void calAngleInteraction (const IndexType numAtom,
				     const CoordType * coord,
				     ScalorType * forcx,
				     ScalorType * forcy, 
				     ScalorType * forcz,
				     const RectangularBox box,
				     const DeviceAngleList anglelist)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  ScalorType fsumx = 0.0f;
  ScalorType fsumy = 0.0f;
  ScalorType fsumz = 0.0f;
  IndexType ii = tid + bid * blockDim.x;
  IndexType myNumAngle;
  
  if (ii < numAtom){
    myNumAngle = anglelist.numAngle[ii];  
  }
  else {
    myNumAngle = 0;
  }
  if (__all(myNumAngle == 0)) return ;
  
  if (ii < numAtom) {
    CoordType ref;
#ifdef COMPILE_NO_TEX
    ref = coord[ii];
#else
    ref = tex1Dfetch(global_texRef_interaction_coord, ii);
#endif
    for (IndexType jj = 0; jj < myNumAngle; ++jj){
      IndexType targetIdx0 =
	  anglelist.angleNeighborIndex[((jj<<1)  ) * anglelist.stride + ii];
      IndexType targetIdx1 =
	  anglelist.angleNeighborIndex[((jj<<1)+1) * anglelist.stride + ii];
      IndexType myPosi     =
	  anglelist.anglePosi[jj * anglelist.stride + ii];
      CoordType target0, target1;
#ifdef COMPILE_NO_TEX
      target0 = coord[targetIdx0];
      target1 = coord[targetIdx1];
#else
      target0 = tex1Dfetch(global_texRef_interaction_coord, targetIdx0);
      target1 = tex1Dfetch(global_texRef_interaction_coord, targetIdx1);
#endif 
      ScalorType diff0x, diff0y, diff0z;
      ScalorType diff1x, diff1y, diff1z;
      bool center (myPosi == 1);
      if (center){
	diff0x = ref.x - target0.x;
	diff0y = ref.y - target0.y;
	diff0z = ref.z - target0.z;
	diff1x = target1.x -  ref.x;
	diff1y = target1.y -  ref.y;
	diff1z = target1.z -  ref.z;
      } else {
	diff0x = target0.x - ref.x;
	diff0y = target0.y - ref.y;
	diff0z = target0.z - ref.z;
	diff1x = target1.x - target0.x;
	diff1y = target1.y - target0.y;
	diff1z = target1.z - target0.z;
      }      
      shortestImage (box, &diff0x, &diff0y, &diff0z);
      shortestImage (box, &diff1x, &diff1y, &diff1z);
      ScalorType fx, fy, fz;
      ForceIndexType angleFindex = anglelist.angleIndex[jj * anglelist.stride + ii];
      angleForce (center,
		  bondedInteractionType[angleFindex],
		  &bondedInteractionParameter
		  [bondedInteractionParameterPosition[angleFindex]],
		  diff0x, diff0y, diff0z,
		  diff1x, diff1y, diff1z,
		  &fx, &fy, &fz);
      fsumx += fx;
      fsumy += fy;
      fsumz += fz;
    }
    forcx[ii] += fsumx;
    forcy[ii] += fsumy;
    forcz[ii] += fsumz;
  }
}


__global__ void calAngleInteraction (const IndexType numAtom,
				     const CoordType * coord,
				     ScalorType * forcx,
				     ScalorType * forcy, 
				     ScalorType * forcz,
				     const RectangularBox box,
				     const DeviceAngleList anglelist,
				     ScalorType * statistic_b_buff0,
				     mdError_t * ptr_de)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  ScalorType fsumx = 0.0f;
  ScalorType fsumy = 0.0f;
  ScalorType fsumz = 0.0f;
  ScalorType myPoten = 0.f;
  IndexType ii = tid + bid * blockDim.x;
  IndexType myNumAngle;
  extern __shared__ volatile ScalorType buff[];
  buff[tid] = 0.f;
  __syncthreads();
  
  if (ii < numAtom) {
    CoordType ref;
#ifdef COMPILE_NO_TEX
    ref = coord[ii];
#else
    ref = tex1Dfetch(global_texRef_interaction_coord, ii);
#endif
    myNumAngle = anglelist.numAngle[ii];  
    for (IndexType jj = 0; jj < myNumAngle; ++jj){
      IndexType targetIdx0 =
	  anglelist.angleNeighborIndex[((jj<<1)  ) * anglelist.stride + ii];
      IndexType targetIdx1 =
	  anglelist.angleNeighborIndex[((jj<<1)+1) * anglelist.stride + ii];
      IndexType myPosi     =
	  anglelist.anglePosi[jj * anglelist.stride + ii];
      CoordType target0, target1;
#ifdef COMPILE_NO_TEX
      target0 = coord[targetIdx0];
      target1 = coord[targetIdx1];
#else
      target0 = tex1Dfetch(global_texRef_interaction_coord, targetIdx0);
      target1 = tex1Dfetch(global_texRef_interaction_coord, targetIdx1);
#endif 
      ScalorType diff0x, diff0y, diff0z;
      ScalorType diff1x, diff1y, diff1z;
      bool center = (myPosi == 1);
      if (center){
	diff0x = ref.x - target0.x;
	diff0y = ref.y - target0.y;
	diff0z = ref.z - target0.z;
	diff1x = target1.x -  ref.x;
	diff1y = target1.y -  ref.y;
	diff1z = target1.z -  ref.z;
      } else {
	diff0x = target0.x - ref.x;
	diff0y = target0.y - ref.y;
	diff0z = target0.z - ref.z;
	diff1x = target1.x - target0.x;
	diff1y = target1.y - target0.y;
	diff1z = target1.z - target0.z;
      }      
      shortestImage (box, &diff0x, &diff0y, &diff0z);
      shortestImage (box, &diff1x, &diff1y, &diff1z);
      ScalorType fx, fy, fz;
      ForceIndexType angleFindex = anglelist.angleIndex[jj * anglelist.stride + ii];
      ScalorType dp;
      angleForcePoten (center,
		       bondedInteractionType[angleFindex],
		       &bondedInteractionParameter
		       [bondedInteractionParameterPosition[angleFindex]],
		       diff0x, diff0y, diff0z,
		       diff1x, diff1y, diff1z,
		       &fx, &fy, &fz, &dp);
      myPoten += dp;
      fsumx += fx;
      fsumy += fy;
      fsumz += fz;
    }
    forcx[ii] += fsumx;
    forcy[ii] += fsumy;
    forcz[ii] += fsumz;
  }

  buff[tid] = myPoten * 0.33333333333333333f;
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_b_buff0[bid] = buff[0];
}



