#include "hip/hip_runtime.h"
#define DEVICE_CODE

#include "Parallel_Integrator.h"
#include "Parallel_Interface.h"
#include "Parallel_Auxiliary.h"
#include "compile_error_mixcode.h"

Parallel::TranslationalFreedomRemover::
~TranslationalFreedomRemover ()
{
  if (malloced){
    hipFree (sums);
  }
}

void Parallel::TranslationalFreedomRemover::
reinit  (const DeviceCellListedMDData & data)
{
  IndexType totalNumCell = data.getNumCell().x * data.getNumCell().y * data.getNumCell().z;
  gridDim = toGridDim (totalNumCell);
  numThreadsInCell = Parallel::Interface::numThreadsInCell();
  sharedBuffSize = numThreadsInCell * sizeof(ScalorType);

  sum_x.reinit (totalNumCell, NThreadForSum);
  sum_y.reinit (totalNumCell, NThreadForSum);
  sum_z.reinit (totalNumCell, NThreadForSum);

  if (! malloced){
    hipMalloc ((void**)&sums, 3 * sizeof(ScalorType));
    Parallel::Auxiliary::setValue <<<1, 3>>> (sums, 3, ScalorType (0.f));
    hipMalloc ((void**)&sumM, 1 * sizeof(ScalorType));
    checkCUDAError ("TranslationalFreedomRemover::reinit, malloc sums");
    malloced = true;
  }

  SumVector<ScalorType > sum_mass;
  sum_mass.reinit (totalNumCell, NThreadForSum);
  Parallel::CudaGlobal::prepareCalTotalMass
      <<<gridDim, numThreadsInCell, sharedBuffSize>>>(
	  data.dptr_numAtomInCell(),
	  data.dptr_mass(),
	  sum_mass.getBuff());
  sum_mass.sumBuff (sumM, 0);
  hipMemcpy (&totalMassi, sumM, sizeof(ScalorType), hipMemcpyDeviceToHost);
  checkCUDAError ("TranslationalFreedomRemover::reinit, cpy sumM");
  totalMassi = 1.f/totalMassi;
}

void Parallel::TranslationalFreedomRemover::
remove (DeviceCellListedMDData & data)
{
  Parallel::CudaGlobal::prepareRemoveTranslationalFreedom
      <<<gridDim, numThreadsInCell, sharedBuffSize>>> (
	  data.dptr_numAtomInCell(),
	  data.dptr_mass(),
	  data.dptr_velocityX(),
	  data.dptr_velocityY(),
	  data.dptr_velocityZ(),
	  sum_x.getBuff(),
	  sum_y.getBuff(),
	  sum_z.getBuff());
  checkCUDAError ("TranslationalFreedomRemover::remove, prepare");
  sum_x.sumBuff (sums, 0);
  sum_y.sumBuff (sums, 1);
  sum_z.sumBuff (sums, 2);
  Parallel::CudaGlobal::removeTranslationalFreedom
      <<<gridDim, numThreadsInCell>>> (
	  data.dptr_numAtomInCell(),
	  totalMassi,
	  sums,
	  data.dptr_velocityX(),
	  data.dptr_velocityY(),
	  data.dptr_velocityZ());
  checkCUDAError("TranslationalFreedomRemover::remove, remove");
}


__global__ void Parallel::CudaGlobal::
prepareCalTotalMass (const IndexType * numAtomInCell,
		     const ScalorType * mass,
		     ScalorType * mass_buff)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;

  IndexType this_numAtomInCell = numAtomInCell[bid];
  if (this_numAtomInCell == 0){
    if (threadIdx.x == 0){
      mass_buff[bid] = 0;
    }
    return;
  }
  extern __shared__ ScalorType buff[];
  if (threadIdx.x < this_numAtomInCell){
    buff[threadIdx.x] = mass[ii];
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  __syncthreads ();
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) mass_buff[bid] = buff[0];
}

__global__ void Parallel::CudaGlobal::
prepareRemoveTranslationalFreedom (const IndexType * numAtomInCell,
				   const ScalorType * mass,
				   const ScalorType * velox,
				   const ScalorType * veloy,
				   const ScalorType * veloz,
				   ScalorType * st_buff_x,
				   ScalorType * st_buff_y,
				   ScalorType * st_buff_z)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;  
  IndexType this_numAtomInCell = numAtomInCell[bid];
  
  if (this_numAtomInCell == 0){
    if (threadIdx.x == 0){
      st_buff_x[bid] = 0;
      st_buff_y[bid] = 0;
      st_buff_z[bid] = 0;
    }
    return;
  }

  extern __shared__  ScalorType buff[];
  if (threadIdx.x < this_numAtomInCell){
    buff[threadIdx.x] = mass[ii] * velox[ii];
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  __syncthreads();
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) st_buff_x[bid] = buff[0];
  __syncthreads();

  if (threadIdx.x < this_numAtomInCell){
    buff[threadIdx.x] = mass[ii] * veloy[ii];
  }
  __syncthreads();
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) st_buff_y[bid] = buff[0];
  __syncthreads();

  if (threadIdx.x < this_numAtomInCell){
    buff[threadIdx.x] = mass[ii] * veloz[ii];
  }
  __syncthreads();
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) st_buff_z[bid] = buff[0];
}

__global__ void Parallel::CudaGlobal::
removeTranslationalFreedom (const IndexType * numAtomInCell,
			    const ScalorType totalMassi,
			    const ScalorType * sums,
			    ScalorType * velox,
			    ScalorType * veloy,
			    ScalorType * veloz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;  
  IndexType this_numAtomInCell = numAtomInCell[bid];
  
  if (this_numAtomInCell == 0){
    return;
  }

  __shared__ ScalorType buffSums[3];
  if (threadIdx.x < 3){
    buffSums[threadIdx.x] = sums[threadIdx.x];
  }
  __syncthreads();

  if (threadIdx.x < this_numAtomInCell){
    velox[ii] -= buffSums[0] * totalMassi;
    veloy[ii] -= buffSums[1] * totalMassi;
    veloz[ii] -= buffSums[2] * totalMassi;
  }
}


void Parallel::Integrator::VelocityVerlet::
init (const DeviceCellListedMDData & ddata)
{
  IndexType totalNumCell = ddata.getNumCell().x *
      ddata.getNumCell().y * ddata.getNumCell().z;
  gridDim = toGridDim (totalNumCell);

  sum_kxx.init (totalNumCell, NThreadForSum);
  sum_kyy.init (totalNumCell, NThreadForSum);
  sum_kzz.init (totalNumCell, NThreadForSum);
  
  sharedBuffSize = Parallel::Interface::numThreadsInCell() * sizeof(ScalorType);
}

void Parallel::Integrator::VelocityVerlet::
step1 (DeviceCellListedMDData & ddata,
       const ScalorType & dt)
{
  Parallel::CudaGlobal::velocityVerlet_step1
      <<<gridDim, Parallel::Interface::numThreadsInCell()>>>(
	  ddata.dptr_numAtomInCell(),
	  ddata.dptr_forceX(),
	  ddata.dptr_forceY(),
	  ddata.dptr_forceZ(),
	  ddata.dptr_mass(),
	  dt,
	  ddata.dptr_coordinate(),
	  ddata.dptr_velocityX(),
	  ddata.dptr_velocityY(),
	  ddata.dptr_velocityZ());
  checkCUDAError ("Integrator::VelocityVerlet::step1");	  
}

void Parallel::Integrator::VelocityVerlet::
step2 (DeviceCellListedMDData & data,
       const ScalorType & dt)
{
  Parallel::CudaGlobal::velocityVerlet_step2
      <<<gridDim, Parallel::Interface::numThreadsInCell()>>>(
	  data.dptr_numAtomInCell(),
	  data.dptr_forceX(),
	  data.dptr_forceY(),
	  data.dptr_forceZ(),
	  data.dptr_mass(),
	  dt,
	  data.dptr_velocityX(),
	  data.dptr_velocityY(),
	  data.dptr_velocityZ());
  checkCUDAError ("interface::VelocityVerlet::step2, no st");
}

void Parallel::Integrator::VelocityVerlet::
step2 (DeviceCellListedMDData & data,
       const ScalorType & dt,
       DeviceStatistic & st)
{
  Parallel::CudaGlobal::velocityVerlet_step2
      <<<gridDim, Parallel::Interface::numThreadsInCell(), sharedBuffSize>>> (
	  data.dptr_numAtomInCell(),
	  data.dptr_forceX(),
	  data.dptr_forceY(),
	  data.dptr_forceZ(),
	  data.dptr_mass(),
	  dt,
	  data.dptr_velocityX(),
	  data.dptr_velocityY(),
	  data.dptr_velocityZ(),
	  sum_kxx.getBuff(),
	  sum_kyy.getBuff(),
	  sum_kzz.getBuff());
  sum_kxx.sumBuffAdd (st.dptr_statisticData(), mdStatistic_KineticEnergyXX, 0);
  sum_kyy.sumBuffAdd (st.dptr_statisticData(), mdStatistic_KineticEnergyYY, 0);
  sum_kzz.sumBuffAdd (st.dptr_statisticData(), mdStatistic_KineticEnergyZZ, 0);

  checkCUDAError ("interface::VelocityVerlet::step2, with st");
}
	  

	  

__global__ void Parallel::CudaGlobal::
velocityVerlet_step1 (const IndexType * numAtomInCell,
		      const ScalorType * forcx,
		      const ScalorType * forcy,
		      const ScalorType * forcz,
		      const ScalorType * mass,
		      const ScalorType   dt,
		      CoordType * coord,
		      ScalorType * velox,
		      ScalorType * veloy,
		      ScalorType * veloz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;

  if (tid < numAtomInCell[bid]){
    ScalorType hdtmi = 0.5 * dt / mass[ii];
    velox[ii]   += hdtmi * forcx[ii];
    coord[ii].x += dt * velox[ii];
    veloy[ii]   += hdtmi * forcy[ii];
    coord[ii].y += dt * veloy[ii];
    veloz[ii]   += hdtmi * forcz[ii];
    coord[ii].z += dt * veloz[ii];
  }
}

__global__ void Parallel::CudaGlobal::
velocityVerlet_step2 (const IndexType * numAtomInCell,
		      const ScalorType * forcx,
		      const ScalorType * forcy,
		      const ScalorType * forcz,
		      const ScalorType * mass,
		      const ScalorType   dt,
		      ScalorType * velox,
		      ScalorType * veloy,
		      ScalorType * veloz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;

  if (tid < numAtomInCell[bid]){
    ScalorType hdtmi = 0.5f * dt / mass[ii];
    velox[ii] += hdtmi * forcx[ii];
    veloy[ii] += hdtmi * forcy[ii];
    veloz[ii] += hdtmi * forcz[ii];
  }
}

__global__ void Parallel::CudaGlobal::
velocityVerlet_step2 (const IndexType * numAtomInCell,
		      const ScalorType * forcx,
		      const ScalorType * forcy,
		      const ScalorType * forcz,
		      const ScalorType * mass,
		      const ScalorType   dt,
		      ScalorType * velox,
		      ScalorType * veloy,
		      ScalorType * veloz,
		      ScalorType * statistic_buffxx,
		      ScalorType * statistic_buffyy,
		      ScalorType * statistic_buffzz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  
  extern __shared__  ScalorType buff [];
  ScalorType vx(0.f), vy(0.f), vz(0.f);
  IndexType this_numAtomInCell = numAtomInCell[bid];
  
  if (threadIdx.x < this_numAtomInCell) {
    ScalorType hdtmi = 0.5f * dt / mass[ii];
    vx = (velox[ii] += hdtmi * forcx[ii]);
    vy = (veloy[ii] += hdtmi * forcy[ii]);
    vz = (veloz[ii] += hdtmi * forcz[ii]);
  }

  ScalorType scalor = 0.5f * mass[ii];
  // if (threadIdx.x < this_numAtomInCell) scalor = 0.5f * mass[ii];
  // else scalor = 0.f;
  
  buff[threadIdx.x] = scalor * vx * vx;
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffxx[bid] = buff[0];
  __syncthreads();
  buff[threadIdx.x] = scalor * vy * vy;
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffyy[bid] = buff[0];
  __syncthreads();
  buff[threadIdx.x] = scalor * vz * vz;
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffzz[bid] = buff[0];
}

  
