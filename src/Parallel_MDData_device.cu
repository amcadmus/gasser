#include "hip/hip_runtime.h"
#define DEVICE_CODE
#include "Parallel_MDData.h"
#include "Parallel_MDData_device.h"

#include "compile_error_mixcode.h"

Parallel::DeviceMDData::
DeviceMDData ()
    : numData_ (0), memSize_(0), malloced(false)
{
}

Parallel::DeviceMDData::
~DeviceMDData ()
{
  clear();
}

void Parallel::DeviceMDData::
easyMalloc (const IndexType & memSize__)
{
  if (memSize__ == 0) return;
  clear ();

  memSize_ = memSize__;
  
  size_t sizef = memSize_ * sizeof(ScalorType);
  size_t sizecoord =memSize_ * sizeof(CoordType);
  size_t sizecoordNoi =memSize_ * sizeof(CoordNoiType);
  size_t sizeIdx = memSize_ * sizeof(IndexType);
  size_t sizet = memSize_ * sizeof(TypeType);
  
  hipMalloc ((void**) &coord, sizecoord);
  checkCUDAError ("initDeviceMDData coord");

  hipMalloc ((void**) &coordNoi, sizecoordNoi);
  checkCUDAError ("initDeviceMDData coordNoi");

  hipMalloc ((void**) &velox, sizef);
  hipMalloc ((void**) &veloy, sizef);
  hipMalloc ((void**) &veloz, sizef);
  checkCUDAError ("initDeviceMDData velo");

  hipMalloc ((void**) &forcx, sizef);
  hipMalloc ((void**) &forcy, sizef);
  hipMalloc ((void**) &forcz, sizef);
  checkCUDAError ("initDeviceMDData forc");

  hipMalloc ((void**) &globalIndex, sizeIdx);
  hipMalloc ((void**) &type, sizet);
  hipMalloc ((void**) &mass, sizef);
  hipMalloc ((void**) &charge, sizef);
  checkCUDAError ("initDeviceMDData top Property");

  malloced = true;
}

void Parallel::DeviceMDData::
clear ()
{
  if (malloced){
    hipFree (coord);
    hipFree (coordNoi);
  
    hipFree (velox);
    hipFree (veloy);
    hipFree (veloz);
  
    hipFree (forcx);
    hipFree (forcy);
    hipFree (forcz);

    hipFree (globalIndex);
    hipFree (type);
    hipFree (mass);
    hipFree (charge);

    memSize_ = 0;
    malloced = false;
  }
}


void Parallel::DeviceMDData::
copyFromHost (const HostMDData & hdata,
	      const MDDataItemMask_t mask)
{
  if (memSize_ < hdata.numData()){
    easyMalloc (hdata.numData() * MemAllocExtension);
  }
  numData_ = hdata.numData();
  setGlobalBox (hdata.getGlobalBox());
		
  size_t sizef = numData_ * sizeof(ScalorType);
  size_t sizecoord = numData_ * sizeof(CoordType);
  size_t sizecoordNoi = numData_ * sizeof(CoordNoiType);
  size_t sizeIdx = numData_ * sizeof(IndexType);
  size_t sizet = numData_ * sizeof(TypeType);

  if (mask & MDDataItemMask_Coordinate){
    hipMemcpy (coord, hdata.coord, sizecoord, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice coord");
  }

  if (mask & MDDataItemMask_CoordinateNoi){
    hipMemcpy (coordNoi, hdata.coordNoi, sizecoordNoi, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice coordNoi");
  }

  if (mask & MDDataItemMask_Velocity){
    hipMemcpy (velox, hdata.velox, sizef, hipMemcpyHostToDevice);
    hipMemcpy (veloy, hdata.veloy, sizef, hipMemcpyHostToDevice);
    hipMemcpy (veloz, hdata.veloz, sizef, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice velo");
  }

  if (mask & MDDataItemMask_Force){
    hipMemcpy (forcx, hdata.forcx, sizef, hipMemcpyHostToDevice);
    hipMemcpy (forcy, hdata.forcy, sizef, hipMemcpyHostToDevice);
    hipMemcpy (forcz, hdata.forcz, sizef, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice forc");
  }

  if (mask & MDDataItemMask_GlobalIndex){
    hipMemcpy (globalIndex, hdata.globalIndex, sizeIdx, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice globalIndex");
  }
  if (mask & MDDataItemMask_Type){
    hipMemcpy (type, hdata.type, sizet, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice type");
  }
  if (mask & MDDataItemMask_Mass){
    hipMemcpy (mass, hdata.mass, sizef, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice mass");
  }
  if (mask & MDDataItemMask_Charge){
    hipMemcpy (charge, hdata.charge, sizef, hipMemcpyHostToDevice);   
    checkCUDAError ("cpyHostMDDataToDevice charge");
  }
}


void Parallel::DeviceMDData::
copyToHost (HostMDData & hdata,
	    const MDDataItemMask_t mask) const
{
  if (hdata.memSize() < numData_){
    hdata.easyRealloc (numData_ * MemAllocExtension);
  }
  hdata.numData_ = numData_;
  hdata.setGlobalBox (globalBox);
  
  size_t sizef = numData_ * sizeof(ScalorType);
  size_t sizecoord = numData_ * sizeof(CoordType);
  size_t sizecoordNoi = numData_ * sizeof(CoordNoiType);
  size_t sizeIdx = numData_ * sizeof(IndexType);
  size_t sizet = numData_ * sizeof(TypeType);

  if (mask & MDDataItemMask_Coordinate){
    hipMemcpy (hdata.coord, coord, sizecoord, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost coord");
  }

  if (mask & MDDataItemMask_CoordinateNoi){
    hipMemcpy (hdata.coordNoi, coordNoi, sizecoordNoi, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost coordNoi");
  }

  if (mask & MDDataItemMask_Velocity){
    hipMemcpy (hdata.velox, velox, sizef, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.veloy, veloy, sizef, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.veloz, veloz, sizef, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost velo");
  }

  if (mask & MDDataItemMask_Force){
    hipMemcpy (hdata.forcx, forcx, sizef, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.forcy, forcy, sizef, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.forcz, forcz, sizef, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost forc");
  }
  
  if (mask & MDDataItemMask_GlobalIndex){
    hipMemcpy (hdata.globalIndex, globalIndex, sizeIdx, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost globalIndex");
  }
  if (mask & MDDataItemMask_Type){
    hipMemcpy (hdata.type, type, sizet, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost type");
  }
  if (mask & MDDataItemMask_Mass){
    hipMemcpy (hdata.mass, mass, sizef, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost mass");
  }
  if (mask & MDDataItemMask_Charge){
    hipMemcpy (hdata.charge, charge, sizef, hipMemcpyDeviceToHost);  
    checkCUDAError ("cpyDeviceMDDataToHost charge");
  }
}


void Parallel::DeviceMDData::
copyFromDevice (const DeviceMDData & ddata,
		const MDDataItemMask_t mask)
{
  if (memSize_ < ddata.numData()){
    easyMalloc (ddata.numData() * MemAllocExtension);
  }
  numData_ = ddata.numData();
  setGlobalBox (ddata.getGlobalBox());
		
  size_t sizef = numData_ * sizeof(ScalorType);
  size_t sizecoord = numData_ * sizeof(CoordType);
  size_t sizecoordNoi = numData_ * sizeof(CoordNoiType);
  size_t sizeIdx = numData_ * sizeof(IndexType);
  size_t sizet = numData_ * sizeof(TypeType);

  if (mask & MDDataItemMask_Coordinate){
    hipMemcpy (coord, ddata.coord, sizecoord, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice coord");
  }

  if (mask & MDDataItemMask_CoordinateNoi){
    hipMemcpy (coordNoi, ddata.coordNoi, sizecoordNoi, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice coordNoi");
  }

  if (mask & MDDataItemMask_Velocity){
    hipMemcpy (velox, ddata.velox, sizef, hipMemcpyDeviceToDevice);
    hipMemcpy (veloy, ddata.veloy, sizef, hipMemcpyDeviceToDevice);
    hipMemcpy (veloz, ddata.veloz, sizef, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice velo");
  }

  if (mask & MDDataItemMask_Force){
    hipMemcpy (forcx, ddata.forcx, sizef, hipMemcpyDeviceToDevice);
    hipMemcpy (forcy, ddata.forcy, sizef, hipMemcpyDeviceToDevice);
    hipMemcpy (forcz, ddata.forcz, sizef, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice forc");
  }

  if (mask & MDDataItemMask_GlobalIndex){
    hipMemcpy (globalIndex, ddata.globalIndex, sizeIdx, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice globalIndex");
  }
  if (mask & MDDataItemMask_Type){
    hipMemcpy (type, ddata.type, sizet, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice globalIndex");
  }
  if (mask & MDDataItemMask_Mass){
    hipMemcpy (mass, ddata.mass, sizef, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice globalIndex");
  }
  if (mask & MDDataItemMask_Charge){
    hipMemcpy (charge, ddata.charge, sizef, hipMemcpyDeviceToDevice);     
    checkCUDAError ("cpyDeviceMDDataToDevice globalIndex");
  }
}


Parallel::DeviceMDData::
DeviceMDData (const DeviceMDData & ddata)
    : numData_ (0), memSize_(0), malloced(false)
{
  copyFromDevice (ddata, MDDataItemMask_All);
}


void Parallel::DeviceMDData::
initZero ()
{
  Parallel::CudaGlobal::initZeroDeviceData
      <<<memSize_ / DefaultNThreadPerBlock + 1, DefaultNThreadPerBlock>>>
      (memSize_,
       coord,
       coordNoi,
       velox,
       veloy,
       veloz,
       forcx,
       forcy,
       forcz,
       globalIndex,
       type,
       mass,
       charge);
  checkCUDAError ("DeviceMDData::initZero initZeroDeviceData");
}


__global__ void Parallel::CudaGlobal::
initZeroDeviceData(const IndexType num,
		   CoordType  * coord,
		   CoordNoiType * coordNoi,
		   ScalorType * velox,
		   ScalorType * veloy,
		   ScalorType * veloz,
		   ScalorType * forcx,
		   ScalorType * forcy,
		   ScalorType * forcz,
		   IndexType  * globalIndex,
		   TypeType   * type,
		   ScalorType * mass,
		   ScalorType * charge)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;

  if (ii < num){
    CoordType tmp;
    tmp.x = 0;
    tmp.y = 0;
    tmp.z = 0;
    tmp.w = -1;
    coord[ii] = tmp;
    coordNoi[ii].x = coordNoi[ii].y = coordNoi[ii].z = 0;
    veloz[ii] = veloy[ii] = veloz[ii] = 0.f;
    forcz[ii] = forcy[ii] = forcz[ii] = 0.f;
    globalIndex[ii] = MaxIndexValue;
    type[ii] = 0;
    mass[ii] = 0;
    charge[ii] = 0;
  }
}


void Parallel::GlobalHostMDData::
initWriteData_xtcFile (const char * filename, float prec)
{
  xdfile = NULL;
  xdfile = xdrfile_open (filename, "w");
  if (xdfile == NULL){
    MDExcptCannotOpenFile ("MDSystem::initWriteXtc", filename);
  }
  for (unsigned i = 0; i < 3; ++i){
    for (unsigned j = 0; j < 3; ++j){
      xdbox[i][j] = 0.f;
    }	      
  }
  xdx = (rvec *) malloc (sizeof(rvec) * memSize_);
  if (xdx == NULL){
    MDExcptFailedMallocOnHost ("MDSystem::initWriteXtc", "xdx", sizeof(rvec) * memSize_);
  }
  xdprec = prec;
}


void Parallel::GlobalHostMDData::
writeData_xtcFile (int step, float time)
{
  for (IndexType i = 0; i < numData_; ++i){
    xdx[i][0] = coord[i].x;
    xdx[i][1] = coord[i].y;
    xdx[i][2] = coord[i].z;
  }
  xdbox[0][0] = globalBox.size.x;
  xdbox[1][1] = globalBox.size.y;
  xdbox[2][2] = globalBox.size.z;
  write_xtc (xdfile, numData_, step, time, xdbox, xdx, xdprec);
}

void Parallel::GlobalHostMDData::
endWriteData_xtcFile ()
{
  xdrfile_close(xdfile);
}
