#include "hip/hip_runtime.h"
#define DEVICE_CODE
#include "Parallel_MDData.h"
#include "Parallel_MDData_device.h"
#include "Parallel_Interface.h"
#include "Parallel_Auxiliary.h"

#include "compile_error_mixcode.h"

Parallel::DeviceMDData::
DeviceMDData ()
    : _numData(0),
      _memSize(0),
      maxNumBond(0),
      maxNumAngle(0),
      maxNumDihedral(0)
{
}

Parallel::DeviceMDData::
~DeviceMDData ()
{
  clear();
}

void Parallel::DeviceMDData::
clear ()
{
  if (memSize() != 0){    
    hipFree (coord);
    hipFree (coordNoi);
    checkCUDAError ("DeviceMDData::clear");
  
    hipFree (velox);
    hipFree (veloy);
    hipFree (veloz);
    checkCUDAError ("DeviceMDData::clear");
  
    hipFree (forcx);
    hipFree (forcy);
    hipFree (forcz);
    checkCUDAError ("DeviceMDData::clear");

    hipFree (globalIndex);
    hipFree (type);
    hipFree (mass);
    hipFree (charge);

    _memSize = 0;
    _numData = 0;
    checkCUDAError ("DeviceMDData::clear");
    
    if (maxNumBond != 0){
      hipFree (numBond);
      hipFree (bondNeighbor_globalIndex);
      hipFree (bondIndex);
      maxNumBond = 0;
    }
    checkCUDAError ("DeviceMDData::clearBondTop");
    if (maxNumAngle != 0){
      hipFree (numAngle);
      hipFree (angleNeighbor_globalIndex);
      hipFree (angleIndex);
      hipFree (anglePosi);
      maxNumAngle = 0;
    }
    checkCUDAError ("DeviceMDData::clearBondTop");
    if (maxNumDihedral != 0){
      hipFree (numDihedral);
      hipFree (dihedralNeighbor_globalIndex);
      hipFree (dihedralIndex);
      hipFree (dihedralPosi);
      maxNumDihedral = 0;
    }
    checkCUDAError ("DeviceMDData::clearBondTop");
  }
}

void Parallel::DeviceMDData::
fillZero ()
{
  CoordType coord0;
  coord0.x = coord0.y = coord0.z = coord0.w = 0.f;
  CoordNoiType coordNoi0;
  coordNoi0.x = coordNoi0.y = coordNoi0.z = 0;
  IndexType numThreadsInCell = Parallel::Interface::numThreadsInCell();
  IndexType numBlock = memSize() / numThreadsInCell + 1;
  dim3 gridDim (toGridDim(numBlock));

  using namespace Parallel::Auxiliary;
  
  setValue <<<gridDim, numThreadsInCell>>> (coord, memSize(), coord0);
  setValue <<<gridDim, numThreadsInCell>>> (coordNoi, memSize(), coordNoi0);
  setValue <<<gridDim, numThreadsInCell>>> (velox, memSize(), 0.f);
  setValue <<<gridDim, numThreadsInCell>>> (veloy, memSize(), 0.f);
  setValue <<<gridDim, numThreadsInCell>>> (veloz, memSize(), 0.f);
  setValue <<<gridDim, numThreadsInCell>>> (forcx, memSize(), 0.f);
  setValue <<<gridDim, numThreadsInCell>>> (forcy, memSize(), 0.f);
  setValue <<<gridDim, numThreadsInCell>>> (forcz, memSize(), 0.f);
  setValue <<<gridDim, numThreadsInCell>>> (globalIndex, memSize(), MaxIndexValue);
  setValue <<<gridDim, numThreadsInCell>>> (type, memSize(), TypeType(0));
  setValue <<<gridDim, numThreadsInCell>>> (mass, memSize(), 0.f);
  setValue <<<gridDim, numThreadsInCell>>> (charge, memSize(), 0.f);
  
  if (maxNumBond != 0){
    setValue<<<gridDim, numThreadsInCell>>> (numBond, bondTopStride(), IndexType(0));
    numBlock = bondTopStride() * maxNumBond / numThreadsInCell + 1;
    dim3 gridDim1 (toGridDim(numBlock));
    setValue<<<gridDim1, numThreadsInCell>>> (
	bondIndex, bondTopStride() * maxNumBond, IndexType(0));
    setValue<<<gridDim1, numThreadsInCell>>> (
	bondNeighbor_globalIndex, bondTopStride() * maxNumBond, MaxIndexValue);
  }
  if (maxNumAngle != 0){
    setValue<<<gridDim, numThreadsInCell>>> (numAngle, bondTopStride(), IndexType(0));
    numBlock = bondTopStride() * maxNumAngle / numThreadsInCell + 1;
    dim3 gridDim1 (toGridDim(numBlock));
    numBlock = bondTopStride() * maxNumAngle * 2 / numThreadsInCell + 1;
    dim3 gridDim2 (toGridDim(numBlock));
    setValue<<<gridDim1, numThreadsInCell>>> (
	angleIndex, bondTopStride() * maxNumAngle, IndexType(0));
    setValue<<<gridDim1, numThreadsInCell>>> (
	anglePosi,  bondTopStride() * maxNumAngle, IndexType(0));
    setValue<<<gridDim2, numThreadsInCell>>> (
	angleNeighbor_globalIndex, bondTopStride() * maxNumAngle * 2, MaxIndexValue);
  }
  if (maxNumDihedral != 0){
    setValue<<<gridDim, numThreadsInCell>>> (numDihedral, bondTopStride(), IndexType(0));
    numBlock = bondTopStride() * maxNumDihedral / numThreadsInCell + 1;
    dim3 gridDim1 (toGridDim(numBlock));
    numBlock = bondTopStride() * maxNumDihedral * 3 / numThreadsInCell + 1;
    dim3 gridDim2 (toGridDim(numBlock));
    setValue<<<gridDim1, numThreadsInCell>>> (
	dihedralIndex, bondTopStride() * maxNumDihedral, IndexType(0));
    setValue<<<gridDim1, numThreadsInCell>>> (
	dihedralPosi,  bondTopStride() * maxNumDihedral, IndexType(0));
    setValue<<<gridDim2, numThreadsInCell>>> (
	dihedralNeighbor_globalIndex,
	bondTopStride() * maxNumDihedral * 3,
	MaxIndexValue);
  }
}


void Parallel::DeviceMDData::
easyMalloc (const IndexType memSize_,
	    const IndexType maxNumBond_,
	    const IndexType maxNumAngle_,
	    const IndexType maxNumDihedral_)
{
  clear ();

  _memSize = memSize_;
  maxNumBond = maxNumBond_;
  maxNumAngle = maxNumAngle_;
  maxNumDihedral = maxNumDihedral_;

  if (_memSize == 0) return;
  
  size_t sizef = memSize() * sizeof(ScalorType);
  size_t sizecoord =memSize() * sizeof(CoordType);
  size_t sizecoordNoi =memSize() * sizeof(CoordNoiType);
  size_t sizeIdx = memSize() * sizeof(IndexType);
  size_t sizet = memSize() * sizeof(TypeType);
  
  hipMalloc ((void**) &coord, sizecoord);
  checkCUDAError ("initDeviceMDData coord");

  hipMalloc ((void**) &coordNoi, sizecoordNoi);
  checkCUDAError ("initDeviceMDData coordNoi");

  hipMalloc ((void**) &velox, sizef);
  hipMalloc ((void**) &veloy, sizef);
  hipMalloc ((void**) &veloz, sizef);
  checkCUDAError ("initDeviceMDData velo");

  hipMalloc ((void**) &forcx, sizef);
  hipMalloc ((void**) &forcy, sizef);
  hipMalloc ((void**) &forcz, sizef);
  checkCUDAError ("initDeviceMDData forc");

  hipMalloc ((void**) &globalIndex, sizeIdx);
  hipMalloc ((void**) &type, sizet);
  hipMalloc ((void**) &mass, sizef);
  hipMalloc ((void**) &charge, sizef);
  checkCUDAError ("initDeviceMDData top Property");

  size_t size0 = sizeof(IndexType) * memSize();
  if (maxNumBond != 0){
    size_t size1 = size0 * maxNumBond;
    hipMalloc ((void**)&numBond, size0);
    hipMalloc ((void**)&bondIndex, size1);
    hipMalloc ((void**)&bondNeighbor_globalIndex, size1);
    checkCUDAError ("DeviceMDData::easyMallocBondTop, bond");
  }
  if (maxNumAngle != 0){
    size_t size1 = size0 * maxNumAngle;
    size_t size2 = size0 * maxNumAngle * 2;
    hipMalloc ((void**)&numAngle, size0);
    hipMalloc ((void**)&angleIndex, size1);
    hipMalloc ((void**)&anglePosi, size1);
    hipMalloc ((void**)&angleNeighbor_globalIndex, size2);
    checkCUDAError ("DeviceMDData::easyMallocAngleTop, angle");
  }
  if (maxNumDihedral != 0){
    size_t size1 = size0 * maxNumDihedral;
    size_t size2 = size0 * maxNumDihedral * 3;
    hipMalloc ((void**)&numDihedral, size0);
    hipMalloc ((void**)&dihedralIndex, size1);
    hipMalloc ((void**)&dihedralPosi, size1);
    hipMalloc ((void**)&dihedralNeighbor_globalIndex, size2);
    checkCUDAError ("DeviceMDData::easyMallocDihedralTop, dihedral");
  }

  fillZero();
}


void Parallel::DeviceMDData::
copyFromHost (const HostMDData & hdata,
	      const MDDataItemMask_t mask)
{
  // if (!mask) return;
  IndexType expectedNumBond(0), expectedNumAngle(0), expectedNumDihedral(0);
  bool copyBond = (mask & MDDataItemMask_Bond);
  bool copyAngle = (mask & MDDataItemMask_Angle);
  bool copyDihedral = (mask & MDDataItemMask_Dihedral);  
  if (copyBond){
    expectedNumBond = hdata.getMaxNumBond();
  }
  if (copyAngle){
    expectedNumAngle = hdata.getMaxNumAngle();
  }
  if (copyDihedral){
    expectedNumDihedral = hdata.getMaxNumDihedral();
  }

  if (memSize() != hdata.memSize() ||
      (copyBond && (maxNumBond != hdata.maxNumBond)) ||
      (copyAngle && (maxNumAngle != hdata.maxNumAngle)) ||
      (copyDihedral && (maxNumDihedral != hdata.maxNumDihedral)) ){
    easyMalloc (hdata.memSize(), expectedNumBond, expectedNumAngle, expectedNumDihedral);
  }
  
  numData() = hdata.numData();
  setGlobalBox (hdata.getGlobalBox());
		
  size_t sizef = hdata.numData() * sizeof(ScalorType);
  size_t sizecoord = hdata.numData() * sizeof(CoordType);
  size_t sizecoordNoi = hdata.numData() * sizeof(CoordNoiType);
  size_t sizeIdx = hdata.numData() * sizeof(IndexType);
  size_t sizet = hdata.numData() * sizeof(TypeType);

  if (mask & MDDataItemMask_Coordinate){
    hipMemcpy (coord, hdata.coord, sizecoord, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice coord");
  }

  if (mask & MDDataItemMask_CoordinateNoi){
    hipMemcpy (coordNoi, hdata.coordNoi, sizecoordNoi, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice coordNoi");
  }

  if (mask & MDDataItemMask_Velocity){
    hipMemcpy (velox, hdata.velox, sizef, hipMemcpyHostToDevice);
    hipMemcpy (veloy, hdata.veloy, sizef, hipMemcpyHostToDevice);
    hipMemcpy (veloz, hdata.veloz, sizef, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice velo");
  }

  if (mask & MDDataItemMask_Force){
    hipMemcpy (forcx, hdata.forcx, sizef, hipMemcpyHostToDevice);
    hipMemcpy (forcy, hdata.forcy, sizef, hipMemcpyHostToDevice);
    hipMemcpy (forcz, hdata.forcz, sizef, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice forc");
  }

  if (mask & MDDataItemMask_GlobalIndex){
    hipMemcpy (globalIndex, hdata.globalIndex, sizeIdx, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice globalIndex");
  }
  if (mask & MDDataItemMask_Type){
    hipMemcpy (type, hdata.type, sizet, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice type");
  }
  if (mask & MDDataItemMask_Mass){
    hipMemcpy (mass, hdata.mass, sizef, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice mass");
  }
  if (mask & MDDataItemMask_Charge){
    hipMemcpy (charge, hdata.charge, sizef, hipMemcpyHostToDevice);   
    checkCUDAError ("cpyHostMDDataToDevice charge");
  }

  size_t size0 = sizeof(IndexType) * hdata.memSize();

  if (copyBond && maxNumBond != 0){
    hipMemcpy (numBond, hdata.numBond, size0, hipMemcpyHostToDevice);
    size_t size1 = size0 * maxNumBond;
    hipMemcpy (bondIndex, hdata.bondIndex, size1, hipMemcpyHostToDevice);
    hipMemcpy (bondNeighbor_globalIndex,
		hdata.bondNeighbor_globalIndex,
		size1,
		hipMemcpyHostToDevice);
  }
  if (copyAngle && maxNumAngle != 0){
    hipMemcpy (numAngle, hdata.numAngle, size0, hipMemcpyHostToDevice);
    size_t size1 = size0 * maxNumAngle;
    size_t size2 = size0 * maxNumAngle * 2;
    hipMemcpy (angleIndex, hdata.angleIndex, size1, hipMemcpyHostToDevice);
    hipMemcpy (anglePosi, hdata.anglePosi, size1, hipMemcpyHostToDevice);
    hipMemcpy (angleNeighbor_globalIndex,
		hdata.angleNeighbor_globalIndex,
		size2,
		hipMemcpyHostToDevice);
  }
  if (copyDihedral && maxNumDihedral != 0){
    hipMemcpy (numDihedral, hdata.numDihedral, size0, hipMemcpyHostToDevice);
    size_t size1 = size0 * maxNumDihedral;
    size_t size2 = size0 * maxNumDihedral * 3;
    hipMemcpy (dihedralIndex, hdata.dihedralIndex, size1, hipMemcpyHostToDevice);
    hipMemcpy (dihedralPosi, hdata.dihedralPosi, size1, hipMemcpyHostToDevice);
    hipMemcpy (dihedralNeighbor_globalIndex,
		hdata.dihedralNeighbor_globalIndex,
		size2,
		hipMemcpyHostToDevice);
  }
}


void Parallel::DeviceMDData::
copyToHost (HostMDData & hdata,
	    const MDDataItemMask_t mask) const
{
  // if (!mask) return;
  IndexType expectedNumBond(0), expectedNumAngle(0), expectedNumDihedral(0);
  bool copyBond = (mask & MDDataItemMask_Bond);
  bool copyAngle = (mask & MDDataItemMask_Angle);
  bool copyDihedral = (mask & MDDataItemMask_Dihedral);  
  if (copyBond){
    expectedNumBond = maxNumBond;
  }
  if (copyAngle){
    expectedNumAngle = maxNumAngle;
  }
  if (copyDihedral){
    expectedNumDihedral = maxNumDihedral;
  }

  if (memSize() != hdata.memSize() ||
      (copyBond && (maxNumBond != hdata.maxNumBond)) ||
      (copyAngle && (maxNumAngle != hdata.maxNumAngle)) ||
      (copyDihedral && (maxNumDihedral != hdata.maxNumDihedral)) ){
    hdata.easyMalloc (memSize(), expectedNumBond, expectedNumAngle, expectedNumDihedral);
  }

  hdata.setGlobalBox (getGlobalBox());
  hdata.numData() = numData();
  
  size_t sizef = numData() * sizeof(ScalorType);
  size_t sizecoord = numData() * sizeof(CoordType);
  size_t sizecoordNoi = numData() * sizeof(CoordNoiType);
  size_t sizeIdx = numData() * sizeof(IndexType);
  size_t sizet = numData() * sizeof(TypeType);

  if (mask & MDDataItemMask_Coordinate){
    hipMemcpy (hdata.coord, coord, sizecoord, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost coord");
  }

  if (mask & MDDataItemMask_CoordinateNoi){
    hipMemcpy (hdata.coordNoi, coordNoi, sizecoordNoi, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost coordNoi");
  }

  if (mask & MDDataItemMask_Velocity){
    hipMemcpy (hdata.velox, velox, sizef, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.veloy, veloy, sizef, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.veloz, veloz, sizef, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost velo");
  }

  if (mask & MDDataItemMask_Force){
    hipMemcpy (hdata.forcx, forcx, sizef, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.forcy, forcy, sizef, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.forcz, forcz, sizef, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost forc");
  }
  
  if (mask & MDDataItemMask_GlobalIndex){
    hipMemcpy (hdata.globalIndex, globalIndex, sizeIdx, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost globalIndex");
  }
  if (mask & MDDataItemMask_Type){
    hipMemcpy (hdata.type, type, sizet, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost type");
  }
  if (mask & MDDataItemMask_Mass){
    hipMemcpy (hdata.mass, mass, sizef, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost mass");
  }
  if (mask & MDDataItemMask_Charge){
    hipMemcpy (hdata.charge, charge, sizef, hipMemcpyDeviceToHost);  
    checkCUDAError ("cpyDeviceMDDataToHost charge");
  }

  size_t size0 = sizeof(IndexType) * memSize();
  if (expectedNumBond != 0){
    hipMemcpy (hdata.numBond, numBond, size0, hipMemcpyDeviceToHost);
    size_t size1 = size0 * maxNumBond;
    hipMemcpy (hdata.bondIndex, bondIndex, size1, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.bondNeighbor_globalIndex,
		bondNeighbor_globalIndex,
		size1,
		hipMemcpyDeviceToHost);
  }
  if (expectedNumAngle != 0){
    hipMemcpy (hdata.numAngle, numAngle, size0, hipMemcpyDeviceToHost);
    size_t size1 = size0 * maxNumAngle;
    size_t size2 = size0 * maxNumAngle * 2;
    hipMemcpy (hdata.angleIndex, angleIndex, size1, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.anglePosi, anglePosi, size1, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.angleNeighbor_globalIndex,
		angleNeighbor_globalIndex,
		size2,
		hipMemcpyDeviceToHost);
  }
  if (expectedNumDihedral != 0){
    hipMemcpy (hdata.numDihedral, numDihedral, size0, hipMemcpyDeviceToHost);
    size_t size1 = size0 * maxNumDihedral;
    size_t size2 = size0 * maxNumDihedral * 3;
    hipMemcpy (hdata.dihedralIndex, dihedralIndex, size1, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.dihedralPosi, dihedralPosi, size1, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.dihedralNeighbor_globalIndex,
		dihedralNeighbor_globalIndex,
		size2,
		hipMemcpyDeviceToHost);
  }
}


void Parallel::DeviceMDData::
copyFromDevice (const DeviceMDData & ddata,
		const MDDataItemMask_t mask)
{
  // if (!mask) return;
  IndexType expectedNumBond(0), expectedNumAngle(0), expectedNumDihedral(0);
  bool copyBond = (mask & MDDataItemMask_Bond);
  bool copyAngle = (mask & MDDataItemMask_Angle);
  bool copyDihedral = (mask & MDDataItemMask_Dihedral);  
  if (copyBond){
    expectedNumBond = ddata.getMaxNumBond();
  }
  if (copyAngle){
    expectedNumAngle = ddata.getMaxNumAngle();
  }
  if (copyDihedral){
    expectedNumDihedral = ddata.getMaxNumDihedral();
  }

  if (memSize() != ddata.memSize() ||
      (copyBond && (maxNumBond != ddata.maxNumBond)) ||
      (copyAngle && (maxNumAngle != ddata.maxNumAngle)) ||
      (copyDihedral && (maxNumDihedral != ddata.maxNumDihedral)) ){
    easyMalloc (ddata.memSize(), expectedNumBond, expectedNumAngle, expectedNumDihedral);
  }

  numData() = ddata.numData();
  setGlobalBox (ddata.getGlobalBox());
		
  size_t sizef = ddata.numData() * sizeof(ScalorType);
  size_t sizecoord = ddata.numData() * sizeof(CoordType);
  size_t sizecoordNoi = ddata.numData() * sizeof(CoordNoiType);
  size_t sizeIdx = ddata.numData() * sizeof(IndexType);
  size_t sizet = ddata.numData() * sizeof(TypeType);

  if (mask & MDDataItemMask_Coordinate){
    hipMemcpy (coord, ddata.coord, sizecoord, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice coord");
  }

  if (mask & MDDataItemMask_CoordinateNoi){
    hipMemcpy (coordNoi, ddata.coordNoi, sizecoordNoi, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice coordNoi");
  }

  if (mask & MDDataItemMask_Velocity){
    hipMemcpy (velox, ddata.velox, sizef, hipMemcpyDeviceToDevice);
    hipMemcpy (veloy, ddata.veloy, sizef, hipMemcpyDeviceToDevice);
    hipMemcpy (veloz, ddata.veloz, sizef, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice velo");
  }

  if (mask & MDDataItemMask_Force){
    hipMemcpy (forcx, ddata.forcx, sizef, hipMemcpyDeviceToDevice);
    hipMemcpy (forcy, ddata.forcy, sizef, hipMemcpyDeviceToDevice);
    hipMemcpy (forcz, ddata.forcz, sizef, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice forc");
  }

  if (mask & MDDataItemMask_GlobalIndex){
    hipMemcpy (globalIndex, ddata.globalIndex, sizeIdx, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice globalIndex");
  }
  if (mask & MDDataItemMask_Type){
    hipMemcpy (type, ddata.type, sizet, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice globalIndex");
  }
  if (mask & MDDataItemMask_Mass){
    hipMemcpy (mass, ddata.mass, sizef, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice globalIndex");
  }
  if (mask & MDDataItemMask_Charge){
    hipMemcpy (charge, ddata.charge, sizef, hipMemcpyDeviceToDevice);     
    checkCUDAError ("cpyDeviceMDDataToDevice globalIndex");
  }

  size_t size0 = sizeof(IndexType) * ddata.memSize();

  if (copyBond && maxNumBond != 0){
    hipMemcpy (numBond, ddata.numBond, size0, hipMemcpyDeviceToDevice);
    size_t size1 = size0 * maxNumBond;
    hipMemcpy (bondIndex, ddata.bondIndex, size1, hipMemcpyDeviceToDevice);
    hipMemcpy (bondNeighbor_globalIndex,
		ddata.bondNeighbor_globalIndex,
		size1,
		hipMemcpyDeviceToDevice);
  }
  if (copyAngle && maxNumAngle != 0){
    hipMemcpy (numAngle, ddata.numAngle, size0, hipMemcpyDeviceToDevice);
    size_t size1 = size0 * maxNumAngle;
    size_t size2 = size0 * maxNumAngle * 2;
    hipMemcpy (angleIndex, ddata.angleIndex, size1, hipMemcpyDeviceToDevice);
    hipMemcpy (anglePosi, ddata.anglePosi, size1, hipMemcpyDeviceToDevice);
    hipMemcpy (angleNeighbor_globalIndex,
		ddata.angleNeighbor_globalIndex,
		size2,
		hipMemcpyDeviceToDevice);
  }
  if (copyDihedral && maxNumDihedral != 0){
    hipMemcpy (numDihedral, ddata.numDihedral, size0, hipMemcpyDeviceToDevice);
    size_t size1 = size0 * maxNumDihedral;
    size_t size2 = size0 * maxNumDihedral * 3;
    hipMemcpy (dihedralIndex, ddata.dihedralIndex, size1, hipMemcpyDeviceToDevice);
    hipMemcpy (dihedralPosi, ddata.dihedralPosi, size1, hipMemcpyDeviceToDevice);
    hipMemcpy (dihedralNeighbor_globalIndex,
		ddata.dihedralNeighbor_globalIndex,
		size2,
		hipMemcpyDeviceToDevice);
  }
}


Parallel::DeviceMDData::
DeviceMDData (const DeviceMDData & ddata)
    : _numData(0),
      _memSize(0), 
      maxNumBond(0),
      maxNumAngle(0),
      maxNumDihedral(0)
{
  copyFromDevice (ddata, MDDataItemMask_All);
}


// void Parallel::DeviceMDData::
// initZero ()
// {
//   Parallel::CudaGlobal::initZeroDeviceData
//       <<<memSize_ / DefaultNThreadPerBlock + 1, DefaultNThreadPerBlock>>>
//       (memSize_,
//        coord,
//        coordNoi,
//        velox,
//        veloy,
//        veloz,
//        forcx,
//        forcy,
//        forcz,
//        globalIndex,
//        type,
//        mass,
//        charge);
//   checkCUDAError ("DeviceMDData::initZero initZeroDeviceData");
// }


// __global__ void Parallel::CudaGlobal::
// initZeroDeviceData(const IndexType num,
// 		   CoordType  * coord,
// 		   CoordNoiType * coordNoi,
// 		   ScalorType * velox,
// 		   ScalorType * veloy,
// 		   ScalorType * veloz,
// 		   ScalorType * forcx,
// 		   ScalorType * forcy,
// 		   ScalorType * forcz,
// 		   IndexType  * globalIndex,
// 		   TypeType   * type,
// 		   ScalorType * mass,
// 		   ScalorType * charge)
// {
//   IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
//   IndexType tid = threadIdx.x;
//   IndexType ii = tid + bid * blockDim.x;

//   if (ii < num){
//     CoordType tmp;
//     tmp.x = 0;
//     tmp.y = 0;
//     tmp.z = 0;
//     tmp.w = -1;
//     coord[ii] = tmp;
//     coordNoi[ii].x = coordNoi[ii].y = coordNoi[ii].z = 0;
//     veloz[ii] = veloy[ii] = veloz[ii] = 0.f;
//     forcz[ii] = forcy[ii] = forcz[ii] = 0.f;
//     globalIndex[ii] = MaxIndexValue;
//     type[ii] = 0;
//     mass[ii] = 0;
//     charge[ii] = 0;
//   }
// }


void Parallel::GlobalHostMDData::
initWriteData_xtcFile (const char * filename, float prec)
{
  xdfile = NULL;
  xdfile = xdrfile_open (filename, "w");
  if (xdfile == NULL){
    MDExcptCannotOpenFile ("MDSystem::initWriteXtc", filename);
  }
  for (unsigned i = 0; i < 3; ++i){
    for (unsigned j = 0; j < 3; ++j){
      xdbox[i][j] = 0.f;
    }	      
  }
  xdx = (rvec *) malloc (sizeof(rvec) * numData());
  if (xdx == NULL){
    MDExcptFailedMallocOnHost ("MDSystem::initWriteXtc", "xdx", sizeof(rvec) * numData());
  }
  xdprec = prec;
}


void Parallel::GlobalHostMDData::
writeData_xtcFile (int step, float time)
{
  for (IndexType i = 0; i < numData(); ++i){
    xdx[i][0] = coord[i].x;
    xdx[i][1] = coord[i].y;
    xdx[i][2] = coord[i].z;
  }
  xdbox[0][0] = globalBox.size.x;
  xdbox[1][1] = globalBox.size.y;
  xdbox[2][2] = globalBox.size.z;
  write_xtc (xdfile, numData(), step, time, xdbox, xdx, xdprec);
}

void Parallel::GlobalHostMDData::
endWriteData_xtcFile ()
{
  free (xdx);
  xdrfile_close(xdfile);
}

void Parallel::DeviceMDData::
mallocFromDevice (const DeviceMDData & ddata)
{
  setGlobalBox (ddata.getGlobalBox());
  easyMalloc (ddata.memSize(), ddata.getMaxNumBond(), ddata.getMaxNumAngle(),
	      ddata.getMaxNumDihedral());
  _numData = 0;
}

void Parallel::DeviceMDData::
mallocFromHost (const HostMDData & hdata)
{
  setGlobalBox (hdata.getGlobalBox());
  easyMalloc (hdata.memSize(), hdata.getMaxNumBond(), hdata.getMaxNumAngle(),
	      hdata.getMaxNumDihedral());
  _numData = 0;
}


void Parallel::DeviceMDData::
mallocToHost (HostMDData & hdata) const 
{
  hdata.setGlobalBox (getGlobalBox());
  hdata.easyMalloc (memSize(), getMaxNumBond(), getMaxNumAngle(), getMaxNumDihedral());
  hdata.numData() = 0;
}









