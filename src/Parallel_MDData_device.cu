#define DEVICE_CODE

#include "Parallel_MDData.h"

Parallel::DeviceMDData::
DeviceMDData ()
    : numAtom_ (0), memSize_(0), malloced(false)
{
}

Parallel::DeviceMDData::
~DeviceMDData ()
{
  clearAll();
}

void Parallel::DeviceMDData::
mallocAll (const IndexType & memSize__)
{
  if (malloced){
    clearAll ();
  }
  if (memSize__ == 0) return;

  memSize_ = memSize__;
  
  size_t sizef = memSize_ * sizeof(ScalorType);
  size_t sizei = memSize_ * sizeof(IntScalorType);
  size_t sizecoord =memSize_ * sizeof(CoordType);

  hipMalloc ((void**) &coord, sizecoord);
  checkCUDAError ("initDeviceMDData coord");

  hipMalloc ((void**) &velox, sizef);
  hipMalloc ((void**) &veloy, sizef);
  hipMalloc ((void**) &veloz, sizef);
  checkCUDAError ("initDeviceMDData velo");

  hipMalloc ((void**) &forcx, sizef);
  hipMalloc ((void**) &forcy, sizef);
  hipMalloc ((void**) &forcz, sizef);
  checkCUDAError ("initDeviceMDData forc");

  hipMalloc ((void**) &coordNoix, sizei);
  hipMalloc ((void**) &coordNoiy, sizei);
  hipMalloc ((void**) &coordNoiz, sizei);
  checkCUDAError ("initDeviceMDData coordNoi");

  hipMalloc ((void**) &type, memSize_ * sizeof(TypeType));
  hipMalloc ((void**) &mass, sizef);
  hipMalloc ((void**) &charge, sizef);
  checkCUDAError ("initDeviceMDData top Property");

  malloced = true;
}

void Parallel::DeviceMDData::
clearAll ()
{
  if (malloced){
    hipFree (coord);
    
    hipFree (coordNoix);
    hipFree (coordNoiy);
    hipFree (coordNoiz);
  
    hipFree (velox);
    hipFree (veloy);
    hipFree (veloz);
  
    hipFree (forcx);
    hipFree (forcy);
    hipFree (forcz);

    hipFree (type);
    hipFree (mass);
    hipFree (charge);
    malloced = false;
  }
}


void Parallel::DeviceMDData::
copyFromHost (const HostMDData & hdata)
{
  if (memSize_ < hdata.numAtom()){
    clearAll();
    mallocAll(hdata.numAtom());
  }

  numAtom_ = hdata.numAtom();

  size_t sizef = numAtom_ * sizeof(ScalorType);
  size_t sizei = numAtom_ * sizeof(IntScalorType);
  size_t sizecoord = numAtom_ * sizeof(CoordType);
  
  hipMemcpy (coord, hdata.coord, sizecoord, hipMemcpyHostToDevice);
  checkCUDAError ("cpyHostMDDataToDevice coord");

  hipMemcpy (coordNoix, hdata.coordNoix, sizei, hipMemcpyHostToDevice);
  hipMemcpy (coordNoiy, hdata.coordNoiy, sizei, hipMemcpyHostToDevice);
  hipMemcpy (coordNoiz, hdata.coordNoiz, sizei, hipMemcpyHostToDevice);
  checkCUDAError ("cpyHostMDDataToDevice coordNoi");
  
  hipMemcpy (velox, hdata.velox, sizef, hipMemcpyHostToDevice);
  hipMemcpy (veloy, hdata.veloy, sizef, hipMemcpyHostToDevice);
  hipMemcpy (veloz, hdata.veloz, sizef, hipMemcpyHostToDevice);
  checkCUDAError ("cpyHostMDDataToDevice velo");

  hipMemcpy (forcx, hdata.forcx, sizef, hipMemcpyHostToDevice);
  hipMemcpy (forcy, hdata.forcy, sizef, hipMemcpyHostToDevice);
  hipMemcpy (forcz, hdata.forcz, sizef, hipMemcpyHostToDevice);
  checkCUDAError ("cpyHostMDDataToDevice forc");

  hipMemcpy (type, hdata.type, numAtom_ * sizeof(TypeType), hipMemcpyHostToDevice);
  hipMemcpy (mass, hdata.mass, sizef, hipMemcpyHostToDevice);
  hipMemcpy (charge, hdata.charge, sizef, hipMemcpyHostToDevice);   
  checkCUDAError ("cpyHostMDDataToDevice other");
}


void Parallel::DeviceMDData::
copyToHost (HostMDData & hdata) const
{
  if (hdata.memSize() < numAtom_){
    hdata.reallocAll (numAtom_);
  }
  hdata.numAtom() = numAtom_;

  size_t sizef = numAtom_ * sizeof(ScalorType);
  size_t sizei = numAtom_ * sizeof(IntScalorType);
  size_t sizecoord = numAtom_ * sizeof(CoordType);
  
  hipMemcpy (hdata.coord, coord, sizecoord, hipMemcpyDeviceToHost);
  checkCUDAError ("cpyDeviceMDDataToHost coord");
  
  hipMemcpy (hdata.coordNoix, coordNoix, sizei, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.coordNoiy, coordNoiy, sizei, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.coordNoiz, coordNoiz, sizei, hipMemcpyDeviceToHost);
  checkCUDAError ("cpyDeviceMDDataToHost coordNoi");
  
  hipMemcpy (hdata.velox, velox, sizef, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.veloy, veloy, sizef, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.veloz, veloz, sizef, hipMemcpyDeviceToHost);
  checkCUDAError ("cpyDeviceMDDataToHost velo");

  hipMemcpy (hdata.forcx, forcx, sizef, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.forcy, forcy, sizef, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.forcz, forcz, sizef, hipMemcpyDeviceToHost);
  checkCUDAError ("cpyDeviceMDDataToHost forc");

  hipMemcpy (hdata.type, type, numAtom_ * sizeof(TypeType), hipMemcpyDeviceToHost);
  hipMemcpy (hdata.mass, mass, sizef, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.charge, charge, sizef, hipMemcpyDeviceToHost);  
  checkCUDAError ("cpyDeviceMDDataToHost other");
}

