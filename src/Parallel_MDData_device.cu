#include "hip/hip_runtime.h"
#define DEVICE_CODE
#include "Parallel_MDData.h"
#include "Parallel_MDData_device.h"

#include "compile_error_mixcode.h"

Parallel::DeviceMDData::
DeviceMDData ()
    : numAtom_ (0), memSize_(0), malloced(false)
{
}

Parallel::DeviceMDData::
~DeviceMDData ()
{
  clearAll();
}

void Parallel::DeviceMDData::
mallocAll (const IndexType & memSize__)
{
  if (malloced){
    clearAll ();
  }
  if (memSize__ == 0) return;

  memSize_ = memSize__;
  
  size_t sizef = memSize_ * sizeof(ScalorType);
  size_t sizei = memSize_ * sizeof(IntScalorType);
  size_t sizecoord =memSize_ * sizeof(CoordType);
  size_t sizeIdx = memSize_ * sizeof(IndexType);
  
  hipMalloc ((void**) &coord, sizecoord);
  checkCUDAError ("initDeviceMDData coord");

  hipMalloc ((void**) &velox, sizef);
  hipMalloc ((void**) &veloy, sizef);
  hipMalloc ((void**) &veloz, sizef);
  checkCUDAError ("initDeviceMDData velo");

  hipMalloc ((void**) &forcx, sizef);
  hipMalloc ((void**) &forcy, sizef);
  hipMalloc ((void**) &forcz, sizef);
  checkCUDAError ("initDeviceMDData forc");

  hipMalloc ((void**) &coordNoix, sizei);
  hipMalloc ((void**) &coordNoiy, sizei);
  hipMalloc ((void**) &coordNoiz, sizei);
  checkCUDAError ("initDeviceMDData coordNoi");

  hipMalloc ((void**) &globalIndex, sizeIdx);
  hipMalloc ((void**) &type, memSize_ * sizeof(TypeType));
  hipMalloc ((void**) &mass, sizef);
  hipMalloc ((void**) &charge, sizef);
  checkCUDAError ("initDeviceMDData top Property");

  malloced = true;
}

void Parallel::DeviceMDData::
clearAll ()
{
  if (malloced){
    hipFree (coord);
    
    hipFree (coordNoix);
    hipFree (coordNoiy);
    hipFree (coordNoiz);
  
    hipFree (velox);
    hipFree (veloy);
    hipFree (veloz);
  
    hipFree (forcx);
    hipFree (forcy);
    hipFree (forcz);

    hipFree (globalIndex);
    hipFree (type);
    hipFree (mass);
    hipFree (charge);
    malloced = false;
  }
}


void Parallel::DeviceMDData::
copyFromHost (const HostMDData & hdata)
{
  if (memSize_ < hdata.numAtom()){
    clearAll();
    mallocAll(hdata.numAtom());
  }
  numAtom_ = hdata.numAtom();
  setGlobalBox (hdata.getGlobalBox());
		
  size_t sizef = numAtom_ * sizeof(ScalorType);
  size_t sizei = numAtom_ * sizeof(IntScalorType);
  size_t sizecoord = numAtom_ * sizeof(CoordType);
  size_t sizeIdx = memSize_ * sizeof(IndexType);
  
  hipMemcpy (coord, hdata.coord, sizecoord, hipMemcpyHostToDevice);
  checkCUDAError ("cpyHostMDDataToDevice coord");

  hipMemcpy (coordNoix, hdata.coordNoix, sizei, hipMemcpyHostToDevice);
  hipMemcpy (coordNoiy, hdata.coordNoiy, sizei, hipMemcpyHostToDevice);
  hipMemcpy (coordNoiz, hdata.coordNoiz, sizei, hipMemcpyHostToDevice);
  checkCUDAError ("cpyHostMDDataToDevice coordNoi");
  
  hipMemcpy (velox, hdata.velox, sizef, hipMemcpyHostToDevice);
  hipMemcpy (veloy, hdata.veloy, sizef, hipMemcpyHostToDevice);
  hipMemcpy (veloz, hdata.veloz, sizef, hipMemcpyHostToDevice);
  checkCUDAError ("cpyHostMDDataToDevice velo");

  hipMemcpy (forcx, hdata.forcx, sizef, hipMemcpyHostToDevice);
  hipMemcpy (forcy, hdata.forcy, sizef, hipMemcpyHostToDevice);
  hipMemcpy (forcz, hdata.forcz, sizef, hipMemcpyHostToDevice);
  checkCUDAError ("cpyHostMDDataToDevice forc");

  hipMemcpy (globalIndex, hdata.globalIndex, sizeIdx, hipMemcpyHostToDevice);
  hipMemcpy (type, hdata.type, numAtom_ * sizeof(TypeType), hipMemcpyHostToDevice);
  hipMemcpy (mass, hdata.mass, sizef, hipMemcpyHostToDevice);
  hipMemcpy (charge, hdata.charge, sizef, hipMemcpyHostToDevice);   
  checkCUDAError ("cpyHostMDDataToDevice other");
}


void Parallel::DeviceMDData::
copyToHost (HostMDData & hdata) const
{
  if (hdata.memSize() < numAtom_){
    hdata.reallocAll (numAtom_);
  }
  hdata.numAtom() = numAtom_;
  hdata.setGlobalBox (globalBox);
  
  size_t sizef = numAtom_ * sizeof(ScalorType);
  size_t sizei = numAtom_ * sizeof(IntScalorType);
  size_t sizecoord = numAtom_ * sizeof(CoordType);
  size_t sizeIdx = memSize_ * sizeof(IndexType);
  
  hipMemcpy (hdata.coord, coord, sizecoord, hipMemcpyDeviceToHost);
  checkCUDAError ("cpyDeviceMDDataToHost coord");
  
  hipMemcpy (hdata.coordNoix, coordNoix, sizei, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.coordNoiy, coordNoiy, sizei, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.coordNoiz, coordNoiz, sizei, hipMemcpyDeviceToHost);
  checkCUDAError ("cpyDeviceMDDataToHost coordNoi");
  
  hipMemcpy (hdata.velox, velox, sizef, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.veloy, veloy, sizef, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.veloz, veloz, sizef, hipMemcpyDeviceToHost);
  checkCUDAError ("cpyDeviceMDDataToHost velo");

  hipMemcpy (hdata.forcx, forcx, sizef, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.forcy, forcy, sizef, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.forcz, forcz, sizef, hipMemcpyDeviceToHost);
  checkCUDAError ("cpyDeviceMDDataToHost forc");

  hipMemcpy (hdata.globalIndex, globalIndex, sizeIdx, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.type, type, numAtom_ * sizeof(TypeType), hipMemcpyDeviceToHost);
  hipMemcpy (hdata.mass, mass, sizef, hipMemcpyDeviceToHost);
  hipMemcpy (hdata.charge, charge, sizef, hipMemcpyDeviceToHost);  
  checkCUDAError ("cpyDeviceMDDataToHost other");
}


void Parallel::DeviceMDData::
copyFromDevice (const DeviceMDData & ddata)
{
  if (memSize_ < ddata.memSize_){
    clearAll();
    mallocAll (ddata.memSize());
  }
  numAtom_ = ddata.numAtom();
  setGlobalBox (ddata.getGlobalBox());
		
  size_t sizef = numAtom_ * sizeof(ScalorType);
  size_t sizei = numAtom_ * sizeof(IntScalorType);
  size_t sizecoord = numAtom_ * sizeof(CoordType);
  size_t sizeIdx = memSize_ * sizeof(IndexType);
  
  hipMemcpy (coord, ddata.coord, sizecoord, hipMemcpyDeviceToDevice);
  checkCUDAError ("cpyDeviceMDDataToDevice coord");

  hipMemcpy (coordNoix, ddata.coordNoix, sizei, hipMemcpyDeviceToDevice);
  hipMemcpy (coordNoiy, ddata.coordNoiy, sizei, hipMemcpyDeviceToDevice);
  hipMemcpy (coordNoiz, ddata.coordNoiz, sizei, hipMemcpyDeviceToDevice);
  checkCUDAError ("cpyDeviceMDDataToDevice coordNoi");
  
  hipMemcpy (velox, ddata.velox, sizef, hipMemcpyDeviceToDevice);
  hipMemcpy (veloy, ddata.veloy, sizef, hipMemcpyDeviceToDevice);
  hipMemcpy (veloz, ddata.veloz, sizef, hipMemcpyDeviceToDevice);
  checkCUDAError ("cpyDeviceMDDataToDevice velo");

  hipMemcpy (forcx, ddata.forcx, sizef, hipMemcpyDeviceToDevice);
  hipMemcpy (forcy, ddata.forcy, sizef, hipMemcpyDeviceToDevice);
  hipMemcpy (forcz, ddata.forcz, sizef, hipMemcpyDeviceToDevice);
  checkCUDAError ("cpyDeviceMDDataToDevice forc");

  hipMemcpy (globalIndex, ddata.globalIndex, sizeIdx, hipMemcpyDeviceToDevice);
  hipMemcpy (type, ddata.type, numAtom_ * sizeof(TypeType), hipMemcpyDeviceToDevice);
  hipMemcpy (mass, ddata.mass, sizef, hipMemcpyDeviceToDevice);
  hipMemcpy (charge, ddata.charge, sizef, hipMemcpyDeviceToDevice);   
  checkCUDAError ("cpyDeviceMDDataToDevice other");
}


Parallel::DeviceMDData::
DeviceMDData (const DeviceMDData & ddata)
    : numAtom_ (0), memSize_(0), malloced(false)
{
  copyFromDevice (ddata);
}


void Parallel::DeviceMDData::
initZero ()
{
  Parallel::CudaGlobal::initZeroDeviceData
      <<<memSize_ / DefaultNThreadPerBlock + 1, DefaultNThreadPerBlock>>>
      (memSize_,
       coord,
       coordNoix,
       coordNoiy,
       coordNoiz,
       velox,
       veloy,
       veloz,
       globalIndex,
       type,
       mass,
       charge);
  checkCUDAError ("DeviceMDData::initZero initZeroDeviceData");
}


__global__ void Parallel::CudaGlobal::
initZeroDeviceData(const IndexType num,
		   CoordType  * coord,
		   IntScalorType * coordNoix,
		   IntScalorType * coordNoiy,
		   IntScalorType * coordNoiz,
		   ScalorType * velox,
		   ScalorType * veloy,
		   ScalorType * veloz,
		   IndexType  * globalIndex,
		   TypeType   * type,
		   ScalorType * mass,
		   ScalorType * charge)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;

  if (ii < num){
    CoordType tmp;
    tmp.x = 0;
    tmp.y = 0;
    tmp.z = 0;
    tmp.w = MaxIndexValue;
    coord[ii] = tmp;
    coordNoix[ii] = coordNoiy[ii] = coordNoiz[ii] = 0;
    veloz[ii] = veloy[ii] = veloz[ii] = 0.f;
    globalIndex[ii] = MaxIndexValue;
    type[ii] = 0;
    mass[ii] = 0;
    charge[ii] = 0;
  }
}



  
