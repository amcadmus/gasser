#include "hip/hip_runtime.h"
#define DEVICE_CODE
#include "Parallel_MDData.h"
#include "Parallel_MDData_device.h"

#include "compile_error_mixcode.h"

Parallel::DeviceMDData::
DeviceMDData ()
    : numData_ (0), memSize_(0), malloced(false)
{
}

Parallel::DeviceMDData::
~DeviceMDData ()
{
  clearAll();
}

void Parallel::DeviceMDData::
mallocAll (const IndexType & memSize__)
{
  if (malloced){
    clearAll ();
  }
  if (memSize__ == 0) return;

  memSize_ = memSize__;
  
  size_t sizef = memSize_ * sizeof(ScalorType);
  size_t sizei = memSize_ * sizeof(IntScalorType);
  size_t sizecoord =memSize_ * sizeof(CoordType);
  size_t sizeIdx = memSize_ * sizeof(IndexType);
  size_t sizet = memSize_ * sizeof(TypeType);
  
  hipMalloc ((void**) &coord, sizecoord);
  checkCUDAError ("initDeviceMDData coord");

  hipMalloc ((void**) &velox, sizef);
  hipMalloc ((void**) &veloy, sizef);
  hipMalloc ((void**) &veloz, sizef);
  checkCUDAError ("initDeviceMDData velo");

  hipMalloc ((void**) &forcx, sizef);
  hipMalloc ((void**) &forcy, sizef);
  hipMalloc ((void**) &forcz, sizef);
  checkCUDAError ("initDeviceMDData forc");

  hipMalloc ((void**) &coordNoix, sizei);
  hipMalloc ((void**) &coordNoiy, sizei);
  hipMalloc ((void**) &coordNoiz, sizei);
  checkCUDAError ("initDeviceMDData coordNoi");

  hipMalloc ((void**) &globalIndex, sizeIdx);
  hipMalloc ((void**) &type, sizet);
  hipMalloc ((void**) &mass, sizef);
  hipMalloc ((void**) &charge, sizef);
  checkCUDAError ("initDeviceMDData top Property");

  malloced = true;
}

void Parallel::DeviceMDData::
clearAll ()
{
  if (malloced){
    hipFree (coord);
    
    hipFree (coordNoix);
    hipFree (coordNoiy);
    hipFree (coordNoiz);
  
    hipFree (velox);
    hipFree (veloy);
    hipFree (veloz);
  
    hipFree (forcx);
    hipFree (forcy);
    hipFree (forcz);

    hipFree (globalIndex);
    hipFree (type);
    hipFree (mass);
    hipFree (charge);
    malloced = false;
  }
}


void Parallel::DeviceMDData::
copyFromHost (const HostMDData & hdata,
	      const MDDataItemMask_t mask)
{
  if (memSize_ < hdata.numData()){
    clearAll();
    mallocAll (hdata.numData());
  }
  numData_ = hdata.numData();
  setGlobalBox (hdata.getGlobalBox());
		
  size_t sizef = numData_ * sizeof(ScalorType);
  size_t sizei = numData_ * sizeof(IntScalorType);
  size_t sizecoord = numData_ * sizeof(CoordType);
  size_t sizeIdx = numData_ * sizeof(IndexType);
  size_t sizet = numData_ * sizeof(TypeType);

  if (mask & MDDataItemMask_Coordinate){
    hipMemcpy (coord, hdata.coord, sizecoord, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice coord");
  }

  if (mask & MDDataItemMask_CoordinateNoi){
    hipMemcpy (coordNoix, hdata.coordNoix, sizei, hipMemcpyHostToDevice);
    hipMemcpy (coordNoiy, hdata.coordNoiy, sizei, hipMemcpyHostToDevice);
    hipMemcpy (coordNoiz, hdata.coordNoiz, sizei, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice coordNoi");
  }

  if (mask & MDDataItemMask_Velocity){
    hipMemcpy (velox, hdata.velox, sizef, hipMemcpyHostToDevice);
    hipMemcpy (veloy, hdata.veloy, sizef, hipMemcpyHostToDevice);
    hipMemcpy (veloz, hdata.veloz, sizef, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice velo");
  }

  if (mask & MDDataItemMask_Force){
    hipMemcpy (forcx, hdata.forcx, sizef, hipMemcpyHostToDevice);
    hipMemcpy (forcy, hdata.forcy, sizef, hipMemcpyHostToDevice);
    hipMemcpy (forcz, hdata.forcz, sizef, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice forc");
  }

  if (mask & MDDataItemMask_GlobalIndex){
    hipMemcpy (globalIndex, hdata.globalIndex, sizeIdx, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice globalIndex");
  }
  if (mask & MDDataItemMask_Type){
    hipMemcpy (type, hdata.type, sizet, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice type");
  }
  if (mask & MDDataItemMask_Mass){
    hipMemcpy (mass, hdata.mass, sizef, hipMemcpyHostToDevice);
    checkCUDAError ("cpyHostMDDataToDevice mass");
  }
  if (mask & MDDataItemMask_Charge){
    hipMemcpy (charge, hdata.charge, sizef, hipMemcpyHostToDevice);   
    checkCUDAError ("cpyHostMDDataToDevice charge");
  }
}


void Parallel::DeviceMDData::
copyToHost (HostMDData & hdata,
	    const MDDataItemMask_t mask) const
{
  if (hdata.memSize() < numData_){
    hdata.reallocAll (numData_);
  }
  hdata.numData_ = numData_;
  hdata.setGlobalBox (globalBox);
  
  size_t sizef = numData_ * sizeof(ScalorType);
  size_t sizei = numData_ * sizeof(IntScalorType);
  size_t sizecoord = numData_ * sizeof(CoordType);
  size_t sizeIdx = numData_ * sizeof(IndexType);
  size_t sizet = numData_ * sizeof(TypeType);

  if (mask & MDDataItemMask_Coordinate){
    hipMemcpy (hdata.coord, coord, sizecoord, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost coord");
  }

  if (mask & MDDataItemMask_CoordinateNoi){
    hipMemcpy (hdata.coordNoix, coordNoix, sizei, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.coordNoiy, coordNoiy, sizei, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.coordNoiz, coordNoiz, sizei, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost coordNoi");
  }

  if (mask & MDDataItemMask_Velocity){
    hipMemcpy (hdata.velox, velox, sizef, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.veloy, veloy, sizef, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.veloz, veloz, sizef, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost velo");
  }

  if (mask & MDDataItemMask_Force){
    hipMemcpy (hdata.forcx, forcx, sizef, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.forcy, forcy, sizef, hipMemcpyDeviceToHost);
    hipMemcpy (hdata.forcz, forcz, sizef, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost forc");
  }
  
  if (mask & MDDataItemMask_GlobalIndex){
    hipMemcpy (hdata.globalIndex, globalIndex, sizeIdx, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost globalIndex");
  }
  if (mask & MDDataItemMask_Type){
    hipMemcpy (hdata.type, type, sizet, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost type");
  }
  if (mask & MDDataItemMask_Mass){
    hipMemcpy (hdata.mass, mass, sizef, hipMemcpyDeviceToHost);
    checkCUDAError ("cpyDeviceMDDataToHost mass");
  }
  if (mask & MDDataItemMask_Charge){
    hipMemcpy (hdata.charge, charge, sizef, hipMemcpyDeviceToHost);  
    checkCUDAError ("cpyDeviceMDDataToHost charge");
  }
}


void Parallel::DeviceMDData::
copyFromDevice (const DeviceMDData & ddata,
		const MDDataItemMask_t mask)
{
  if (numData_ < ddata.memSize()){
    clearAll();
    mallocAll (ddata.memSize());
  }
  numData_ = ddata.numData();
  setGlobalBox (ddata.getGlobalBox());
		
  size_t sizef = numData_ * sizeof(ScalorType);
  size_t sizei = numData_ * sizeof(IntScalorType);
  size_t sizecoord = numData_ * sizeof(CoordType);
  size_t sizeIdx = numData_ * sizeof(IndexType);
  size_t sizet = numData_ * sizeof(TypeType);

  if (mask & MDDataItemMask_Coordinate){
    hipMemcpy (coord, ddata.coord, sizecoord, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice coord");
  }

  if (mask & MDDataItemMask_CoordinateNoi){
    hipMemcpy (coordNoix, ddata.coordNoix, sizei, hipMemcpyDeviceToDevice);
    hipMemcpy (coordNoiy, ddata.coordNoiy, sizei, hipMemcpyDeviceToDevice);
    hipMemcpy (coordNoiz, ddata.coordNoiz, sizei, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice coordNoi");
  }

  if (mask & MDDataItemMask_Velocity){
    hipMemcpy (velox, ddata.velox, sizef, hipMemcpyDeviceToDevice);
    hipMemcpy (veloy, ddata.veloy, sizef, hipMemcpyDeviceToDevice);
    hipMemcpy (veloz, ddata.veloz, sizef, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice velo");
  }

  if (mask & MDDataItemMask_Force){
    hipMemcpy (forcx, ddata.forcx, sizef, hipMemcpyDeviceToDevice);
    hipMemcpy (forcy, ddata.forcy, sizef, hipMemcpyDeviceToDevice);
    hipMemcpy (forcz, ddata.forcz, sizef, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice forc");
  }

  if (mask & MDDataItemMask_GlobalIndex){
    hipMemcpy (globalIndex, ddata.globalIndex, sizeIdx, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice globalIndex");
  }
  if (mask & MDDataItemMask_Type){
    hipMemcpy (type, ddata.type, sizet, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice globalIndex");
  }
  if (mask & MDDataItemMask_Mass){
    hipMemcpy (mass, ddata.mass, sizef, hipMemcpyDeviceToDevice);
    checkCUDAError ("cpyDeviceMDDataToDevice globalIndex");
  }
  if (mask & MDDataItemMask_Charge){
    hipMemcpy (charge, ddata.charge, sizef, hipMemcpyDeviceToDevice);     
    checkCUDAError ("cpyDeviceMDDataToDevice globalIndex");
  }
}


Parallel::DeviceMDData::
DeviceMDData (const DeviceMDData & ddata)
    : numData_ (0), memSize_(0), malloced(false)
{
  copyFromDevice (ddata);
}


void Parallel::DeviceMDData::
initZero ()
{
  Parallel::CudaGlobal::initZeroDeviceData
      <<<memSize_ / DefaultNThreadPerBlock + 1, DefaultNThreadPerBlock>>>
      (memSize_,
       coord,
       coordNoix,
       coordNoiy,
       coordNoiz,
       velox,
       veloy,
       veloz,
       forcx,
       forcy,
       forcz,
       globalIndex,
       type,
       mass,
       charge);
  checkCUDAError ("DeviceMDData::initZero initZeroDeviceData");
}


__global__ void Parallel::CudaGlobal::
initZeroDeviceData(const IndexType num,
		   CoordType  * coord,
		   IntScalorType * coordNoix,
		   IntScalorType * coordNoiy,
		   IntScalorType * coordNoiz,
		   ScalorType * velox,
		   ScalorType * veloy,
		   ScalorType * veloz,
		   ScalorType * forcx,
		   ScalorType * forcy,
		   ScalorType * forcz,
		   IndexType  * globalIndex,
		   TypeType   * type,
		   ScalorType * mass,
		   ScalorType * charge)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;

  if (ii < num){
    CoordType tmp;
    tmp.x = 0;
    tmp.y = 0;
    tmp.z = 0;
    tmp.w = MaxIndexValue;
    coord[ii] = tmp;
    coordNoix[ii] = coordNoiy[ii] = coordNoiz[ii] = 0;
    veloz[ii] = veloy[ii] = veloz[ii] = 0.f;
    forcz[ii] = forcy[ii] = forcz[ii] = 0.f;
    globalIndex[ii] = MaxIndexValue;
    type[ii] = 0;
    mass[ii] = 0;
    charge[ii] = 0;
  }
}



  
