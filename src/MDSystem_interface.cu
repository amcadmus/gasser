#include "hip/hip_runtime.h"
#define CPLUSPLUS
#include "MDSystem_interface.h"
#include "GromacsFileManager.h"
#include "FileManager.h"
#include "MDException.h"

MDSystem::MDSystem()
{
  xdfile = NULL;
  xdx = NULL;
  tmpNAtomType = 0;
  // setNULL (&hdata);
  // setNULL (&ddata);
}


void MDSystem::initConfig (const char * configfile, const char * mapfile,
			   const IndexType & maxNumAtom)
{
  FILE * fpc = fopen (configfile, "r");
  if (fpc == NULL) {
    throw MDExcptCannotOpenFile ("MDSystem::initConfig:", configfile);
  }
  while (fgetc(fpc) != '\n');

  IndexType numAtom, numMem;
  if (fscanf (fpc, "%d", &(numAtom)) != 1){
    throw MDExcptWrongFileFormat ("MDSystem::initConfig", configfile);
  }
  if (maxNumAtom != 0) {
    numMem = maxNumAtom;
  }
  else {
    numMem = numAtom;
  }
  mallocHostMDData (numAtom, numMem, &hdata);

  IndexType * tmpatomIndex = (IndexType * )malloc(sizeof(IndexType) * numMem);
  if (tmpatomIndex == NULL){
    throw MDExcptFailedMallocOnHost ("MDSystem::initConfig", "tmpatomIndex",
				     sizeof(IndexType) * numMem);
  }
  ScalorType bx, by, bz;
#ifdef COORD_IN_ONE_VEC
  ScalorType * tmpx, * tmpy, * tmpz;
  tmpx = (ScalorType *)malloc (sizeof(ScalorType) * numMem);
  tmpy = (ScalorType *)malloc (sizeof(ScalorType) * numMem);
  tmpz = (ScalorType *)malloc (sizeof(ScalorType) * numMem);
#endif
  GromacsFileManager::readGroFile (configfile,
				   hdata.resdIndex, hdata.resdName, 
				   hdata.atomName, tmpatomIndex,
#ifndef COORD_IN_ONE_VEC
				   hdata.coordx, hdata.coordy, hdata.coordz,
#else
				   tmpx, tmpy, tmpz,
#endif
				   hdata.velox,  hdata.veloy,  hdata.veloz,
				   &bx, &by, &bz) ;
#ifdef COORD_IN_ONE_VEC
  for (IndexType i = 0; i < numAtom; ++i){
    hdata.coord[i].x = tmpx[i];
    hdata.coord[i].y = tmpy[i];
    hdata.coord[i].z = tmpz[i];
  }
  free (tmpx);
  free (tmpy);
  free (tmpz);
#endif
  freeAPointer ((void**)&tmpatomIndex);
  RectangularBoxGeometry::setBoxSize (bx, by, bz, &box);
  
  tmpNAtomType = readAtomNameMapFile (mapfile, hdata.numAtom, hdata.atomName,
				      hdata.type, hdata.mass, hdata.charge) ;
  initMass (&hdata);

  printf ("# total %d atoms found, %d types are presented in mapping file\n",
	  hdata.numAtom, tmpNAtomType);

  for (IndexType i = 0; i < hdata.numAtom; ++i){
    hdata.forcx[i] = 0.f;
    hdata.forcy[i] = 0.f;
    hdata.forcz[i] = 0.f;
  }
  
  hdata.NFreedom = hdata.numAtom * 3;

  fclose (fpc);
  
  ////////////////////////////////////////////////////////////
  // init device system
  ////////////////////////////////////////////////////////////

  initDeviceMDData (&hdata, &ddata);
  initDeviceMDData (&hdata, &recoveredDdata);
}


void MDSystem::initNBForce (const IndexType & NAtomType)
{
  if (NAtomType != 0){
    nbForce.init (NAtomType);
    printf ("# %d types of atoms will be registed for non-bonded interation\n",
	    NAtomType);
  }
  else if (tmpNAtomType != 0){
    nbForce.init (tmpNAtomType);
    printf ("# %d types of atoms will be registed for non-bonded interation\n",
	    tmpNAtomType);
  }
  else {
    throw MDExcpt0AtomType();
  }
}

void MDSystem::addNBForce (const TypeType &atomi, const TypeType &atomj, 
			   const mdNBInteraction_t & forceType,
			   const ScalorType * param)
{
  nbForce.addNBForce (atomi, atomj, forceType, param);
}

ScalorType MDSystem::
calMaxNBRcut()
{
  ScalorType max  = 0.f;
  for (IndexType i = 0; i < nbForce.setting.NNBForce; ++i){
    ScalorType tmp = calRcut (nbForce.setting.type[i],
			      &nbForce.setting.param[nbForce.setting.paramPosi[i]]);
    if (tmp > max) max = tmp;
  }
  return max;
}


MDSystem::~MDSystem()
{
  freeAPointer ((void **)&xdx);
}

  
void MDSystem::initBond (const IndexType & maxNBond)
{
  bdlist.init (ddata, maxNBond);
}

void MDSystem::addBond (const IndexType & ii, const IndexType & jj,
			const mdBondInteraction_t & type,
			const ScalorType * param)
{
  bdlist.addBond(ii, jj, type, param);
}

void MDSystem::buildBond ()
{
  bdlist.build();
}

void MDSystem::updateHost(MDTimer *timer)
{
  if (timer != NULL) timer->tic(mdTimeDataTransfer);
  cpyDeviceMDDataToHost (&ddata, &hdata);
  if (timer != NULL) timer->toc(mdTimeDataTransfer);
}

void MDSystem::initWriteXtc (const char * filename, float prec)
{
  xdfile = NULL;
  xdfile = xdrfile_open (filename, "w");
  if (xdfile == NULL){
    MDExcptCannotOpenFile ("MDSystem::initWriteXtc", filename);
  }
  for (unsigned i = 0; i < 3; ++i){
      for (unsigned j = 0; j < 3; ++j){
      	  xdbox[i][j] = 0.f;
	  }	      
	  }
  xdbox[0][0] = box.size.x;
  xdbox[1][1] = box.size.y;
  xdbox[2][2] = box.size.z;
  xdx = (rvec *) malloc (sizeof(rvec) * hdata.numMem);
  if (xdx == NULL){
    MDExcptFailedMallocOnHost ("MDSystem::initWriteXtc", "xdx", sizeof(rvec) * hdata.numMem);
  }
  xdprec = prec;
}

void MDSystem::writeHostDataXtc (int step, float time, MDTimer *timer)
{
  if (timer != NULL) timer->tic(mdTimeDataIO);
  for (IndexType i = 0; i < hdata.numAtom; ++i){
#ifndef COORD_IN_ONE_VEC
    xdx[i][0] = hdata.coordx[i];
    xdx[i][1] = hdata.coordy[i];
    xdx[i][2] = hdata.coordz[i];
#else
    xdx[i][0] = hdata.coord[i].x;
    xdx[i][1] = hdata.coord[i].y;
    xdx[i][2] = hdata.coord[i].z;
#endif
  }
  write_xtc (xdfile, hdata.numAtom, step, time, xdbox, xdx, xdprec);
  if (timer != NULL) timer->tic(mdTimeDataIO);
}

void MDSystem::endWriteXtc()
{
  xdrfile_close(xdfile);
}
