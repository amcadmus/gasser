#include "hip/hip_runtime.h"
#define DEVICE_CODE

#include "Parallel_CellList.h"
#include "Parallel_Interface.h"
#include "Parallel_CellList_device.h"
#include "Parallel_Algorithm.h"
#include "Auxiliary.h"
#include "Parallel_Timer.h"

#include "compile_error_mixcode.h"

void Parallel::DeviceCellListedMDData::
initZeroCell ()
{
  IndexType numThreadBlock = Parallel::Interface::numThreadsInCell();
  dim3 gridDim = toGridDim(numCell.x*numCell.y*numCell.z);
  Parallel::CudaGlobal::initZeroCell
      <<<gridDim, numThreadBlock>>>(
	  numCell, 
	  numAtomInCell);
  checkCUDAError ("DeviceCellListedMDData::initZeroCell");
}

void Parallel::DeviceCellListedMDData::
initCellStructure (const ScalorType & rlist_,
		   const IndexType & devideLevel_,
		   const BoxDirection_t & bdir)
{
  int Nx, Ny, Nz;
  Parallel::Interface::numProcDim (Nx, Ny, Nz);
  int ix, iy, iz;
  Parallel::Interface::rankToCartCoord (Parallel::Interface::myRank(), ix, iy, iz);
  double dx, dy, dz;
  dx = getGlobalBoxSize().x / double(Nx);
  dy = getGlobalBoxSize().y / double(Ny);
  dz = getGlobalBoxSize().z / double(Nz);
  frameLow.x = dx * ix;
  frameLow.y = dy * iy;
  frameLow.z = dz * iz;
  frameUp.x = frameLow.x + dx;
  frameUp.y = frameLow.y + dy;
  frameUp.z = frameLow.z + dz;
  
  bool CellOnX, CellOnY, CellOnZ;
  CellOnX = bdir & RectangularBoxGeometry::mdRectBoxDirectionX;
  CellOnY = bdir & RectangularBoxGeometry::mdRectBoxDirectionY;
  CellOnZ = bdir & RectangularBoxGeometry::mdRectBoxDirectionZ;
  rlist = rlist_;
  double rlisti = 1./rlist;

  if (CellOnX ) numCell.x = int ( floor(dx * rlisti) );
  else numCell.x = 1;
  if (CellOnY ) numCell.y = int ( floor(dy * rlisti) );
  else numCell.y = 1;
  if (CellOnZ ) numCell.z = int ( floor(dz * rlisti) );
  else numCell.z = 1;

  if ((CellOnX && numCell.x < 3) ||
      (CellOnY && numCell.y < 3) ||
      (CellOnZ && numCell.z < 3) ){
    throw MDExcptCellList ("Number of cell on one direction is less than 3");
  }

  // add ghost cell
  VectorType dcell;
  dcell.x = (frameUp.x - frameLow.x) / numCell.x;
  dcell.y = (frameUp.y - frameLow.y) / numCell.y;
  dcell.z = (frameUp.z - frameLow.z) / numCell.z;
  frameUp.x += dcell.x;
  frameUp.y += dcell.y;
  frameUp.z += dcell.z;
  frameLow.x -= dcell.x;
  frameLow.y -= dcell.y;
  frameLow.z -= dcell.z;
  numCell.x += 2;
  numCell.y += 2;
  numCell.z += 2;
  
  devideLevel = devideLevel_;
  if (CellOnX) numCell.x *= devideLevel;
  if (CellOnY) numCell.y *= devideLevel;
  if (CellOnZ) numCell.z *= devideLevel;

  DeviceMDData bkData (*this);
  
  IndexType numThreadsInCell = Parallel::Interface::numThreadsInCell ();
  IndexType totalNumCell = numCell.x * numCell.y * numCell.z;
  // maxNumNeighborCell = 1;
  // if (CellOnX) maxNumNeighborCell *= devideLevel * 2 + 1;
  // if (CellOnY) maxNumNeighborCell *= devideLevel * 2 + 1;
  // if (CellOnZ) maxNumNeighborCell *= devideLevel * 2 + 1;
  
  if (numThreadsInCell * totalNumCell > DeviceMDData::memSize_){
    DeviceMDData::easyMalloc (numThreadsInCell * totalNumCell);
    DeviceMDData::initZero();
  }
  numData_ = totalNumCell * numThreadsInCell;

  // printf ("rank %d, numcell %d\n", Parallel::Interface::myRank(), totalNumCell);
  // getchar ();
  // mallocCell (totalNumCell, maxNumNeighborCell);
  easyMallocCell (totalNumCell);
  initZeroCell ();

  IndexType numThreadBlock = numThreadsInCell;
  dim3 gridDim = toGridDim(numCell.x*numCell.y*numCell.z);

  Parallel::CudaGlobal::formCellStructure
      <<<gridDim, numThreadBlock >>>(
	  frameLow,
	  frameUp,
	  numCell,
	  numAtomInCell,
	  bkData.numData(),
	  bkData.dptr_coordinate(),
	  bkData.dptr_coordinateNoi(),
	  bkData.dptr_velocityX(),
	  bkData.dptr_velocityY(),
	  bkData.dptr_velocityZ(),
	  bkData.dptr_globalIndex(),
	  bkData.dptr_type(),
	  bkData.dptr_mass(),
	  bkData.dptr_charge(),
	  coord,
	  coordNoi,
	  velox,
	  veloy,
	  veloz,
	  globalIndex,
	  type,
	  mass,
	  charge,
	  err.ptr_de);
  checkCUDAError ("Parallel::formCellStructure");
  err.updateHost();
  err.check ("Parallel::formCellSturcture");
}

void Parallel::DeviceCellListedMDData::
rebuild ()
{
  IndexType numThreadBlock = Parallel::Interface::numThreadsInCell();
  IndexType totalNumCell = numCell.x*numCell.y*numCell.z;
  dim3 gridDim = toGridDim(totalNumCell);

  IndexType * bk_numAtomInCell;
  hipMalloc ((void**)&bk_numAtomInCell, totalNumCell * sizeof(IndexType));
  hipMemcpy (bk_numAtomInCell, numAtomInCell, totalNumCell * sizeof(IndexType),
	      hipMemcpyDeviceToDevice);
  checkCUDAError ("Parallel::rebuild malloc backup");
  
  Parallel::CudaGlobal::rebuildCellList_step1
      <<<gridDim, numThreadBlock>>> (
	  frameLow,
	  frameUp,
	  numCell,
	  bk_numAtomInCell,
	  numAtomInCell,
	  coord,
	  coordNoi,
	  velox,
	  veloy,
	  veloz,
	  forcx,
	  forcy,
	  forcz,
	  globalIndex,
	  type,
	  mass,
	  charge,
	  err.ptr_de);
  checkCUDAError ("Parallel::rebuild step1");
  err.updateHost();
  err.check ("Parallel::rebuild step1");
  Parallel::CudaGlobal::rebuildCellList_step2
      <<<gridDim, numThreadBlock, numThreadBlock*sizeof(IndexType)*3>>> (
	  numAtomInCell,
	  coord,
	  coordNoi,
	  velox,
	  veloy,
	  veloz,
	  forcx,
	  forcy,
	  forcz,
	  globalIndex,
	  type,
	  mass,
	  charge,
	  err.ptr_de);
  checkCUDAError ("Parallel::rebuild step2");
  err.updateHost();
  err.check ("Parallel::rebuild step2");
  hipFree (bk_numAtomInCell);
  checkCUDAError ("Parallel::rebuild free backup");  
}
	  

__global__ void Parallel::CudaGlobal::
initZeroCell (const IntVectorType numCell,
	      IndexType * numAtomInCell)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  IndexType totalNumCell = numCell.x * numCell.y * numCell.z;

  if (ii < totalNumCell){
    numAtomInCell[ii] = 0;
    // numNeighborCell[ii] = 0;
  }
}


__global__ void Parallel::CudaGlobal::
formCellStructure (const VectorType frameLow,
		   const VectorType frameUp,
		   const IntVectorType numCell,
		   IndexType * numAtomInCell,
		   const IndexType numAtom,
		   const CoordType  * bk_coord,
		   const CoordNoiType * bk_coordNoi,
		   const ScalorType * bk_velox,
		   const ScalorType * bk_veloy,
		   const ScalorType * bk_veloz,
		   const IndexType  * bk_globalIndex,
		   const TypeType   * bk_type,
		   const ScalorType * bk_mass,
		   const ScalorType * bk_charge,
		   CoordType  * coord,
		   CoordNoiType * coordNoi,
		   ScalorType * velox,
		   ScalorType * veloy,
		   ScalorType * veloz,
		   IndexType  * globalIndex,
		   TypeType   * type,
		   ScalorType * mass,
		   ScalorType * charge,
		   mdError_t * ptr_de)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;

  __shared__ ScalorType dcellxi;
  __shared__ ScalorType dcellyi;
  __shared__ ScalorType dcellzi;
  if (tid == 0) {
    dcellxi = ScalorType(numCell.x) / (frameUp.x - frameLow.x);
  }
  if (tid == 1){
    dcellyi = ScalorType(numCell.y) / (frameUp.y - frameLow.y);
  }
  if (tid == 2){
    dcellzi = ScalorType(numCell.z) / (frameUp.z - frameLow.z);
  }
  __syncthreads();
  
  IndexType targetIndex;
  if (ii < numAtom){
    IndexType targetCellx, targetCelly, targetCellz;
    targetCellx = IndexType((bk_coord[ii].x - frameLow.x) * dcellxi);
    targetCelly = IndexType((bk_coord[ii].y - frameLow.y) * dcellyi);
    targetCellz = IndexType((bk_coord[ii].z - frameLow.z) * dcellzi);
    // if (targetCellx == numCell.x){
    //   targetCellx = numCell.x - 1;
    // }
    // if (targetCelly == numCell.y){
    //   targetCelly = numCell.y - 1;
    // }
    // if (targetCellz == numCell.z){
    //   targetCellz = numCell.z - 1;
    // }
    if (ptr_de != NULL && 
	(targetCellx >= numCell.x || 
	 targetCelly >= numCell.y || 
	 targetCellz >= numCell.z)){
      *ptr_de = mdErrorOverFlowCellIdx;
      return;
    }

    IndexType cellid = CudaDevice::D3toD1
	(numCell, targetCellx, targetCelly, targetCellz);

    IndexType pid = atomicAdd (&numAtomInCell[cellid], 1);
    if (pid >= blockDim.x){
      *ptr_de = mdErrorShortCellList;
      pid = 0;
    }
    targetIndex = pid + cellid * blockDim.x;
    coord[targetIndex] = bk_coord[ii];
    coordNoi[targetIndex].x = bk_coordNoi[ii].x;
    coordNoi[targetIndex].y = bk_coordNoi[ii].y;
    coordNoi[targetIndex].z = bk_coordNoi[ii].z;
    velox[targetIndex] = bk_velox[ii];
    veloy[targetIndex] = bk_veloy[ii];
    veloz[targetIndex] = bk_veloz[ii];
    globalIndex[targetIndex] = bk_globalIndex[ii];
    type[targetIndex] = bk_type[ii];
    mass[targetIndex] = bk_mass[ii];
    charge[targetIndex] = bk_charge[ii];
  }
}

__global__ void Parallel::CudaGlobal::
rebuildCellList_step1 (const VectorType frameLow,
		       const VectorType frameUp,
		       const IntVectorType numCell,
		       const IndexType * bk_numAtomInCell,
		       IndexType * numAtomInCell,
		       CoordType * coord,
		       CoordNoiType * coordNoi,
		       ScalorType * velox,
		       ScalorType * veloy,
		       ScalorType * veloz,
		       ScalorType * forcx,
		       ScalorType * forcy,
		       ScalorType * forcz,
		       IndexType  * globalIndex,
		       TypeType   * type,
		       ScalorType * mass,
		       ScalorType * charge,
		       mdError_t * ptr_de)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;

  __shared__ ScalorType dcellxi;
  __shared__ ScalorType dcellyi;
  __shared__ ScalorType dcellzi;
  if (tid == 0) {
    dcellxi = ScalorType(numCell.x) / (frameUp.x - frameLow.x);
  }
  if (tid == 1){
    dcellyi = ScalorType(numCell.y) / (frameUp.y - frameLow.y);
  }
  if (tid == 2){
    dcellzi = ScalorType(numCell.z) / (frameUp.z - frameLow.z);
  }
  __syncthreads();

  // IndexType mark = MaxIndexValue - (MaxIndexValue >> 1);
  // IndexType mystat ;
  
  if (tid < bk_numAtomInCell[bid]){
    // mystat = globalIndex[ii];
    IndexType targetCellx, targetCelly, targetCellz;
    targetCellx = IndexType((coord[ii].x - frameLow.x) * dcellxi);
    targetCelly = IndexType((coord[ii].y - frameLow.y) * dcellyi);
    targetCellz = IndexType((coord[ii].z - frameLow.z) * dcellzi);
    // printf ("%d %d %d %d %f %f %f\n", ii, targetCellx, targetCelly, targetCellz,
    // 	    coord[ii].x, coord[ii].y, coord[ii].z);
    // if (targetCellx == numCell.x){
    //   targetCellx = numCell.x - 1;
    // }
    // if (targetCelly == numCell.y){
    //   targetCelly = numCell.y - 1;
    // }
    // if (targetCellz == numCell.z){
    //   targetCellz = numCell.z - 1;
    // }
    if (ptr_de != NULL && 
	(targetCellx >= numCell.x || 
	 targetCelly >= numCell.y || 
	 targetCellz >= numCell.z)){
      *ptr_de = mdErrorOverFlowCellIdx;
      return;
    }
    IndexType cellid = CudaDevice::D3toD1
	(numCell, targetCellx, targetCelly, targetCellz);
    if (cellid != bid){
      // IndexType pid = atomicAdd (&numAtomInCell[cellid], 1);
      // if (pid >= blockDim.x){
      // 	*ptr_de = mdErrorShortCellList;
      // 	pid = 0;
      // }
      IndexType pid = atomicAdd (&numAtomInCell[cellid], 1);
      if (pid >= blockDim.x){
	*ptr_de = mdErrorShortCellList;
	pid = 0;
      }
      IndexType targetIndex = pid + cellid * blockDim.x;
      coord[targetIndex] = coord[ii];
      coordNoi[targetIndex].x = coordNoi[ii].x;
      coordNoi[targetIndex].y = coordNoi[ii].y;
      coordNoi[targetIndex].z = coordNoi[ii].z;
      velox[targetIndex] = velox[ii];
      veloy[targetIndex] = veloy[ii];
      veloz[targetIndex] = veloz[ii];
      forcx[targetIndex] = forcx[ii];
      forcy[targetIndex] = forcy[ii];
      forcz[targetIndex] = forcz[ii];
      globalIndex[targetIndex] = globalIndex[ii];
      globalIndex[ii] = MaxIndexValue;
      type[targetIndex] = type[ii];
      mass[targetIndex] = mass[ii];
      charge[targetIndex] = charge[ii];
    }
  }
  // globalIndex[ii] = mystat;
  return;
}


static __device__ IndexType
headSort (volatile IndexType * index,
	  volatile IndexType * sbuff)
{
  IndexType k = NUintBit - 1;
  IndexType tid = threadIdx.x;
  sbuff[tid] = getKthBit(index[tid], k);
  sbuff[tid+blockDim.x] = 0;
  
  __syncthreads();
  IndexType total1 = sumVectorBlockBuffer (sbuff, blockDim.x);
  IndexType target, mydata = index[tid];
  __syncthreads();
  
  if (getKthBit(index[tid], k)) {
    target = blockDim.x - sbuff[tid];
  }
  else {
    target = tid + sbuff[tid] - total1;
  }
  __syncthreads();
  index[target] = mydata;
  __syncthreads();

  return total1;
}


__global__ void Parallel::CudaGlobal::
rebuildCellList_step2 (IndexType * numAtomInCell,
		       CoordType  * coord,
		       CoordNoiType * coordNoi,
		       ScalorType * velox,
		       ScalorType * veloy,
		       ScalorType * veloz,
		       ScalorType * forcx,
		       ScalorType * forcy,
		       ScalorType * forcz,
		       IndexType  * globalIndex,
		       TypeType   * type,
		       ScalorType * mass,
		       ScalorType * charge,
		       mdError_t * ptr_de)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  // IndexType k = NUintBit - 1;
  
  extern __shared__ volatile IndexType sbuff[];
  volatile IndexType * myIndex = (volatile IndexType * )sbuff;

  if (tid >= numAtomInCell[bid] || globalIndex[ii] == MaxIndexValue){
    myIndex[tid] = MaxIndexValue;
  }
  else {
    myIndex[tid] = tid;
  }
  __syncthreads();
  IndexType total = headSort (myIndex, &sbuff[blockDim.x]);
  total = blockDim.x - total;

  IndexType fromId;
  CoordType bk_coord;
  CoordNoiType bk_coordNoi;
  ScalorType bk_velox, bk_veloy, bk_veloz;
  ScalorType bk_forcx, bk_forcy, bk_forcz;
  IndexType bk_globalIndex;
  TypeType bk_type;
  ScalorType bk_mass;
  ScalorType bk_charge;
  
  if (tid < total){
    fromId = myIndex[tid] + bid * blockDim.x;
    if (ii != fromId){
      bk_coord = coord[fromId];
      bk_coordNoi.x = coordNoi[fromId].x;
      bk_coordNoi.y = coordNoi[fromId].y;
      bk_coordNoi.z = coordNoi[fromId].z;
      bk_velox = velox[fromId];
      bk_veloy = veloy[fromId];
      bk_veloz = veloz[fromId];
      bk_forcx = forcx[fromId];
      bk_forcy = forcy[fromId];
      bk_forcz = forcz[fromId];
      bk_globalIndex = globalIndex[fromId];
      bk_type = type[fromId];
      bk_mass = mass[fromId];
      bk_charge = charge[fromId];
    }
  }
  __syncthreads();

  if (tid < total && ii != fromId){
    coord[ii] = bk_coord;
    coordNoi[ii].x = bk_coordNoi.x;
    coordNoi[ii].y = bk_coordNoi.y;
    coordNoi[ii].z = bk_coordNoi.z;
    velox[ii] = bk_velox;
    veloy[ii] = bk_veloy;
    veloz[ii] = bk_veloz;
    forcx[ii] = bk_forcx;
    forcy[ii] = bk_forcy;
    forcz[ii] = bk_forcz;
    globalIndex[ii] = bk_globalIndex;
    type[ii] = bk_type;
    mass[ii] = bk_mass;
    charge[ii] = bk_charge;
  }

    
  // else {
  //   globalIndex[ii] = MaxIndexValue;
  // }  

  if (tid == 0){
    numAtomInCell[bid] = total;
  }
}


void Parallel::DeviceCellListedMDData::
easyMallocCell (const IndexType & totalNumCell)
{
  if (totalNumCell == 0) return;
  // if (totalNumCell == numCell.x * numCell.y * numCell.z) return;
  // maxNumNeighborCell = maxNumNeighborCell_;
  clearCell ();
  hipMalloc ((void**)&numAtomInCell, sizeof(IndexType) * totalNumCell);
  // hipMalloc ((void**)&numNeighborCell, sizeof(IndexType) * totalNumCell);
  // hipMalloc ((void**)&neighborCellIndex,
  // 	      sizeof(IndexType) * totalNumCell * maxNumNeighborCell);
  memSize = totalNumCell;
  checkCUDAError ("malloc Cell");
  malloced = true;
}

void Parallel::DeviceCellListedMDData::
clearCell()
{
  if (malloced){
    hipFree (numAtomInCell);
    memSize = 0;
    // hipFree (numNeighborCell);
    // hipFree (neighborCellIndex);
    malloced = false;
  }
}

Parallel::DeviceCellListedMDData::
DeviceCellListedMDData ()
{
  rlist = 0;
  devideLevel = 0;
  frameLow.x = frameLow.y = frameLow.z = 0;
  frameUp.x  = frameUp.y  = frameUp.z  = 0;
  numCell.x  = numCell.y  = numCell.z  = 0;
  memSize = 0;
  // maxNumNeighborCell = 0;
  malloced = false;
}

Parallel::DeviceCellListedMDData::
~DeviceCellListedMDData()
{
  clearCell();
}

Parallel::SubCellList::
SubCellList ()
{
}


void Parallel::SubCellList::
build ()
{
  Parallel::Interface::sort (this->begin(), this->end());
}

bool Parallel::SubCellList::
isBuilt ()
{
  return (Parallel::Interface::is_sorted (this->begin(), this->end()));
}

void Parallel::SubCellList::
add (const SubCellList & a)
{
  for (std::vector<IndexType>::const_iterator it = a.begin();
       it != a.end(); ++it){
    push_back (*it);
  }
  Parallel::Interface::unique (this->begin(), this->end());
  Parallel::Interface::sort   (this->begin(), this->end());
}

void Parallel::SubCellList::
sub (const SubCellList & a)
{
  std::vector<IndexType > result (this->size());
  std::vector<IndexType >::iterator newend =
      Parallel::Interface::set_difference (this->begin(), this->end(),
					   a.begin(), a.end(),
					   result.begin());
  std::vector<IndexType >::iterator newend2 =
      Parallel::Interface::copy (result.begin(), newend, this->begin());
  this->erase (newend2, this->end());
}



void Parallel::DeviceCellListedMDData::
buildSubList (const IndexType & xIdLo,
	      const IndexType & xIdUp,
	      const IndexType & yIdLo,
	      const IndexType & yIdUp,
	      const IndexType & zIdLo,
	      const IndexType & zIdUp,
	      SubCellList & subList)
{
  if (xIdUp > numCell.x){
    throw MDExcptCellList ("x up index exceeds number of cells on x");
  }
  if (yIdUp > numCell.y){
    throw MDExcptCellList ("y up index exceeds number of cells on y");
  }
  if (zIdUp > numCell.z){
    throw MDExcptCellList ("z up index exceeds number of cells on z");
  }

  subList.clear();
  
  for (IndexType i = xIdLo; i < xIdUp; ++i){
    for (IndexType j = yIdLo; j < yIdUp; ++j){
      for (IndexType k = zIdLo; k < zIdUp; ++k){
	subList.push_back ( D3toD1 (i, j, k));
      }
    }
  }
}


Parallel::DeviceTransferPackage::
DeviceTransferPackage ()
    : numCell (0), memSize(0), hcellIndex(NULL), hcellStartIndex(NULL),
      myMask (MDDataItemMask_All)
{
}

void Parallel::DeviceTransferPackage::
clearMe ()
{
  if (memSize != 0){
    hipFree (cellIndex);
    hipFree (cellStartIndex);
    freeAPointer ((void**)&hcellIndex);
    freeAPointer ((void**)&hcellStartIndex);
    memSize = 0;
    numCell = 0;
  }
}

Parallel::DeviceTransferPackage::
~DeviceTransferPackage ()
{
  clearMe();
}

void Parallel::DeviceTransferPackage::
easyMallocMe (IndexType memSize_)
{
  if (memSize_ == 0) return;
  // if (memSize == memSize_) return;
  clearMe ();
  memSize = memSize_;
  size_t size = memSize * sizeof(IndexType);
  size_t size1 = (memSize+1) * sizeof(IndexType);
  hipMalloc ((void**)&cellIndex, size);
  hipMalloc ((void**)&cellStartIndex, size1);
  checkCUDAError ("DeviceTransferPackage::mallocMe failed malloc");
  hcellIndex = (IndexType *) malloc (size);
  if (hcellIndex == NULL){
    throw MDExcptFailedMallocOnHost ("DeviceTransferPackage::reinit",
				     "hcellIndex", size);
  }
  hcellStartIndex = (IndexType *) malloc (size1);
  if (hcellStartIndex == NULL){
    throw MDExcptFailedMallocOnHost ("DeviceTransferPackage::reinit",
				     "hcellStartIndex", size1);
  }
}

void Parallel::DeviceTransferPackage::
reinit (const SubCellList & subCellList)
{
  if (memSize < subCellList.size()){
    easyMallocMe (subCellList.size()*MemAllocExtension);
  }
  numCell = subCellList.size();
  for (IndexType i = 0; i < numCell; ++i){
    hcellIndex[i] = subCellList[i];
  }
  size_t size = memSize * sizeof(IndexType);
  hipMemcpy (cellIndex, hcellIndex, size, hipMemcpyHostToDevice);
  checkCUDAError ("DeviceTransferPackage::reinit memcpy");
}

void Parallel::DeviceTransferPackage::
pack (const DeviceCellListedMDData & ddata,
      const MDDataItemMask_t mask)
{
  if (numCell == 0) return;
  myMask = mask;
  
  IndexType totalNumCell = ddata.numCell.x * ddata.numCell.y * ddata.numCell.z;
  IndexType * numAtomInCell ;
  size_t size = totalNumCell * sizeof(IndexType);
  
  numAtomInCell = (IndexType *) malloc (size);
  if (numAtomInCell == NULL){
    throw MDExcptFailedMallocOnHost ("DeviceTransferPackage::reinit",
				     "numAtomInCell", size);
  }
  hipMemcpy (numAtomInCell, ddata.numAtomInCell, size,
	      hipMemcpyDeviceToHost);
  checkCUDAError ("DeviceTransferPackage::pack cpy numAtomInCell to host");
  
  hcellStartIndex[0] = 0;
  for (IndexType i = 1; i < numCell+1; ++i){
    hcellStartIndex[i] = hcellStartIndex[i-1] + numAtomInCell[hcellIndex[i-1]];
  }
  this->numData() = hcellStartIndex[numCell];
  hipMemcpy (cellStartIndex, hcellStartIndex, (numCell+1) * sizeof(IndexType),
	      hipMemcpyHostToDevice);
  checkCUDAError ("DeviceTransferPackage::pack cpy cellStartIndex to device");
  
  free (numAtomInCell);

  this->DeviceMDData::setGlobalBox (ddata.getGlobalBox());
  if (this->DeviceMDData::numData() > this->DeviceMDData::memSize()){
    printf ("# DeviceTransferPackage::pack, realloc\n");
    
    this->DeviceMDData::easyMalloc (this->DeviceMDData::numData() * MemAllocExtension);
  }

  checkCUDAError ("DeviceTransferPackage::pack, packDeviceMDData, before");
  Parallel::CudaGlobal::packDeviceMDData
      <<<numCell, Parallel::Interface::numThreadsInCell()>>> (
	  cellIndex,
	  ddata.dptr_numAtomInCell(),
	  cellStartIndex,
	  mask,
	  ddata.dptr_coordinate(),
	  ddata.dptr_coordinateNoi(),
	  ddata.dptr_velocityX(),
	  ddata.dptr_velocityY(),
	  ddata.dptr_velocityZ(),
	  ddata.dptr_forceX(),
	  ddata.dptr_forceY(),
	  ddata.dptr_forceZ(),
	  ddata.dptr_globalIndex(),
	  ddata.dptr_type(),
	  ddata.dptr_mass(),
	  ddata.dptr_charge(),
	  this->dptr_coordinate(),
	  this->dptr_coordinateNoi(),
	  this->dptr_velocityX(),
	  this->dptr_velocityY(),
	  this->dptr_velocityZ(),
	  this->dptr_forceX(),
	  this->dptr_forceY(),
	  this->dptr_forceZ(),
	  this->dptr_globalIndex(),
	  this->dptr_type(),
	  this->dptr_mass(),
	  this->dptr_charge());
  checkCUDAError ("DeviceTransferPackage::pack, packDeviceMDData");
}


__global__ void Parallel::CudaGlobal::
packDeviceMDData (const IndexType * cellIndex,
		  const IndexType * numAtomInCell,
		  const IndexType * cellStartIndex,
		  const MDDataItemMask_t mask,
		  const CoordType  * source_coord,
		  const CoordNoiType * source_coordNoi,
		  const ScalorType * source_velox,
		  const ScalorType * source_veloy,
		  const ScalorType * source_veloz,
		  const ScalorType * source_forcx,
		  const ScalorType * source_forcy,
		  const ScalorType * source_forcz,
		  const IndexType  * source_globalIndex,
		  const TypeType   * source_type,
		  const ScalorType * source_mass,
		  const ScalorType * source_charge,
		  CoordType  * coord,
		  CoordNoiType * coordNoi,
		  ScalorType * velox,
		  ScalorType * veloy,
		  ScalorType * veloz,
		  ScalorType * forcx,
		  ScalorType * forcy,
		  ScalorType * forcz,
		  IndexType  * globalIndex,
		  TypeType   * type,
		  ScalorType * mass,
		  ScalorType * charge)		  
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;

  IndexType cellIdx = cellIndex[bid];
  IndexType fromid = tid + cellIdx * blockDim.x;
  IndexType toid = tid + cellStartIndex[bid];
  
  if (tid < numAtomInCell[cellIdx]){
    if (mask & MDDataItemMask_Coordinate){
      coord[toid] = source_coord[fromid];
    }
    if (mask & MDDataItemMask_CoordinateNoi){
      coordNoi[toid].x = source_coordNoi[fromid].x;
    }
    if (mask & MDDataItemMask_Velocity){
      velox[toid] = source_velox[fromid];
      veloy[toid] = source_veloy[fromid];
      veloz[toid] = source_veloz[fromid];
    }
    if (mask & MDDataItemMask_Force){
      forcx[toid] = source_forcx[fromid];
      forcy[toid] = source_forcy[fromid];
      forcz[toid] = source_forcz[fromid];
    }
    if (mask & MDDataItemMask_GlobalIndex){
      globalIndex[toid] = source_globalIndex[fromid];
    }
    if (mask & MDDataItemMask_Type){
      type[toid] = source_type[fromid];
    }
    if (mask & MDDataItemMask_Mass){
      mass[toid] = source_mass[fromid];
    }
    if (mask & MDDataItemMask_Charge){
      charge[toid] = source_charge[fromid];
    }
  }
}


void Parallel::DeviceTransferPackage::
copyToHost (HostTransferPackage & hpkg) const
{
  HostMDData & hdata(hpkg);
  const DeviceMDData & ddata(*this);  
  ddata.copyToHost (hdata, myMask);
  // alloc memory
  if (hpkg.getMemSize() < numCell){
    hpkg.easyMallocMe (numCell * MemAllocExtension);
  }
  hpkg.getTotalNumCell() = numCell;
  hpkg.getMask() = myMask;
  for (IndexType i = 0; i < numCell; ++i){
    hpkg.getCellIndex()[i] = hcellIndex[i];
    hpkg.getCellStartIndex()[i] = hcellStartIndex[i];
  }
  hpkg.getCellStartIndex()[numCell] = hcellStartIndex[numCell];  
}

void Parallel::DeviceTransferPackage::
copyFromHost (const HostTransferPackage & hpkg)
{
  const HostMDData & hdata(hpkg);
  DeviceMDData & ddata(*this);
  myMask = hpkg.getMask();
  ddata.copyFromHost (hdata, myMask);

  if (memSize < hpkg.getTotalNumCell()){
    easyMallocMe (hpkg.getTotalNumCell() * MemAllocExtension);
  }
  numCell = hpkg.getTotalNumCell();
  for (IndexType i = 0; i < numCell; ++i){
    hcellIndex[i] = hpkg.getCellIndex()[i];
    hcellStartIndex[i] = hpkg.getCellStartIndex()[i];
  }
  hcellStartIndex[numCell] = hpkg.getCellStartIndex()[numCell];

  hipMemcpy (cellIndex, hcellIndex, sizeof(IndexType)*numCell,
	      hipMemcpyHostToDevice);
  hipMemcpy (cellStartIndex, hcellStartIndex, sizeof(IndexType)*(numCell+1),
	      hipMemcpyHostToDevice);
  checkCUDAError ("DeviceTransferPackage::copyFromHost cpy from host");
}


void Parallel::DeviceTransferPackage::
unpack_replace (DeviceCellListedMDData & ddata) const
{
  Parallel::CudaGlobal::unpackDeviceMDData_replace
      <<<numCell, Parallel::Interface::numThreadsInCell()>>> (
	  cellIndex,
	  cellStartIndex,
	  myMask,
	  this->dptr_coordinate(),
	  this->dptr_coordinateNoi(),
	  this->dptr_velocityX(),
	  this->dptr_velocityY(),
	  this->dptr_velocityZ(),
	  this->dptr_forceX(),
	  this->dptr_forceY(),
	  this->dptr_forceZ(),
	  this->dptr_globalIndex(),
	  this->dptr_type(),
	  this->dptr_mass(),
	  this->dptr_charge(),
	  ddata.numAtomInCell,
	  ddata.dptr_coordinate(),
	  ddata.dptr_coordinateNoi(),
	  ddata.dptr_velocityX(),
	  ddata.dptr_velocityY(),
	  ddata.dptr_velocityZ(),
	  ddata.dptr_forceX(),
	  ddata.dptr_forceY(),
	  ddata.dptr_forceZ(),
	  ddata.dptr_globalIndex(),
	  ddata.dptr_type(),
	  ddata.dptr_mass(),
	  ddata.dptr_charge());
  checkCUDAError ("DeviceTransferPackage::unpack_replace");
}


void Parallel::DeviceTransferPackage::
unpack_add (DeviceCellListedMDData & ddata) const
{
  Parallel::CudaGlobal::unpackDeviceMDData_add
      <<<numCell, Parallel::Interface::numThreadsInCell()>>> (
	  cellIndex,
	  cellStartIndex,
	  myMask,
	  this->dptr_coordinate(),
	  this->dptr_coordinateNoi(),
	  this->dptr_velocityX(),
	  this->dptr_velocityY(),
	  this->dptr_velocityZ(),
	  this->dptr_forceX(),
	  this->dptr_forceY(),
	  this->dptr_forceZ(),
	  this->dptr_globalIndex(),
	  this->dptr_type(),
	  this->dptr_mass(),
	  this->dptr_charge(),
	  ddata.numAtomInCell,
	  ddata.dptr_coordinate(),
	  ddata.dptr_coordinateNoi(),
	  ddata.dptr_velocityX(),
	  ddata.dptr_velocityY(),
	  ddata.dptr_velocityZ(),
	  ddata.dptr_forceX(),
	  ddata.dptr_forceY(),
	  ddata.dptr_forceZ(),
	  ddata.dptr_globalIndex(),
	  ddata.dptr_type(),
	  ddata.dptr_mass(),
	  ddata.dptr_charge(),
	  err.ptr_de);
  checkCUDAError ("DeviceTransferPackage::unpack_add");
  err.updateHost ();
  err.check ("DeviceTransferPackage::unpack_add");
}



__global__ void Parallel::CudaGlobal::
unpackDeviceMDData_replace (const IndexType * cellIndex,
			  const IndexType * cellStartIndex,
			  const MDDataItemMask_t mask,
			  const CoordType  * source_coord,
			  const CoordNoiType * source_coordNoi,
			  const ScalorType * source_velox,
			  const ScalorType * source_veloy,
			  const ScalorType * source_veloz,
			  const ScalorType * source_forcx,
			  const ScalorType * source_forcy,
			  const ScalorType * source_forcz,
			  const IndexType  * source_globalIndex,
			  const TypeType   * source_type,
			  const ScalorType * source_mass,
			  const ScalorType * source_charge,
			  IndexType * numAtomInCell,
			  CoordType  * coord,
			  CoordNoiType * coordNoi,
			  ScalorType * velox,
			  ScalorType * veloy,
			  ScalorType * veloz,
			  ScalorType * forcx,
			  ScalorType * forcy,
			  ScalorType * forcz,
			  IndexType  * globalIndex,
			  TypeType   * type,
			  ScalorType * mass,
			  ScalorType * charge)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;

  IndexType cellIdx = cellIndex[bid];
  IndexType startIdx = cellStartIndex[bid];
  IndexType numAtomInThisCell = cellStartIndex[bid+1] - startIdx;
  IndexType toid   = tid + cellIdx * blockDim.x;
  IndexType fromid = tid + startIdx;
  if (tid == blockDim.x-1){
    numAtomInCell[cellIdx] = numAtomInThisCell;
  }
  
  if (tid < numAtomInThisCell){
    if (mask & MDDataItemMask_Coordinate){
      coord[toid] = source_coord[fromid];
    }
    if (mask & MDDataItemMask_CoordinateNoi){
      coordNoi[toid].x = source_coordNoi[fromid].x;
      coordNoi[toid].y = source_coordNoi[fromid].y;
      coordNoi[toid].z = source_coordNoi[fromid].z;
    }
    if (mask & MDDataItemMask_Velocity){
      velox[toid] = source_velox[fromid];
      veloy[toid] = source_veloy[fromid];
      veloz[toid] = source_veloz[fromid];
    }
    if (mask & MDDataItemMask_Force){
      forcx[toid] = source_forcx[fromid];
      forcy[toid] = source_forcy[fromid];
      forcz[toid] = source_forcz[fromid];
    }
    if (mask & MDDataItemMask_GlobalIndex){
      globalIndex[toid] = source_globalIndex[fromid];
    }
    if (mask & MDDataItemMask_Type){
      type[toid] = source_type[fromid];
    }
    if (mask & MDDataItemMask_Mass){
      mass[toid] = source_mass[fromid];
    }
    if (mask & MDDataItemMask_Charge){
      charge[toid] = source_charge[fromid];
    }
  }
}


__global__ void Parallel::CudaGlobal::
unpackDeviceMDData_add (const IndexType * cellIndex,
			const IndexType * cellStartIndex,
			const MDDataItemMask_t mask,
			const CoordType  * source_coord,
			const CoordNoiType * source_coordNoi,
			const ScalorType * source_velox,
			const ScalorType * source_veloy,
			const ScalorType * source_veloz,
			const ScalorType * source_forcx,
			const ScalorType * source_forcy,
			const ScalorType * source_forcz,
			const IndexType  * source_globalIndex,
			const TypeType   * source_type,
			const ScalorType * source_mass,
			const ScalorType * source_charge,
			IndexType * numAtomInCell,
			CoordType  * coord,
			CoordNoiType * coordNoi,
			ScalorType * velox,
			ScalorType * veloy,
			ScalorType * veloz,
			ScalorType * forcx,
			ScalorType * forcy,
			ScalorType * forcz,
			IndexType  * globalIndex,
			TypeType   * type,
			ScalorType * mass,
			ScalorType * charge,
			mdError_t * ptr_de)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;

  __shared__ volatile IndexType tmpbuff[2];
  if (tid < 2){
    tmpbuff[tid] = cellStartIndex[bid+tid];
  }
  __syncthreads();
  IndexType startIdx = tmpbuff[0];
  IndexType numAdded = tmpbuff[1] - startIdx;
  // IndexType startIdx = cellStartIndex[bid];
  // IndexType numAdded = cellStartIndex[bid+1] - startIdx;
  if (numAdded == 0) return;
  
  IndexType cellIdx = cellIndex[bid];
  IndexType alreadyInCell = numAtomInCell[cellIdx];
  IndexType toid   = tid + cellIdx * blockDim.x + alreadyInCell;
  IndexType fromid = tid + startIdx;
  // __shared__ IndexType numAtomInThisCell;
  
  __syncthreads();
  __shared__ volatile bool failed ;
  if (tid == 0){
    if (((numAtomInCell[cellIdx] += numAdded)) > blockDim.x &&
	ptr_de != NULL){
      *ptr_de = mdErrorShortCellList;
      failed = true;
    }
    else {
      failed = false;
    }
  }
  __syncthreads();
  if (failed) return;
  
  if (tid < numAdded){
    if (mask & MDDataItemMask_Coordinate){
      coord[toid] = source_coord[fromid];
    }
    if (mask & MDDataItemMask_CoordinateNoi){
      coordNoi[toid].x = source_coordNoi[fromid].x;
      coordNoi[toid].y = source_coordNoi[fromid].y;
      coordNoi[toid].z = source_coordNoi[fromid].z;
    }
    if (mask & MDDataItemMask_Velocity){
      velox[toid] = source_velox[fromid];
      veloy[toid] = source_veloy[fromid];
      veloz[toid] = source_veloz[fromid];
    }
    if (mask & MDDataItemMask_Force){
      forcx[toid] = source_forcx[fromid];
      forcy[toid] = source_forcy[fromid];
      forcz[toid] = source_forcz[fromid];
    }
    if (mask & MDDataItemMask_GlobalIndex){
      globalIndex[toid] = source_globalIndex[fromid];
    }
    if (mask & MDDataItemMask_Type){
      type[toid] = source_type[fromid];
    }
    if (mask & MDDataItemMask_Mass){
      mass[toid] = source_mass[fromid];
    }
    if (mask & MDDataItemMask_Charge){
      charge[toid] = source_charge[fromid];
    }
  }
}


void Parallel::DeviceCellListedMDData::
copyToHost (HostCellListedMDData & hdata,
	    const MDDataItemMask_t mask) const
{
  const DeviceMDData & ddata (*this);
  ddata.copyToHost (hdata, mask);

  hdata.rlist = rlist;
  hdata.devideLevel = devideLevel;
  hdata.numCell.x = numCell.x;
  hdata.numCell.y = numCell.y;
  hdata.numCell.z = numCell.z;
  hdata.frameUp.x = frameUp.x;
  hdata.frameUp.y = frameUp.y;
  hdata.frameUp.z = frameUp.z;
  hdata.frameLow.x = frameLow.x;
  hdata.frameLow.y = frameLow.y;
  hdata.frameLow.z = frameLow.z;

  IndexType totalNumCell = numCell.x * numCell.y * numCell.z;
  size_t size = totalNumCell * sizeof (IndexType);

  if (hdata.HostCellListedMDData::memSize < totalNumCell){
    hdata.easyReallocCell (totalNumCell * MemAllocExtension);
  }
  hipMemcpy (hdata.numAtomInCell, numAtomInCell, size,
	      hipMemcpyDeviceToHost);
  checkCUDAError ("DeviceCellListedMDData::copyToHost copy numAtomInCell");
}

  
void Parallel::DeviceCellListedMDData::
copyFromHost (const HostCellListedMDData & hdata,
	      const MDDataItemMask_t mask)
{
  DeviceMDData & ddata(*this);
  ddata.copyFromHost (hdata, mask);

  rlist = hdata.rlist;
  devideLevel = hdata.devideLevel;
  numCell.x = hdata.numCell.x;
  numCell.y = hdata.numCell.y;
  numCell.z = hdata.numCell.z;
  frameUp.x = hdata.frameUp.x;
  frameUp.y = hdata.frameUp.y;
  frameUp.z = hdata.frameUp.z;
  frameLow.x = hdata.frameLow.x;
  frameLow.y = hdata.frameLow.y;
  frameLow.z = hdata.frameLow.z;
  
  IndexType totalNumCell = numCell.x * numCell.y * numCell.z;
  size_t size = totalNumCell * sizeof (IndexType);

  if (memSize < totalNumCell){
    easyMallocCell (totalNumCell * MemAllocExtension);
  }
  hipMemcpy (numAtomInCell, hdata.numAtomInCell, size,
	      hipMemcpyHostToDevice);
  checkCUDAError ("DeviceCellListedMDData::copyFromHost copy numAtomInCell");
}

void Parallel::DeviceCellListedMDData::
copyFromDevice (const DeviceCellListedMDData & ddata,
		const MDDataItemMask_t mask)
{  
  DeviceMDData & me(*this);
  me.copyFromDevice (ddata, mask);

  rlist = ddata.rlist;
  devideLevel = ddata.devideLevel;
  numCell.x = ddata.numCell.x;
  numCell.y = ddata.numCell.y;
  numCell.z = ddata.numCell.z;
  frameUp.x = ddata.frameUp.x;
  frameUp.y = ddata.frameUp.y;
  frameUp.z = ddata.frameUp.z;
  frameLow.x = ddata.frameLow.x;
  frameLow.y = ddata.frameLow.y;
  frameLow.z = ddata.frameLow.z;
  
  IndexType totalNumCell = numCell.x * numCell.y * numCell.z;
  size_t size = totalNumCell * sizeof (IndexType);

  if (memSize < totalNumCell){
    easyMallocCell (totalNumCell * MemAllocExtension);
  }

  hipMemcpy (numAtomInCell, ddata.numAtomInCell, size,
	      hipMemcpyDeviceToDevice);
  checkCUDAError ("DeviceCellListedMDData::copyFromDevice copy numAtomInCell");
}


void Parallel::DeviceCellListedMDData::
clearData (const SubCellList & subList)
{
  IndexType * tmpList;
  IndexType * deviceList;
  IndexType num = subList.size();
  size_t size = sizeof(IndexType) * num;
  
  tmpList = (IndexType * )malloc (size);
  if (tmpList == NULL){
    throw MDExcptFailedMallocOnHost ("DeviceCellListedMDData::clearData",
				     "tmpList", size);
  }
  hipMalloc ((void**)&deviceList, size);
  checkCUDAError ("DeviceCellListedMDData::clearData, malloc deviceList");

  for (IndexType i = 0; i < num; ++i){
    tmpList[i] = subList[i];
  }
  hipMemcpy (deviceList, tmpList, size, hipMemcpyHostToDevice);
  freeAPointer ((void**)&tmpList);
  
  Parallel::CudaGlobal::clearCellListData
      <<<(num + DefaultNThreadPerBlock -1) / DefaultNThreadPerBlock,
      DefaultNThreadPerBlock>>> (
	  deviceList,
	  num,
	  numAtomInCell);
  checkCUDAError ("DeviceCellListedMDData::clearData, clearCellListData");
  hipFree (deviceList);
  checkCUDAError ("DeviceCellListedMDData::clearData, free deviceList");  
}


void __global__
Parallel::CudaGlobal::
clearCellListData (const IndexType * deviceList,
		   IndexType num,
		   IndexType * numAtomInCell)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;

  if (ii < num){
    numAtomInCell[deviceList[ii]] = 0;
  }
}



__global__ void Parallel::CudaGlobal::
normalizeSystem_CellListed (RectangularBox box,
			    const IndexType * numAtomInCell,
			    CoordType * coord,
			    CoordNoiType * coordNoi)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
 
  if (tid < numAtomInCell[bid]) {
    RectangularBoxGeometry::moveParticleToBox_1image (
	box.size.x, &(coord[ii].x), &(coordNoi[ii].x));
    RectangularBoxGeometry::moveParticleToBox_1image (
	box.size.y, &(coord[ii].y), &(coordNoi[ii].y));
    RectangularBoxGeometry::moveParticleToBox_1image (
	box.size.z, &(coord[ii].z), &(coordNoi[ii].z));
  }
}


void Parallel::DeviceCellListedMDData::
applyPeriodicBondaryCondition ()
{
  IndexType totalNumCell = numCell.x * numCell.y * numCell.z;
  IndexType numThreadsInCell = Parallel::Interface::numThreadsInCell ();
  
  Parallel::CudaGlobal::normalizeSystem_CellListed
      <<<totalNumCell, numThreadsInCell>>> (
	  globalBox,
	  numAtomInCell,
	  coord,
	  coordNoi);
  checkCUDAError ("DeviceCellListedMDData::applyPeriodicBondaryCondition");
}

void Parallel::DeviceCellRelation::
easyMalloc (const IndexType & totalNumCell,
	    const IndexType & MaxNeiPerCell)
{
  if (malloced){
    clear ();
  }
  hipMalloc ((void**)&numNeighbor, totalNumCell * sizeof(IndexType));
  hipMalloc ((void**)&neighborCellIndex,
	      totalNumCell * MaxNeiPerCell * sizeof (IndexType));
  hipMalloc ((void**)&neighborShift,
	      totalNumCell * MaxNeiPerCell * sizeof (CoordType));
  checkCUDAError ("DeviceCellRelation::easyMalloc");
  malloced = true;
}

void Parallel::DeviceCellRelation::
clear ()
{
  hipFree (numNeighbor);
  hipFree (neighborCellIndex);
  hipFree (neighborShift);
  malloced = false;
}

Parallel::DeviceCellRelation::
~DeviceCellRelation ()
{
  clear();
}

void Parallel::DeviceCellRelation::
build (DeviceCellListedMDData & list)
{
  ptr_list = &list;

  IntVectorType numCell = list.getNumCell ();
  IndexType totalNumCell = numCell.x * numCell.y * numCell.z;
  MaxNeiPerCell = 2 * list.getDevideLevel() + 1;
  MaxNeiPerCell = MaxNeiPerCell * MaxNeiPerCell * MaxNeiPerCell;
  int Nx, Ny, Nz;
  Parallel::Interface::numProcDim (Nx, Ny, Nz);
  
  easyMalloc (totalNumCell, MaxNeiPerCell);
  // setValue <<<
  //     (totalNumCell + DefaultNThreadPerBlock - 1) / DefaultNThreadPerBlock,
  //     DefaultNThreadPerBlock >>> (
  // 	  numNeighbor,
  // 	  totalNumCell,
  // 	  0);
  int rankx, ranky, rankz;
  Parallel::Interface::rankToCartCoord (Parallel::Interface::myRank(), rankx, ranky, rankz);
  
  Parallel::CudaGlobal::buildCellNeighborhood
      <<<toGridDim(totalNumCell), 1>>> (
	  numCell,
	  list.getDevideLevel(),
	  list.getRlist(),
	  list.getGlobalBoxSize(),
	  rankx, ranky, rankz,
	  Nx, Ny, Nz,
	  numNeighbor,
	  neighborCellIndex,
	  neighborShift,
	  MaxNeiPerCell);
  checkCUDAError ("DeviceCellRelation::build, buildCellNeighborhood");
}


__global__ void Parallel::CudaGlobal::
buildCellNeighborhood (const IntVectorType numCell,
		       const IndexType devideLevel,
		       const ScalorType rlist,
		       const HostVectorType boxSize,
		       IndexType * numNeighbor,
		       IndexType * neighborCellIndex,
		       const IndexType stride)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  __shared__ bool stop;

  numNeighbor[bid] = 0;
  int centerx, centery, centerz;
  bool oneCellX(false), oneCellY(false), oneCellZ(false);
  if (numCell.x == 3) oneCellX = true;
  if (numCell.y == 3) oneCellY = true;
  if (numCell.z == 3) oneCellZ = true;

  if (tid == 0){
    stop = false;
    Parallel::CudaDevice::D1toD3 (numCell, int(bid), centerx, centery, centerz);
    if (oneCellX){
      if (centerx == 0 || centerx == 2){
	stop = true;
      }
    }
    else {
      if (centerx < devideLevel || centerx >= numCell.x - devideLevel){
	stop = true;
      }
    }
    if (oneCellY){
      if (centery == 0 || centery == 2){
	stop = true;
      }
    }
    else {
      if (centery < devideLevel || centery >= numCell.y - devideLevel){
	stop = true;
      }
    }
    if (oneCellZ){
      if (centerz == 0 || centerz == 2){
	stop = true;
      }
    }
    else {
      if (centerz < devideLevel || centerz >= numCell.z - devideLevel){
	stop = true;
      }
    }
  }
  __syncthreads();
  if (stop) return;

  if (tid == 0){
    int lowerX (-devideLevel);
    int lowerY (-devideLevel);
    int lowerZ (-devideLevel);
    if (oneCellX) lowerX = -1;
    if (oneCellY) lowerY = -1;
    if (oneCellZ) lowerZ = -1;
    int upperX (devideLevel+1);
    int upperY (devideLevel+1);
    int upperZ (devideLevel+1);
    if (oneCellX) upperX = 2;
    if (oneCellY) upperY = 2;
    if (oneCellZ) upperZ = 2;
    ScalorType scalorx, scalory, scalorz;
    oneCellX ? scalorx = boxSize.x :
	scalorx = boxSize.x / ScalorType(numCell.x - (devideLevel << 1));
    oneCellY ? scalory = boxSize.y :
	scalory = boxSize.y / ScalorType(numCell.y - (devideLevel << 1));
    oneCellZ ? scalorz = boxSize.z :
	scalorz = boxSize.z / ScalorType(numCell.z - (devideLevel << 1));
    
    ScalorType rlist2 = rlist * rlist;
    for (int ix = lowerX; ix < upperX; ++ix){
      for (int iy = lowerY; iy < upperY; ++iy){
	for (int iz = lowerZ; iz < upperZ; ++iz){
	  int myx = ix + int(centerx);
	  int myy = iy + int(centery);
	  int myz = iz + int(centerz);
	  ScalorType min = 1e9;
#pragma unroll 27
	  for (int dx = -1; dx <= 1; ++dx){
	    for (int dy = -1; dy <= 1; ++dy){
	      for (int dz = -1; dz <= 1; ++dz){
		ScalorType diffx ((-centerx + myx + dx) * scalorx);
		ScalorType diffy ((-centery + myy + dy) * scalory);
		ScalorType diffz ((-centerz + myz + dz) * scalorz);
		// shortestImage (box, &diffx, &diffy, &diffz);
		ScalorType diff2 (diffx * diffx + diffy * diffy + diffz * diffz);
		if (diff2 < min){
		  min = diff2;
		}
	      }
	    }
	  }
	  if (min < rlist2){
	    IndexType tmp = Parallel::CudaDevice::D3toD1 (numCell, myx, myy, myz);
	    neighborCellIndex[(numNeighbor[bid]++) + bid * stride] = tmp;
	  }
	}
      }
    }
  }
}


__global__ void Parallel::CudaGlobal::
buildCellNeighborhood (const IntVectorType numCell,
		       const IndexType devideLevel,
		       const ScalorType rlist,
		       const HostVectorType globalBoxSize,
		       const int rankx,
		       const int ranky,
		       const int rankz,
		       const int nProcDimx,
		       const int nProcDimy,
		       const int nProcDimz,
		       IndexType * numNeighbor,
		       IndexType * neighborCellIndex,
		       CoordType * neighborShift,
		       const IndexType stride)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  __shared__ bool stop;

  numNeighbor[bid] = 0;
  int centerx, centery, centerz;
  bool oneCellX(false), oneCellY(false), oneCellZ(false);
  if (numCell.x == 3) oneCellX = true;
  if (numCell.y == 3) oneCellY = true;
  if (numCell.z == 3) oneCellZ = true;

  HostVectorType boxSize;
  if (tid == 0){
    boxSize.x = globalBoxSize.x / nProcDimx;
    boxSize.y = globalBoxSize.y / nProcDimy;
    boxSize.z = globalBoxSize.z / nProcDimz;
    stop = false;
    Parallel::CudaDevice::D1toD3 (numCell, int(bid), centerx, centery, centerz);
    if (oneCellX){
      if (centerx == 0 || centerx == 2){
	stop = true;
      }
    }
    else {
      if (centerx < devideLevel || centerx >= numCell.x - devideLevel){
	stop = true;
      }
    }
    if (oneCellY){
      if (centery == 0 || centery == 2){
	stop = true;
      }
    }
    else {
      if (centery < devideLevel || centery >= numCell.y - devideLevel){
	stop = true;
      }
    }
    if (oneCellZ){
      if (centerz == 0 || centerz == 2){
	stop = true;
      }
    }
    else {
      if (centerz < devideLevel || centerz >= numCell.z - devideLevel){
	stop = true;
      }
    }
  }
  __syncthreads();
  if (stop) return;

  if (tid == 0){
    int lowerX (-devideLevel);
    int lowerY (-devideLevel);
    int lowerZ (-devideLevel);
    if (oneCellX) lowerX = -1;
    if (oneCellY) lowerY = -1;
    if (oneCellZ) lowerZ = -1;
    int upperX (devideLevel+1);
    int upperY (devideLevel+1);
    int upperZ (devideLevel+1);
    if (oneCellX) upperX = 2;
    if (oneCellY) upperY = 2;
    if (oneCellZ) upperZ = 2;
    ScalorType scalorx, scalory, scalorz;
    oneCellX ? scalorx = boxSize.x :
	scalorx = boxSize.x / ScalorType(numCell.x - (devideLevel << 1));
    oneCellY ? scalory = boxSize.y :
	scalory = boxSize.y / ScalorType(numCell.y - (devideLevel << 1));
    oneCellZ ? scalorz = boxSize.z :
	scalorz = boxSize.z / ScalorType(numCell.z - (devideLevel << 1));
    
    ScalorType rlist2 = rlist * rlist;
    CoordType myshift ;
    myshift.x = myshift.y = myshift.z = 0;
    for (int ix = lowerX; ix < upperX; ++ix){
      for (int iy = lowerY; iy < upperY; ++iy){
	for (int iz = lowerZ; iz < upperZ; ++iz){
	  int myx = ix + int(centerx);
	  int myy = iy + int(centery);
	  int myz = iz + int(centerz);
	  myshift.x = myshift.y = myshift.z = 0.f;
	  if (rankx == 0 && myx < devideLevel) {
	    myshift.x = - globalBoxSize.x;
	  }
	  else if (rankx == nProcDimx - 1 && myx >= int((numCell.x-1) * devideLevel)){
	    myshift.x = globalBoxSize.x;
	  }
	  if (ranky == 0 && myy < devideLevel) {
	    myshift.y = - globalBoxSize.y;
	  }
	  else if (ranky == nProcDimy - 1 && myy >= int((numCell.y-1) * devideLevel)){
	    myshift.y = globalBoxSize.y;
	  }
	  if (rankz == 0 && myz < devideLevel) {
	    myshift.z = - globalBoxSize.z;
	  }
	  else if (rankz == nProcDimz - 1 && myz >= int((numCell.z-1) * devideLevel)){
	    myshift.z = globalBoxSize.z;
	  }
	  ScalorType min = 1e9;
#pragma unroll 27
	  for (int dx = -1; dx <= 1; ++dx){
	    for (int dy = -1; dy <= 1; ++dy){
	      for (int dz = -1; dz <= 1; ++dz){
		ScalorType diffx ((-centerx + myx + dx) * scalorx);
		ScalorType diffy ((-centery + myy + dy) * scalory);
		ScalorType diffz ((-centerz + myz + dz) * scalorz);
		// shortestImage (box, &diffx, &diffy, &diffz);
		ScalorType diff2 (diffx * diffx + diffy * diffy + diffz * diffz);
		if (diff2 < min){
		  min = diff2;
		}
	      }
	    }
	  }
	  if (min < rlist2){
	    IndexType tmp = Parallel::CudaDevice::D3toD1 (numCell, myx, myy, myz);
	    neighborShift    [(numNeighbor[bid]  ) + bid * stride] = myshift;
	    neighborCellIndex[(numNeighbor[bid]++) + bid * stride] = tmp;
	  }
	}
      }
    }
  }
}

  

  






