#include "SPMERec.h"

void SPMERecIk::
freeAll ()
{
  if (malloced){
    hipFree (vecbx);
    hipFree (vecby);
    hipFree (vecbz);
    hipFree (Q);
    hipFree (psiF);
    hipFree (phiF0);
    hipFree (phiF1);
    hipFree (phiF2);
    hipFree (QF);
    hipFree (QFxPsiF);
    hipFree (QFxPhiF0);
    hipFree (QFxPhiF1);
    hipFree (QFxPhiF2);
    hipFree (QConvPsi);
    hipFree (QConvPhi0);
    hipFree (QConvPhi1);
    hipFree (QConvPhi2);
    hipfftDestroy (planForward);
    hipfftDestroy (planBackward);
    malloced = false;
  }
}

SPMERecIk::
SPMERecIk()
    : malloced (false)
{
}

SPMERecIk::
~SPMERecIk ()
{
  freeAll();
}

void SPMERecIk::
calB ()
{
  cal_Bx
      <<<((K.x + meshBlockDim.x - 1) / meshBlockDim.x), meshBlockDim>>> (
	  K,
	  order,
	  vecbx);
  checkCUDAError ("SPMERecIk::calB x");
  cal_By
      <<<((K.y + meshBlockDim.x - 1) / meshBlockDim.x), meshBlockDim>>> (
	  K,
	  order,
	  vecby);
  checkCUDAError ("SPMERecIk::calB y");
  cal_Bz
      <<<((K.z + meshBlockDim.x - 1) / meshBlockDim.x), meshBlockDim>>> (
	  K,
	  order,
	  vecbz);
  checkCUDAError ("SPMERecIk::calB z");
}


void SPMERecIk::
buildNeighborList (const MDSystem & sys)
{
}


void  SPMERecIk::
reinit (const MatrixType & vecA_,
	const IntVectorType & K_,
	const IndexType & order_,
	const ScalorType & beta_,
	const IndexType & natom,
	const IndexType & meshNThread,
	const IndexType & atomNThread)
{
  freeAll();
  vecA = vecA_;
  K = K_;
  order = order_;
  beta = beta_;
  calV();
  calAStar();

  IndexType nele = K.x * K.y * K.z;
  IntVectorType N(K);
  N.z = (N.z >> 1) + 1;
  IndexType nelehalf = N.x * N.y * N.z;

  IndexType nob;
  meshBlockDim.x = meshNThread;
  nob = (nele  + meshBlockDim.x - 1) / meshBlockDim.x;
  meshGridDim = toGridDim (nob);
  nob = (nelehalf  + meshBlockDim.x - 1) / meshBlockDim.x;
  meshGridDim_half = toGridDim (nob);
  atomBlockDim.x = atomNThread;
  nob = (natom + atomBlockDim.x - 1) / atomBlockDim.x;
  atomGridDim = toGridDim (nob);

  hipMalloc ((void**)&vecbx, sizeof(ScalorType) * K.x);
  hipMalloc ((void**)&vecby, sizeof(ScalorType) * K.y);
  hipMalloc ((void**)&vecbz, sizeof(ScalorType) * K.z);
  calB ();
  checkCUDAError ("SPMERecIk::reinit malloc");
  hipMalloc ((void**)&Q, sizeof(hipfftReal) * nele);
  checkCUDAError ("SPMERecIk::reinit malloc");
  hipMalloc ((void**)&psiF,  sizeof(hipfftComplex) * nelehalf);
  hipMalloc ((void**)&phiF0, sizeof(hipfftComplex) * nelehalf);
  hipMalloc ((void**)&phiF1, sizeof(hipfftComplex) * nelehalf);
  hipMalloc ((void**)&phiF2, sizeof(hipfftComplex) * nelehalf);
  checkCUDAError ("SPMERecIk::reinit malloc");
  hipMalloc ((void**)&QF,  sizeof(hipfftComplex) * nelehalf);
  hipMalloc ((void**)&QFxPsiF,  sizeof(hipfftComplex) * nelehalf);
  hipMalloc ((void**)&QFxPhiF0, sizeof(hipfftComplex) * nelehalf);
  hipMalloc ((void**)&QFxPhiF1, sizeof(hipfftComplex) * nelehalf);
  hipMalloc ((void**)&QFxPhiF2, sizeof(hipfftComplex) * nelehalf);
  checkCUDAError ("SPMERecIk::reinit malloc");
  hipMalloc ((void**)&QConvPsi,  sizeof(hipfftReal) * nele);
  hipMalloc ((void**)&QConvPhi0, sizeof(hipfftReal) * nele);
  hipMalloc ((void**)&QConvPhi1, sizeof(hipfftReal) * nele);
  hipMalloc ((void**)&QConvPhi2, sizeof(hipfftReal) * nele);
  checkCUDAError ("SPMERecIk::reinit malloc");

  hipfftPlan3d (&planForward,  K.x, K.y, K.z, HIPFFT_R2C);
  hipfftPlan3d (&planBackward, K.x, K.y, K.z, HIPFFT_C2R);

  malloced = true;

  cal_PsiFPhiF
      <<<meshGridDim_half, meshBlockDim>>> (
	  K,
	  vecAStar,
	  beta,
	  volume,
	  vecbx,
	  vecby,
	  vecbz,
	  psiF,
	  phiF0,
	  phiF1,
	  phiF2);
  checkCUDAError ("SPMERecIk::reinit cal_PsiFPhiF");

  nlist_stride = nele;
  nlist_length = order * order * order * 2;
  hipMalloc ((void**)&nlist_n, sizeof(IndexType) * nlist_stride);
  hipMalloc ((void**)&nlist_list, sizeof(IndexType) * nlist_stride * nlist_length);
  checkCUDAError ("SPMERecIk::reinit malloc nlist");

  sum_e.  reinit (nele, NThreadForSum);
  sum_vxx.reinit (nele, NThreadForSum);
  sum_vyy.reinit (nele, NThreadForSum);
  sum_vzz.reinit (nele, NThreadForSum);
  checkCUDAError ("EwaldSumRec::reinit reinit sums");
}

void SPMERecIk::
applyInteraction (MDSystem & sys,
		  MDStatistic * pst,
		  MDTimer * timer)
{
  calQ (sys);
  
  hipfftExecR2C (planForward, Q, QF);
  checkCUDAError ("SPMERecIk::applyInteraction Q->QF");
  
  IntVectorType N(K);
  N.z = (N.z >> 1) + 1;
  IndexType nelehalf = N.x * N.y * N.z;
  ScalorType sizei = 1./(K.x*K.y*K.z);
  timeQFPhiF
      <<<meshGridDim_half, meshBlockDim>>> (
	  QF,
	  phiF0,
	  phiF1,
	  phiF2,
	  QFxPhiF0,
	  QFxPhiF1,
	  QFxPhiF2,
	  nelehalf,
	  sizei);
  checkCUDAError ("SPMERecIk::applyInteraction timeQFPhiF");

  hipfftExecC2R (planBackward, QFxPhiF0, QConvPhi0);
  hipfftExecC2R (planBackward, QFxPhiF1, QConvPhi1);
  hipfftExecC2R (planBackward, QFxPhiF2, QConvPhi2);
  checkCUDAError ("SPMERecIk::applyInteraction QFxPhiF->QConvPhi");

  calForce
      <<<atomGridDim, atomBlockDim>>> (
	  K,
	  vecAStar,
	  order,
	  sys.ddata.coord,
	  sys.ddata.charge,
	  sys.ddata.numAtom,
	  QConvPhi0,
	  QConvPhi1,
	  QConvPhi2,
	  sys.ddata.forcx,
	  sys.ddata.forcy,
	  sys.ddata.forcz,
	  err.ptr_de);
  checkCUDAError ("SPMERecIk::applyInteraction calForce");
  err.check ("SPMERecIk::applyInteraction calForce");

  if (pst != NULL){
    timeQFPsiF
	<<<meshGridDim_half, meshBlockDim>>> (
	    QF,
	    psiF,
	    QFxPsiF,
	    nelehalf,
	    sizei);
    checkCUDAError ("SPMERecIk::applyInteraction time QF PhiF");
    hipfftExecC2R (planBackward, QFxPsiF, QConvPsi);
    checkCUDAError ("SPMERecIk::applyInteraction QFxPsiF->QConvPsi");
    calEnergy
	<<<meshGridDim, meshBlockDim>>> (
	    Q,
	    QConvPsi,
	    sum_e.buff,
	    K.x * K.y * K.z);
    checkCUDAError ("SPMERecIk::applyInteraction cal energy");
    sum_e.  sumBuffAdd(pst->ddata, mdStatisticNonBondedPotential);
    checkCUDAError ("SPMERecIk::applyInteraction sum energy");
  }
}



void SPMERecIk::
calQ (const MDSystem & sys)
{
//
// fast algorithm !!!!
//
  initMeshNeighborList
      <<<meshGridDim, meshBlockDim>>>(
	  K,
	  vecAStar,
	  nlist_n,
	  nlist_list,
	  nlist_stride,
	  nlist_length);
  buildMeshNeighborList
      <<<atomGridDim, atomBlockDim>>> (
	  K,
	  vecAStar,
	  order,
	  sys.ddata.coord,
	  sys.ddata.numAtom,
	  nlist_n,
	  nlist_list,
	  nlist_stride,
	  nlist_length,
	  err.ptr_de);
  checkCUDAError ("SPMERecIk::calQ buildNeighborList");
  err.check ("SPMERecIk::calQ buildNeighborList");
  cal_Q
      <<<meshGridDim, meshBlockDim>>> (
	  K,
	  vecAStar,
	  order,
	  nlist_n,
	  nlist_list,
	  nlist_stride,
	  Q);
  checkCUDAError ("SPMERecIk::calQ cal_Q");
  // FILE * fp = fopen ("tmpQ.out", "w");
  // for (unsigned i = 0; i < K.x * K.y * K.z; ++i){
  //   fprintf (fp, "%.12e\n", Q[i]);
  // }
  // fclose (fp);
}
  
void SPMERecIk::
calV()
{
  volume =
      vecA.xx * (vecA.yy*vecA.zz - vecA.zy*vecA.yz) - 
      vecA.xy * (vecA.yx*vecA.zz - vecA.zx*vecA.yz) +
      vecA.xz * (vecA.yx*vecA.zy - vecA.zx*vecA.yy);
}
  
void SPMERecIk::
calAStar ()
{
  ScalorType volumei = ScalorType(1.) / volume;
  vecAStar.xx = ( vecA.yy*vecA.zz - vecA.zy*vecA.yz) * volumei;
  vecAStar.yy = ( vecA.xx*vecA.zz - vecA.zx*vecA.xz) * volumei;
  vecAStar.zz = ( vecA.xx*vecA.yy - vecA.yx*vecA.xy) * volumei;
  vecAStar.yx = (-vecA.yx*vecA.zz + vecA.zx*vecA.yz) * volumei;
  vecAStar.zx = ( vecA.yx*vecA.zy - vecA.zx*vecA.yy) * volumei;
  vecAStar.xy = (-vecA.xy*vecA.zz + vecA.zy*vecA.xz) * volumei;
  vecAStar.zy = (-vecA.xx*vecA.zy + vecA.zx*vecA.xy) * volumei;
  vecAStar.xz = ( vecA.xy*vecA.yz - vecA.yy*vecA.xz) * volumei;
  vecAStar.yz = (-vecA.xx*vecA.yz + vecA.yx*vecA.xz) * volumei;
}

