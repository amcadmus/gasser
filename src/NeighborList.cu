#include "hip/hip_runtime.h"
#define DEVICE_CODE

#include "NeighborList.h"
#include<stdio.h>
#include "Auxiliary.h"
#include "NonBondedInteraction.h"

// extern texture<float, 1, hipReadModeElementType> texRef;

__global__ void
prepare_naivelyBuildDeviceCellList (DeviceCellList clist)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  clist.data[bid * clist.stride + threadIdx.x] = MaxIndexValue;
  if (threadIdx.x == 0) clist.numbers[bid] = 0;
}



//////////////////////////////////////////////////
// coord in one vec
//////////////////////////////////////////////////

__global__ void
naivelyBuildDeviceCellList (const IndexType		numAtom,
			    const CoordType *		coord,
			    const RectangularBox	box,
			    DeviceCellList		clist,
			    mdError_t *			ptr_de,
			    IndexType *			erridx,
			    ScalorType *		errsrc)
{
  // normalizeSystem (box, numAtom, coordx, coordy, coordz, coordNoix, coordNoiy, coordNoiz);
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  
  // calculate target cell id
  IndexType cellid ;
  if (ii < numAtom){
    IndexType targetCelli, targetCellj, targetCellk;
    targetCelli = IndexType(coord[ii].x * box.sizei.x * ScalorType (clist.NCell.x));
    targetCellj = IndexType(coord[ii].y * box.sizei.y * ScalorType (clist.NCell.y));
    targetCellk = IndexType(coord[ii].z * box.sizei.z * ScalorType (clist.NCell.z));
    cellid = D3toD1 (clist.NCell, targetCelli, targetCellj, targetCellk);
    if (ptr_de != NULL && 
	(targetCelli >= clist.NCell.x || 
	 targetCellj >= clist.NCell.y || 
	 targetCellk >= clist.NCell.z)){
      *ptr_de = mdErrorOverFlowCellIdx;
      if (targetCelli >= IndexType(clist.NCell.x)){
	*erridx = targetCelli;
	*errsrc = coord[ii].x;
	return;
      }
      if (targetCellj >= IndexType(clist.NCell.y)){
	*erridx = targetCellj;
	*errsrc = coord[ii].y;
	return;
      }
      if (targetCellk >= IndexType(clist.NCell.z)){
	*erridx = targetCellk;
	*errsrc = coord[ii].z;
	return;
      }
    }
  }
  else {
    cellid = MaxIndexValue;
  }
  
  // write indexes to clist 
  if (cellid != MaxIndexValue){
    IndexType pid = atomicInc(&(clist.numbers[cellid]), blockDim.x);
    clist.data[cellid * clist.stride + pid]
	= ii;
    if (pid == blockDim.x && *ptr_de != NULL){
      *ptr_de = mdErrorShortCellList;
    }
  }
}


__global__ void
naivelyBuildDeviceCellList2 (const IndexType		numAtom,
			     CoordType *		coord,
			     IntScalorType *		coordNoix,
			     IntScalorType *		coordNoiy,
			     IntScalorType *		coordNoiz,
			     const RectangularBox	box,
			     DeviceCellList		clist,
			     mdError_t *		ptr_de,
			     IndexType *		erridx,
			     ScalorType *		errsrc)
{
  // normalizeSystem (box, numAtom, coordx, coordy, coordz, coordNoix, coordNoiy, coordNoiz);
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  
  // calculate target cell id
  extern __shared__ volatile IndexType sbuff[];
  volatile IndexType * targetCellid = (volatile IndexType *) sbuff;
  
  if (ii < numAtom){
    IndexType targetCelli, targetCellj, targetCellk;
    targetCelli = IndexType(coord[ii].x * box.sizei.x * ScalorType (clist.NCell.x));
    targetCellj = IndexType(coord[ii].y * box.sizei.y * ScalorType (clist.NCell.y));
    targetCellk = IndexType(coord[ii].z * box.sizei.z * ScalorType (clist.NCell.z));
    if (targetCelli == clist.NCell.x){
      targetCelli -= clist.NCell.x;
      coord[ii].x -= box.size.x;
      coordNoix[ii] ++;
    }
    if (targetCellj == clist.NCell.y){
      targetCellj -= clist.NCell.y;
      coord[ii].y -= box.size.y;
      coordNoiy[ii] ++;
    }
    if (targetCellk == clist.NCell.z){
      targetCellk -= clist.NCell.z;
      coord[ii].z -= box.size.z;
      coordNoiz[ii] ++;
    }
    targetCellid[tid] = D3toD1 (clist.NCell, targetCelli, targetCellj, targetCellk);
    if (ptr_de != NULL && 
	(targetCelli >= clist.NCell.x || 
	 targetCellj >= clist.NCell.y || 
	 targetCellk >= clist.NCell.z)){
      *ptr_de = mdErrorOverFlowCellIdx;
      if (targetCelli >= IndexType(clist.NCell.x)){
	*erridx = targetCelli;
	*errsrc = coord[ii].x;
	return;
      }
      if (targetCellj >= IndexType(clist.NCell.y)){
	*erridx = targetCellj;
	*errsrc = coord[ii].y;
	return;
      }
      if (targetCellk >= IndexType(clist.NCell.z)){
	*erridx = targetCellk;
	*errsrc = coord[ii].z;
	return;
      }      
    }
  }
  else {
    targetCellid[tid] = MaxIndexValue;
  }
  __syncthreads();
  // write indexes to clist only the first thread in the block to that
  if (threadIdx.x == 0){
    for (IndexType i = 0; i < blockDim.x; ++i){
      IndexType cellid = targetCellid[i];
      if (cellid != MaxIndexValue){
	IndexType pid = atomicInc(&clist.numbers[cellid], clist.stride);
  	clist.data[cellid * clist.stride + pid] = i + bid * blockDim.x;
	if (pid == clist.stride && ptr_de != NULL){
	  *ptr_de = mdErrorShortCellList;
	}
      }
      else
  	break;
    }
  }
}


__global__ void
buildCellNeighborhood (DeviceCellList clist,
		       const IndexType divide,
		       const HostVectorType boxSize)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;

  bool oneCellX(false), oneCellY(false), oneCellZ(false);
  if (clist.NCell.x == 1) oneCellX = true;
  if (clist.NCell.y == 1) oneCellY = true;
  if (clist.NCell.z == 1) oneCellZ = true;

  IndexType upperX, upperY, upperZ;
  oneCellX ? upperX = 1 : upperX = 2*divide+1;
  oneCellY ? upperY = 1 : upperY = 2*divide+1;
  oneCellZ ? upperZ = 1 : upperZ = 2*divide+1;
  
  
  if (tid == 0) {
    ScalorType rlist2 = clist.rlist * clist.rlist;
    clist.numNeighborCell[bid] = 0;
    int centerx, centery, centerz;
    D1toD3 (clist.NCell, int(bid), centerx, centery, centerz);
    for (int ix = 0; ix < upperX; ++ix){
      for (int iy = 0; iy < upperY; ++iy){
	for (int iz = 0; iz < upperZ; ++iz){
	  int myx = ix - clist.divide + int(centerx) ;
	  int myy = iy - clist.divide + int(centery) ;
	  int myz = iz - clist.divide + int(centerz) ;
	  ScalorType scalorx = boxSize.x / clist.NCell.x ;
	  ScalorType scalory = boxSize.y / clist.NCell.y ;
	  ScalorType scalorz = boxSize.z / clist.NCell.z ;
      
	  ScalorType min = 1e9;
#pragma unroll 27
	  for (int dx = -1; dx <= 1; ++dx){
	    for (int dy = -1; dy <= 1; ++dy){
	      for (int dz = -1; dz <= 1; ++dz){
		ScalorType diffx ((-centerx + myx + dx) * scalorx);
		ScalorType diffy ((-centery + myy + dy) * scalory);
		ScalorType diffz ((-centerz + myz + dz) * scalorz);
		// shortestImage (box, &diffx, &diffy, &diffz);
		ScalorType diff2 (diffx * diffx + diffy * diffy + diffz * diffz);
		if (diff2 < min){
		  min = diff2;
		}
	      }
	    }
	  }
	  CoordNoiType shift;
	  shift.x = shift.y = shift.z = 0;
	  if (min < rlist2){
	    if (myx < 0) {
	      myx += clist.NCell.x;
	      shift.x += 1;
	    }
	    else if (myx >= clist.NCell.x){
	      myx -= clist.NCell.x;
	      shift.x -= 1;
	    }
	    if (myy < 0) {
	      myy += clist.NCell.y;
	      shift.y += 1;
	    }
	    else if (myy >= clist.NCell.y){
	      myy -= clist.NCell.y;
	      shift.y -= 1;
	    }
	    if (myz < 0) {
	      myz += clist.NCell.z;
	      shift.z += 1;
	    }
	    else if (myz >= clist.NCell.z){
	      myz -= clist.NCell.z;
	      shift.z -= 1;
	    }

	    pushNeighborCell (clist,
			      bid,
			      D3toD1 (clist.NCell, myx, myy, myz),
			      shift);
	  }
	}
      }
    }
  }
}

  
  // if (tid == 0) clist.numNeighborCell[bid] = 0;
  // __syncthreads();
  
  // int centerx, centery, centerz;
  // D1toD3 (clist.NCell, int(bid), centerx, centery, centerz);
  // IntVectorType thisCubic;
  // thisCubic.z = thisCubic.y = thisCubic.x = 2 * clist.divide + 1;
  // int myx, myy, myz;
  // D1toD3 (thisCubic, int(tid), myx, myy, myz);
  // myx += int(centerx) - clist.divide;
  // myy += int(centery) - clist.divide ;
  // myz += int(centerz) - clist.divide ;
  // ScalorType scalorx = boxSize.x / clist.NCell.x / divide;
  // ScalorType scalory = boxSize.y / clist.NCell.y / divide;
  // ScalorType scalorz = boxSize.z / clist.NCell.z / divide;

  // ScalorType rlist2 = clist.rlist * clist.rlist;
  // ScalorType min = 1e9;
  // for (int dx = -1; dx <= 1; ++dx){
  //   for (int dy = -1; dy <= 1; ++dy){
  //     for (int dz = -1; dz <= 1; ++dz){
  // 	ScalorType diffx ((centerx - myx + dx) * scalorx);
  // 	ScalorType diffy ((centery - myy + dy) * scalory);
  // 	ScalorType diffz ((centerz - myz + dz) * scalorz);
  // 	ScalorType diff2 (diffx * diffx + diffy * diffy + diffz * diffz);
  // 	if (diff2 < min){
  // 	  min = diff2;
  // 	}
  //     }
  //   }
  // }
  // CoordType shift;
  // shift.x = shift.y = shift.z = 0.f;
  // if (min < rlist2){
  //   IndexType index = atomicInc (&clist.numNeighborCell[bid], blockDim.x);
  //   if (myx < 0) {
  //     myx += clist.NCell.x;
  //     shift.x -= boxSize.x;
  //   }
  //   else if (myx > clist.NCell.x){
  //     myx -= clist.NCell.x;
  //     shift.x += boxSize.x;
  //   }
  //   if (myy < 0) {
  //     myy += clist.NCell.y;
  //     shift.y -= boxSize.y;
  //   }
  //   else if (myy > clist.NCell.y){
  //     myy -= clist.NCell.y;
  //     shift.y += boxSize.y;
  //   }
  //   if (myz < 0) {
  //     myz += clist.NCell.z;
  //     shift.z -= boxSize.z;
  //   }
  //   else if (myz > clist.NCell.z){
  //     myz -= clist.NCell.z;
  //     shift.z += boxSize.z;
  //   }

  //   clist.neighborCellIndex[index] = D3toD1 (clist.NCell, myx, myy, myz);
  //   clist.neighborCellShift[index] = shift;
  // }
// }  



__global__ void
buildDeviceNeighborList_AllPair  (const IndexType		numAtom,
				  const CoordType *		coord,
				  const TypeType *		type,
				  const RectangularBox		box,
				  DeviceNeighborList		nlist,
				  const IndexType *		nbForceTable,
				  const IndexType		NatomType,
				  const bool			sharednbForceTable,
				  mdError_t *			ptr_de)
{
  // RectangularBoxGeometry::normalizeSystem (box, &ddata);
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  
  IndexType Nneighbor = 0;
  IndexType numberAtom = numAtom;
  IndexType ii = tid + bid * blockDim.x;

  extern __shared__ volatile char pub_sbuff[];

  volatile IndexType * targetIndexes =
      (volatile IndexType *) pub_sbuff;
  volatile CoordType * target =
      (volatile CoordType *) &targetIndexes[roundUp4(blockDim.x)];
  volatile TypeType * targettype =
      (volatile TypeType *) &target[roundUp4(blockDim.x)];
  IndexType * nbForceTableBuff = NULL;

  IndexType nbForceTableLength = AtomNBForceTable::dCalDataLength(NatomType);
  if (sharednbForceTable){
    nbForceTableBuff = (IndexType *) &targettype[roundUp4(blockDim.x)];
    cpyGlobalDataToSharedBuff (nbForceTable, nbForceTableBuff, nbForceTableLength);
  }
  __syncthreads();
  
  CoordType ref;
  TypeType reftype;
  if (ii < numberAtom){
    ref = coord[ii];
    reftype = type[ii];
  }

  for (IndexType targetBlockId = 0;
       targetBlockId * blockDim.x < numberAtom; ++targetBlockId){
    IndexType jj = tid + targetBlockId * blockDim.x;
    __syncthreads();
    if (jj < numberAtom){
      target[tid].x = coord[jj].x;
      target[tid].y = coord[jj].y;
      target[tid].z = coord[jj].z;
      targettype[tid] = type[jj];
    }
    __syncthreads();
    if (ii < numberAtom){
      for (IndexType kk = 0; kk < blockDim.x; ++kk){
	if (kk + targetBlockId * blockDim.x >= numberAtom) break;
	ScalorType diffx = target[kk].x - ref.x;
	ScalorType diffy = target[kk].y - ref.y;
	ScalorType diffz = target[kk].z - ref.z;
	RectangularBoxGeometry::shortestImage (box, &diffx, &diffy, &diffz);
	if ((diffx*diffx+diffy*diffy+diffz*diffz) < nlist.rlist*nlist.rlist &&
	    kk + targetBlockId * blockDim.x != ii){
	  IndexType listIdx = Nneighbor * nlist.stride + ii;
	  nlist.data[listIdx] = kk + targetBlockId * blockDim.x;
	  if (sharednbForceTable){
	    nlist.forceIndex[listIdx] 
		= AtomNBForceTable::calForceIndex (
		    nbForceTableBuff, NatomType, reftype, targettype[kk]);
	  }
	  else {
	    nlist.forceIndex[listIdx] 
		= AtomNBForceTable::calForceIndex (
		    nbForceTable, NatomType, reftype, TypeType(targettype[kk]));
	  }	
	  // if (nlist.forceIndex[listIdx] == 0){
	  //   printf ("%d  %d  reftype:%d targettype:%d\n",
	  // 	    ii, kk, reftype, targettype[kk]);
	  // }
	  Nneighbor ++;
	}
      }
    }
  }
  if (ii < numberAtom){
    nlist.Nneighbor[ii] = Nneighbor;
    if (Nneighbor > nlist.listLength && ptr_de != NULL){
      *ptr_de = mdErrorShortNeighborList;
    }
  }
}


__global__ void
Reshuffle_backupDeviceNeighborList (const IndexType numAtom,
				    const IndexType * nlistData1,
				    const IndexType * nbForceIndex1,
				    const IndexType stride,
				    const IndexType * Nneighbor1,
				    IndexType * nlistData2,
				    IndexType * nbForceIndex2,
				    IndexType * Nneighbor2)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;

  // __shared__ volatile IndexType myNumbers [MaxThreadsPerBlock * 2];
  extern __shared__ volatile IndexType myNumbers[];
  
  IndexType N;
  if ((bid + 1) * blockDim.x < numAtom) N = blockDim.x;
  else if (bid * blockDim.x >= numAtom) N = 0;
  else N = numAtom - bid * blockDim.x;

  myNumbers[tid] = 0;
  myNumbers[tid + blockDim.x] = 0;
  if (ii < numAtom){
    Nneighbor2[ii] = myNumbers[tid] = Nneighbor1[ii];
  }
  __syncthreads();
  IndexType maxNum = maxVectorBlockBuffer (myNumbers, N);
  __syncthreads();

  for (IndexType jj = 0; jj < maxNum; ++jj){
    if (jj < myNumbers[tid] && ii < numAtom ){
      nlistData2   [jj * stride + ii] = nlistData1   [jj * stride + ii];
      nbForceIndex2[jj * stride + ii] = nbForceIndex1[jj * stride + ii];
    }
  }
}


__global__ void
Reshuffle_reshuffleDeviceCellList (IndexType * clistData,
				   const IndexType * idxTable)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;

  IndexType fromid = clistData[bid * blockDim.x + tid];
  if (fromid != MaxIndexValue){
    IndexType toid = idxTable[fromid];
    clistData[bid * blockDim.x + tid] = toid;
  }
}


__global__ void
Reshuffle_reshuffleDeviceNeighborList (const IndexType numAtom,
				       const IndexType * nlistData1,
				       const IndexType* nbForceIndex1,
				       const IndexType stride,
				       const IndexType * Nneighbor1,
				       const IndexType * idxTable,
				       IndexType * nlistData2,
				       IndexType * nbForceIndex2,
				       IndexType * Nneighbor2)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  // fromid is  ii

  extern __shared__ volatile IndexType myNumbers [];
  // __shared__ volatile IndexType myNumbers [MaxThreadsPerBlock * 2];

  IndexType toid;
  if (ii < numAtom)
    toid = idxTable[ii];
  IndexType myNum = 0;
  
  myNumbers[tid] = 0;
  myNumbers[tid + blockDim.x] = 0;
  if (ii < numAtom){
    myNum = myNumbers[tid] = Nneighbor1[ii];
  }
  __syncthreads();
  IndexType maxNum = maxVectorBlockBuffer (myNumbers, blockDim.x);
  __syncthreads();

  if (ii < numAtom){
    Nneighbor2[toid] = Nneighbor1[ii];
  }
  for (unsigned jj = 0; jj < maxNum; ++jj){
    if (jj < myNum && ii < numAtom){
      nlistData2[jj * stride + toid] = idxTable[nlistData1[jj * stride + ii]];
      nbForceIndex2[jj * stride + toid] = nbForceIndex1[jj * stride + ii];
    }
  }
}




