#include "MDTimer_interface.h"
#include "common.h"

MDTimer::MDTimer()
{
  for (IndexType i = 0; i < NumberOfMemberInMdTime; ++i){
    hipEventCreate (&start[i]);
    hipEventCreate (&stop [i]);
    timeRecord[i] = 0.f;
  }
}

MDTimer::~MDTimer()
{
  for (IndexType i = 0; i < NumberOfMemberInMdTime; ++i){
    hipEventDestroy (start[i]);
    hipEventDestroy (stop [i]);
  }
}

void MDTimer::printRecord (FILE * fp)
{
  fprintf(fp, "Normalize system:                    %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeNormalizeSys],
	  100 * timeRecord[mdTimeNormalizeSys] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Build cell list:                     %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeBuildCellList],
	  100 * timeRecord[mdTimeBuildCellList] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Build neighbor list:                 %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeBuildNeighborList],
	  100 * timeRecord[mdTimeBuildNeighborList] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Judge rebuild:                       %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeJudgeRebuild],
	  100 * timeRecord[mdTimeJudgeRebuild] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Reshuffle system:                    %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeReshuffleSystem],
	  100 * timeRecord[mdTimeReshuffleSystem] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Integrate:                           %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeIntegrator],
	  100 * timeRecord[mdTimeIntegrator] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Remove translational freedom:        %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeRemoveTransFreedom],
	  100 * timeRecord[mdTimeRemoveTransFreedom] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Cal. non-bonded interaction:         %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeNonBondedInteraction],
	  100 * timeRecord[mdTimeNonBondedInteraction] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Cal. non-bonded interaction(st):     %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeNBInterStatistic],
	  100 * timeRecord[mdTimeNBInterStatistic] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Cal. non-bonded twin range corr:     %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeNBInterTwinRange],
	  100 * timeRecord[mdTimeNBInterTwinRange] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Cal. bond interaction:               %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeBondedInteraction],
	  100 * timeRecord[mdTimeBondedInteraction] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Cal. bond interaction(st):           %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeBInterStatistic],
	  100 * timeRecord[mdTimeBInterStatistic] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Cal. angle interaction:              %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeAngleInteraction],
	  100 * timeRecord[mdTimeAngleInteraction] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Cal. angle interaction(st):          %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeAngleInterStatistic],
	  100 * timeRecord[mdTimeAngleInterStatistic] / timeRecord[mdTimeTotal]);
  fprintf(fp, "SPME cal Q: build mesh nlist:        %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeSPMERecMeshNeighborList],
	  100 * timeRecord[mdTimeSPMERecMeshNeighborList] / timeRecord[mdTimeTotal]);
  fprintf(fp, "SPME cal Q: from mesh nlist:         %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeSPMECalQFromNList],
	  100 * timeRecord[mdTimeSPMECalQFromNList] / timeRecord[mdTimeTotal]);
  fprintf(fp, "SPME cal Q:                          %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeSPMERecCalQ],
	  100 * timeRecord[mdTimeSPMERecCalQ] / timeRecord[mdTimeTotal]);
  fprintf(fp, "SPME FFT:                            %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeSPMERecFFT],
	  100 * timeRecord[mdTimeSPMERecFFT] / timeRecord[mdTimeTotal]);
  fprintf(fp, "SPME time matrix:                    %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeSPMERecTimeMatrix],
	  100 * timeRecord[mdTimeSPMERecTimeMatrix] / timeRecord[mdTimeTotal]);
  fprintf(fp, "SPME cal force:                      %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeSPMERecForce],
	  100 * timeRecord[mdTimeSPMERecForce] / timeRecord[mdTimeTotal]);
  fprintf(fp, "SPME cal energy:                     %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeSPMERecEnergy],
	  100 * timeRecord[mdTimeSPMERecEnergy] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Data transfer:                       %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeDataTransfer],
	  100 * timeRecord[mdTimeDataTransfer] / timeRecord[mdTimeTotal]);
  fprintf(fp, "Data IO:                             %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeDataIO],
	  100 * timeRecord[mdTimeDataIO] / timeRecord[mdTimeTotal]);
  TimeType otherTime = TimeType(0.);
  for (IndexType i = 1; i < NumberOfMemberInMdTime; ++i){
    otherTime += timeRecord[i];
  }
  otherTime = timeRecord[mdTimeTotal] - otherTime;
  fprintf(fp, "Others uncounted:                    %1.3e s   %3.1f %\n",
	  0.001 * otherTime,
	  100 * otherTime / timeRecord[mdTimeTotal]);
  fprintf(fp, "Total time:                          %1.3e s   %3.1f %\n",
	  0.001 * timeRecord[mdTimeTotal],
	  100 * timeRecord[mdTimeTotal] / timeRecord[mdTimeTotal]);
}

