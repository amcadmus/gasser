#include "hip/hip_runtime.h"
#define DEVICE_CODE

#include "Statistic_interface.h"


void MDStatistic::deviceCopy (const MDStatistic & st)
{
  hipMemcpy (ddata, st.ddata, sizeof(ScalorType) * NumberOfStatisticItems,
	      hipMemcpyDeviceToDevice);
  checkCUDAError ("Statistic::deviceCopy");
}

MDStatistic::
MDStatistic ()
    : hdata (NULL), dmalloced (false)
{
}

MDStatistic::
MDStatistic (const MDSystem & sys)
    : hdata (NULL), dmalloced (false)
{
  reinit(sys);
}

void MDStatistic::
clear ()
{
  freeAPointer ((void**)&hdata);
  if (dmalloced){
    hipFree (ddata);
    dmalloced = false;
  }
}


void MDStatistic::
reinit (const MDSystem & sys)
{
  clear ();
  
  // malloc and init system
  hdata = (ScalorType *) malloc (sizeof(ScalorType) * NumberOfStatisticItems);
  if (hdata == NULL){
    throw MDExcptFailedMallocOnHost ("MDStatistic::MDStatistic", "hdata",
				     sizeof(ScalorType) * NumberOfStatisticItems);
  }
  hipMalloc ((void**)&ddata, sizeof(ScalorType) * NumberOfStatisticItems);
  checkCUDAError("MDStatistic::MDStatistic allocate for ddata");
  dmalloced = true;

  for (IndexType i = 0; i < NumberOfStatisticItems; ++i){
    hdata[i] = 0.f;
  }
  hipMemcpy (ddata, hdata, sizeof(ScalorType) * NumberOfStatisticItems,
	      hipMemcpyHostToDevice);
  checkCUDAError("MDStatistic::MDStatistic cpy form host to device");
}

MDStatistic::
~MDStatistic ()
{
  clear ();
}

__global__ void clearStatisticData (ScalorType *ddata)
{
  if (threadIdx.x < NumberOfStatisticItems){
    ddata[threadIdx.x] = 0.f;
  }
}

__global__ void addStatisticData (ScalorType * ddata, const ScalorType * cddata)
{
  if (threadIdx.x < NumberOfStatisticItems){
    ddata[threadIdx.x] += cddata[threadIdx.x];
  }
}

void MDStatistic::
clearDevice ()
{
  clearStatisticData <<<1, NumberOfStatisticItems>>> (ddata);
  checkCUDAError("Statistic::clearDevice");
}

void MDStatistic::
updateHost () const
{
  hipMemcpy (hdata, ddata, sizeof(ScalorType) * NumberOfStatisticItems, 
	      hipMemcpyDeviceToHost);
  checkCUDAError("Statistic::updateHost");
}

void MDStatistic::
deviceAdd (const MDStatistic & st)
{
  addStatisticData <<<1, NumberOfStatisticItems>>> (ddata, st.ddata);
}


void MDStatistic::
copy (const MDStatistic & st,
      const IndexType num,
      const mdStatisticItem_t items[NumberOfStatisticItems])
{
  ScalorType tmp[NumberOfStatisticItems];
  hipMemcpy (tmp, st.ddata, sizeof(ScalorType) * NumberOfStatisticItems,
	      hipMemcpyDeviceToHost);
  checkCUDAError ("MDStatistic::deviceCopy");
  updateHost();
  for (unsigned i = 0; i < num; ++i){
    hdata[items[i]] = tmp[items[i]];
  }
  hipMemcpy (ddata, hdata, sizeof(ScalorType) * NumberOfStatisticItems,
	      hipMemcpyHostToDevice);
}

void MDStatistic::
add  (const MDStatistic & st,
      const IndexType num,
      const mdStatisticItem_t items[NumberOfStatisticItems])
{
  ScalorType tmp[NumberOfStatisticItems];
  hipMemcpy (tmp, st.ddata, sizeof(ScalorType) * NumberOfStatisticItems,
	      hipMemcpyDeviceToHost);
  checkCUDAError ("MDStatistic::deviceCopy");
  updateHost();
  for (unsigned i = 0; i < num; ++i){
    hdata[items[i]] += tmp[items[i]];
  }
  hipMemcpy (ddata, hdata, sizeof(ScalorType) * NumberOfStatisticItems,
	      hipMemcpyHostToDevice);
}

  
__global__ static void
syncData (ScalorType * tmpddata,
	  ScalorType * ddata,
	  int * flag)
{
  if (threadIdx.x < NumberOfStatisticItems){
    int myflag = flag[threadIdx.x];
    if (myflag != -1){
      ddata[myflag] = tmpddata[myflag];
    }
  }
}

void MDStatistic::
setEnergyCorr (const ScalorType & energyCorr_)
{
  hdata[mdStatisticEnergyCorrection] = energyCorr_;
  int flag[NumberOfStatisticItems];
  for (unsigned i = 0; i < NumberOfStatisticItems; ++i){
    flag[i] = -1;
  }
  flag[0] = mdStatisticEnergyCorrection;
  ScalorType * tmpddata;
  int * dflag;
  hipMalloc ((void**)&tmpddata, sizeof(ScalorType) * NumberOfStatisticItems);
  hipMalloc ((void**)&dflag, sizeof(int) * NumberOfStatisticItems);
  checkCUDAError("MDStatistic::setEnergyCorr allocate for tmpddata");
  hipMemcpy (tmpddata, hdata, sizeof(ScalorType) * NumberOfStatisticItems,
	      hipMemcpyHostToDevice);
  hipMemcpy (dflag, flag, sizeof(int) * NumberOfStatisticItems,
	      hipMemcpyHostToDevice);
  syncData<<<1, NumberOfStatisticItems>>> (tmpddata, ddata, dflag);
  checkCUDAError("MDStatistic::setEnergyCorr sync for tmpddata");
  hipFree (tmpddata);
  hipFree (dflag);
}

void MDStatistic::
setPressureCorr (const ScalorType & pressureCorr_)
{
  hdata[mdStatisticPressureCorrection] = pressureCorr_;
  // printf ("# setting pressureCorr_ to %f\n", pressureCorr_);
  int flag[NumberOfStatisticItems];
  for (unsigned i = 0; i < NumberOfStatisticItems; ++i){
    flag[i] = -1;
  }
  flag[0] = mdStatisticPressureCorrection;
  ScalorType * tmpddata;
  int * dflag;
  hipMalloc ((void**)&tmpddata, sizeof(ScalorType) * NumberOfStatisticItems);
  hipMalloc ((void**)&dflag, sizeof(int) * NumberOfStatisticItems);
  checkCUDAError("MDStatistic::setPressureCorr allocate for tmpddata");
  hipMemcpy (tmpddata, hdata, sizeof(ScalorType) * NumberOfStatisticItems,
	      hipMemcpyHostToDevice);
  hipMemcpy (dflag, flag, sizeof(int) * NumberOfStatisticItems,
	      hipMemcpyHostToDevice);
  syncData<<<1, NumberOfStatisticItems>>> (tmpddata, ddata, dflag);
  checkCUDAError("MDStatistic::setPressureCorr sync for tmpddata");
  hipFree (tmpddata);
  hipFree (dflag);
}
