#include "hip/hip_runtime.h"
#include "Statistic_interface.h"


// __global__ void initBuff (ScalorType * buff, IndexType n);
// void Statistic::init(const MDSystem & sys, 
// 		     const IndexType & NThread)
// {
//   myBlockDim.y = 1;
//   myBlockDim.z = 1;
//   myBlockDim.x = NThread;
//   IndexType nob;
//   if (sys.ddata.numAtom % myBlockDim.x == 0){
//     nob = sys.ddata.numAtom / myBlockDim.x;
//   } else {
//     nob = sys.ddata.numAtom / myBlockDim.x + 1;
//   }
//   atomGridDim = toGridDim (nob);

//   hostData.data = (ScalorType *) malloc (sizeof(ScalorType) * Size_StatisticData);

//   hipMalloc((void**)&(deviceData.data), sizeof(ScalorType) * Size_StatisticData);
//   clearStatisticData <<<1, 1>>> (deviceData);

//   updateHost();
  
//   hipMalloc((void**)&(statistic_buff), sizeof(ScalorType) * nob);
//   initBuff <<<1, 1>>> (statistic_buff, nob);
//   checkCUDAError("Statistic::init");
// }

// void Statistic::clearDevice ()
// {
//   clearStatisticData <<<1, 1>>> (deviceData);
//   checkCUDAError("Statistic::clearDevice");
// }

// void Statistic::updateHost()
// {
//   hipMemcpy (hostData.data, deviceData.data, 
// 	      sizeof(ScalorType) * Size_StatisticData, hipMemcpyDeviceToHost);
//   checkCUDAError("Statistic::updateHost");
// }

// Statistic::~Statistic()
// {
//   free (hostData.data);
//   hipFree (deviceData.data);
//   hipFree (statistic_buff);
//   checkCUDAError("Statistic::~Statistic");
// }


// __global__ void initBuff (ScalorType * buff, IndexType n)
// {
//   IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
//   IndexType tid = threadIdx.x;
//   if (bid + tid == 0)
//     for (IndexType i = 0; i < n; ++i){
//       buff[i] = 0;
//     }
// }


MDStatistic::
MDStatistic ()
{
  hdata = NULL;
  dmalloced = false;
  volume = 0;
}


void MDStatistic::
init (const MDSystem & sys)
{
  // recorde system infomation
  volume = sys.box.size.x * sys.box.size.y * sys.box.size.z;

  // malloc and init system
  hdata = (ScalorType *) malloc (sizeof(ScalorType) * NumberOfStatisticItems);
  if (hdata == NULL){
    throw MDExcptFailedMallocOnHost ("MDStatistic::MDStatistic", "hdata",
				     sizeof(ScalorType) * NumberOfStatisticItems);
  }
  hipMalloc ((void**)&ddata, sizeof(ScalorType) * NumberOfStatisticItems);
  checkCUDAError("MDStatistic::MDStatistic allocate for ddata");
  dmalloced = true;

  for (IndexType i = 0; i < NumberOfStatisticItems; ++i){
    hdata[i] = 0.f;
  }
  hipMemcpy (ddata, hdata, sizeof(ScalorType) * NumberOfStatisticItems,
	      hipMemcpyHostToDevice);
  checkCUDAError("MDStatistic::MDStatistic cpy form host to device");
}

MDStatistic::
~MDStatistic ()
{
  freeAPointer ((void**)&hdata);
  if (dmalloced){
    hipFree (ddata);
  }
}

__global__ void clearStatisticData (ScalorType *ddata)
{
  if (threadIdx.x < NumberOfStatisticItems){
    ddata[threadIdx.x] = 0.f;
  }
}

__global__ void addStatisticData (ScalorType * ddata, const ScalorType * cddata)
{
  if (threadIdx.x < NumberOfStatisticItems){
    ddata[threadIdx.x] += cddata[threadIdx.x];
  }
}

void MDStatistic::
clearDevice ()
{
  clearStatisticData <<<1, NumberOfStatisticItems>>> (ddata);
  checkCUDAError("Statistic::clearDevice");
}

void MDStatistic::
updateHost ()
{
  hipMemcpy (hdata, ddata, sizeof(ScalorType) * NumberOfStatisticItems, 
	      hipMemcpyDeviceToHost);
  checkCUDAError("Statistic::updateHost");
}

void MDStatistic::
deviceAdd (const MDStatistic & st)
{
  addStatisticData <<<1, NumberOfStatisticItems>>> (ddata, st.ddata);
}

  
