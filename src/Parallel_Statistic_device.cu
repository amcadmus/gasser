#include "hip/hip_runtime.h"
#define DEVICE_CODE

#include "Parallel_Statistic.h"
#include "compile_error_mixcode.h"

Parallel::DeviceStatistic::
~DeviceStatistic ()
{
  if (dmalloced){
    hipFree (ddata);
  }
}


void Parallel::DeviceStatistic::
reinit (const DeviceCellListedMDData & data)
{
  volume = data.getGlobalBox().size.x * data.getGlobalBox().size.y *
      data.getGlobalBox().size.z;

  size = sizeof (ScalorType) * NumberOfStatisticItems;
  if (!dmalloced){
    hipMalloc ((void**)&ddata, size);
    checkCUDAError("DeviceStatistic::init, malloc");
    dmalloced = true;
  }
  
  clearData ();
}

__global__ void Parallel::CudaGlobal::
clearStatisticData (ScalorType *ddata)
{
  if (threadIdx.x < NumberOfStatisticItems){
    ddata[threadIdx.x] = 0.f;
  }
}

__global__ void Parallel::CudaGlobal::
addStatisticData (ScalorType * ddata, const ScalorType * cddata)
{
  if (threadIdx.x < NumberOfStatisticItems){
    ddata[threadIdx.x] += cddata[threadIdx.x];
  }
}

void Parallel::DeviceStatistic::
clearData ()
{
  Parallel::CudaGlobal::clearStatisticData <<<1, NumberOfStatisticItems>>> (ddata);
  checkCUDAError("DeviceStatistic::clearDevice");
}

void Parallel::DeviceStatistic::
add (const DeviceStatistic & st)
{
  Parallel::CudaGlobal::addStatisticData <<<1, NumberOfStatisticItems>>> (ddata, st.ddata);
  checkCUDAError("DeviceStatistic::add");
}

void Parallel::DeviceStatistic::
copy (const DeviceStatistic & st)
{
  hipMemcpy (ddata, st.ddata, sizeof(ScalorType) * NumberOfStatisticItems,
	      hipMemcpyDeviceToDevice);
  checkCUDAError ("DeviceStatistic::copy");
}

void Parallel::DeviceStatistic::
copyToHost (HostStatistic & hst)
{
  hipMemcpy (hst.cptr_localStatisticData(), ddata, size, hipMemcpyDeviceToHost);
  checkCUDAError ("DeviceStatistic::copyToHost");
}

