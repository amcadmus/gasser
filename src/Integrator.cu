#include "hip/hip_runtime.h"
#include "MDSystem_interface.h"
#include "common.h"
#include "Integrator.h"
#include "Auxiliary.h"
#include <stdio.h>
#include "Statistic_interface.h"


__global__ void leapFrog1Step (const IndexType numAtom,
			       const ScalorType * massi,
			       CoordType * coord,
			       ScalorType * velox,
			       ScalorType * veloy, 
			       ScalorType * veloz,
			       const ScalorType * forcx,
			       const ScalorType * forcy, 
			       const ScalorType * forcz,
			       const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  if (ii >= numAtom) return;
  ScalorType mi = massi[ii];
  ScalorType vx, vy, vz;
  vx = (velox[ii] += dt * forcx[ii] * mi);
  coord[ii].x += dt * vx;
  vy = (veloy[ii] += dt * forcy[ii] * mi);
  coord[ii].y += dt * vy;
  vz = (veloz[ii] += dt * forcz[ii] * mi);
  coord[ii].z += dt * vz;
}

__global__ void leapFrog1Step (const IndexType numAtom,
			       const ScalorType * mass,
			       const ScalorType * massi,
			       CoordType * coord,
			       ScalorType * velox,
			       ScalorType * veloy, 
			       ScalorType * veloz,
			       const ScalorType * forcx,
			       const ScalorType * forcy, 
			       const ScalorType * forcz,
			       const ScalorType dt,
			       ScalorType * statistic_buffxx,
			       ScalorType * statistic_buffyy,
			       ScalorType * statistic_buffzz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  ScalorType vx, vy, vz;
  
  if (ii < numAtom) {
    ScalorType mi = massi[ii];
    vx = (velox[ii] += dt * forcx[ii] * mi);
    coord[ii].x += dt * vx;
    vy = (veloy[ii] += dt * forcy[ii] * mi);
    coord[ii].y += dt * vy;
    vz = (veloz[ii] += dt * forcz[ii] * mi);
    coord[ii].z += dt * vz;
  }

  extern __shared__ volatile ScalorType buff [];

  ScalorType scalor;
  if (ii < numAtom) scalor = 0.5f * mass[ii];
  else scalor = 0.f;
  
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vx * vx;
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffxx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vy * vy;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffyy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vz * vz;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffzz[bid] = buff[0];
}

#ifndef COORD_IN_ONE_VEC
__global__ void leapFrogStepX (const IndexType numAtom,
			       const ScalorType * massi,
			       ScalorType * coordx,
			       ScalorType * coordy, 
			       ScalorType * coordz,
			       const ScalorType * velox,
			       const ScalorType * veloy, 
			       const ScalorType * veloz,
			       const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  if (ii >= numAtom) return;
  coordx[ii] += dt * velox[ii];
  coordy[ii] += dt * veloy[ii];
  coordz[ii] += dt * veloz[ii];
}
#else
__global__ void leapFrogStepX (const IndexType numAtom,
			       const ScalorType * massi,
			       CoordType * coord,
			       const ScalorType * velox,
			       const ScalorType * veloy, 
			       const ScalorType * veloz,
			       const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  if (ii >= numAtom) return;
  coord[ii].x += dt * velox[ii];
  coord[ii].y += dt * veloy[ii];
  coord[ii].z += dt * veloz[ii];
}
#endif


__global__ void leapFrogStepV (const IndexType numAtom,
			       const ScalorType * massi,
			       ScalorType * velox,
			       ScalorType * veloy, 
			       ScalorType * veloz,
			       const ScalorType * forcx,
			       const ScalorType * forcy, 
			       const ScalorType * forcz,
			       const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  
  if (ii < numAtom) {
    ScalorType mi = massi[ii];
    (velox[ii] += dt * forcx[ii] * mi);
    (veloy[ii] += dt * forcy[ii] * mi);
    (veloz[ii] += dt * forcz[ii] * mi);
  }
}

__global__ void leapFrogStepV (const IndexType numAtom,
			       const ScalorType * mass,
			       const ScalorType * massi,
			       ScalorType * velox,
			       ScalorType * veloy, 
			       ScalorType * veloz,
			       const ScalorType * forcx,
			       const ScalorType * forcy, 
			       const ScalorType * forcz,
			       const ScalorType dt,
			       ScalorType * statistic_buffxx,
			       ScalorType * statistic_buffyy,
			       ScalorType * statistic_buffzz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  ScalorType vx, vy, vz;
  
  if (ii < numAtom) {
    ScalorType mi = massi[ii];
    vx = (velox[ii] += dt * forcx[ii] * mi);
    vy = (veloy[ii] += dt * forcy[ii] * mi);
    vz = (veloz[ii] += dt * forcz[ii] * mi);
  }

  extern __shared__ volatile ScalorType buff [];

  ScalorType scalor;
  if (ii < numAtom) scalor = 0.5f * mass[ii];
  else scalor = 0.f;
  
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vx * vx;
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffxx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vy * vy;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffyy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vz * vz;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffzz[bid] = buff[0];
}


__device__ IndexType integrator_counter_prepare_x = 0;
__device__ IndexType integrator_counter_prepare_y = 0;
__device__ IndexType integrator_counter_prepare_z = 0;
__global__ void initRemoveTranslationalFreedom ()
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  if (tid + bid == 0){
    integrator_counter_prepare_x = 0;
    integrator_counter_prepare_y = 0;
    integrator_counter_prepare_z = 0;
  }
}

__global__ void prepareRemoveTranslationalFreedom (IndexType numAtom,
						   ScalorType * mass,
						   ScalorType * velox,
						   ScalorType * veloy,
						   ScalorType * veloz,
						   ScalorType * buffx,
						   ScalorType * buffy,
						   ScalorType * buffz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = threadIdx.x + bid * blockDim.x;

  extern __shared__ volatile ScalorType buff[];
  buff[tid] = 0.f;
  buff[tid+blockDim.x] = 0.f;
  __syncthreads();
  if (ii < numAtom){
    buff[tid] = mass[ii] * velox[ii];
  }
  __syncthreads();
  sumVectorBlockBuffer (buff, blockDim.x);
  if (tid == 0) buffx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[tid] = mass[ii] * veloy[ii];
  }
  __syncthreads();
  sumVectorBlockBuffer (buff, blockDim.x);
  if (tid == 0) buffy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[tid] = mass[ii] * veloz[ii];
  }
  __syncthreads();
  sumVectorBlockBuffer (buff, blockDim.x);
  if (tid == 0) buffz[bid] = buff[0];
}

__global__ void removeFreedom (IndexType numAtom,
			       ScalorType * velox, 
			       ScalorType * veloy,
			       ScalorType * veloz,
			       ScalorType totalMassi,
			       ScalorType * sums)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  if (ii < numAtom) {
    velox[ii] -= sums[0] * totalMassi;
    veloy[ii] -= sums[1] * totalMassi;
    veloz[ii] -= sums[2] * totalMassi;
  }
}


#ifndef COORD_IN_ONE_VEC
__global__ void velocityVerlet_part1 (const IndexType numAtom,
				      const ScalorType * massi,
				      ScalorType * coordx,
				      ScalorType * coordy, 
				      ScalorType * coordz,
				      ScalorType * velox,
				      ScalorType * veloy, 
				      ScalorType * veloz,
				      const ScalorType * forcx,
				      const ScalorType * forcy, 
				      const ScalorType * forcz,
				      const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  if (ii >= numAtom) return;
  
  ScalorType hdtmi = 0.5f*dt*massi[ii];
  velox[ii]  += hdtmi * forcx[ii];
  coordx[ii] += dt * velox[ii];
  veloy[ii]  += hdtmi * forcy[ii];
  coordy[ii] += dt * veloy[ii];
  veloz[ii]  += hdtmi * forcz[ii];
  coordz[ii] += dt * veloz[ii];
}
#else
__global__ void velocityVerlet_part1 (const IndexType numAtom,
				      const ScalorType * massi,
				      CoordType * coord,
				      ScalorType * velox,
				      ScalorType * veloy, 
				      ScalorType * veloz,
				      const ScalorType * forcx,
				      const ScalorType * forcy, 
				      const ScalorType * forcz,
				      const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  if (ii >= numAtom) return;
  
  ScalorType hdtmi = 0.5f*dt*massi[ii];
  velox[ii]  += hdtmi * forcx[ii];
  coord[ii].x += dt * velox[ii];
  veloy[ii]  += hdtmi * forcy[ii];
  coord[ii].y += dt * veloy[ii];
  veloz[ii]  += hdtmi * forcz[ii];
  coord[ii].z += dt * veloz[ii];
}
#endif


__global__ void velocityVerlet_part2 (const IndexType numAtom,
				      const ScalorType * massi,
				      ScalorType * velox,
				      ScalorType * veloy, 
				      ScalorType * veloz,
				      const ScalorType * forcx,
				      const ScalorType * forcy, 
				      const ScalorType * forcz,
				      const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  if (ii >= numAtom) return;

  ScalorType hdtmi = 0.5f*dt*massi[ii];
  velox[ii] += hdtmi * forcx[ii];
  veloy[ii] += hdtmi * forcy[ii];
  veloz[ii] += hdtmi * forcz[ii];
}


__global__ void velocityVerlet_part2 (const IndexType numAtom,
				      const ScalorType * mass,
				      const ScalorType * massi,
				      ScalorType * velox,
				      ScalorType * veloy, 
				      ScalorType * veloz,
				      const ScalorType * forcx,
				      const ScalorType * forcy, 
				      const ScalorType * forcz,
				      const ScalorType dt,
				      ScalorType * statistic_buffxx,
				      ScalorType * statistic_buffyy,
				      ScalorType * statistic_buffzz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  extern __shared__ volatile ScalorType buff [];
ScalorType vx(0.f), vy(0.f), vz(0.f);
  if (ii < numAtom) {
    ScalorType hdtmi = 0.5f*dt*massi[ii];
    vx = (velox[ii] += hdtmi * forcx[ii]);
    vy = (veloy[ii] += hdtmi * forcy[ii]);
    vz = (veloz[ii] += hdtmi * forcz[ii]);
  }

  ScalorType scalor;
  if (ii < numAtom) scalor = 0.5f * mass[ii];
  else scalor = 0.f;
  
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vx * vx;
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffxx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vy * vy;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffyy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vz * vz;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffzz[bid] = buff[0];
}

__global__ void velocityVerlet_part2a (const IndexType numAtom,
				       const ScalorType * mass,
				       const ScalorType * massi,
				       ScalorType * velox,
				       ScalorType * veloy, 
				       ScalorType * veloz,
				       const ScalorType * forcx,
				       const ScalorType * forcy, 
				       const ScalorType * forcz,
				       const ScalorType dt,
				       ScalorType * statistic_buff)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;

  ScalorType vx, vy, vz;
  if (ii < numAtom) { 
    ScalorType hdtmi = 0.5f*dt*massi[ii];
    vx = (velox[ii] += hdtmi * forcx[ii]);
    vy = (veloy[ii] += hdtmi * forcy[ii]);
    vz = (veloz[ii] += hdtmi * forcz[ii]);
  }

  extern __shared__ volatile ScalorType buff [];
  if (ii < numAtom)
    buff[tid] = 0.5 * mass[ii] * (vx*vx + vy*vy + vz*vz);
  else
    buff[tid] = 0.f;
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buff[bid] = buff[0];  
}





__global__ void velocityRescale_rescale (const IndexType numAtom,
					 ScalorType * velox,
					 ScalorType * veloy, 
					 ScalorType * veloz,
					 const ScalorType alpha)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  
  if (ii < numAtom) {
    (velox[ii] *= alpha);
    (veloy[ii] *= alpha);
    (veloz[ii] *= alpha);
  }
}

__global__ void velocityRescale_rescale (const IndexType numAtom,
					 const ScalorType * mass,
					 ScalorType * velox,
					 ScalorType * veloy, 
					 ScalorType * veloz,
					 const ScalorType alpha,
					 ScalorType * statistic_buffxx,
					 ScalorType * statistic_buffyy,
					 ScalorType * statistic_buffzz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  ScalorType vx, vy, vz;
  
  if (ii < numAtom) {
    vx = (velox[ii] *= alpha);
    vy = (veloy[ii] *= alpha);
    vz = (veloz[ii] *= alpha);
  }

  extern __shared__ volatile ScalorType buff [];

  ScalorType scalor;
  if (ii < numAtom) scalor = 0.5 * mass[ii];
  else scalor = 0.f;
  
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vx * vx;
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffxx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vy * vy;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffyy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vz * vz;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffzz[bid] = buff[0];
}



__global__ void rescaleData (const IndexType numAtom,
			     ScalorType * data,
			     ScalorType alpha)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  
  if (ii < numAtom) data[ii] *= alpha;
}



__global__ void
leapFrogStepV_VCouple (const IndexType numAtom,
		       const ScalorType * massi,
		       ScalorType * velox,
		       ScalorType * veloy, 
		       ScalorType * veloz,
		       const ScalorType * forcx,
		       const ScalorType * forcy, 
		       const ScalorType * forcz,
		       const ScalorType lambda0,
		       const ScalorType lambda1,
		       const ScalorType lambda2,
		       const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  
  if (ii < numAtom) {
    ScalorType mi = massi[ii];
    ScalorType hdt = 0.5f * dt;
    ScalorType tmp = 1.f - hdt * lambda0;
    ScalorType tmp1= 1.f + hdt * lambda0;
    velox[ii] = (tmp * velox[ii] + dt * mi * forcx[ii]) / tmp1;
    tmp = 1.f - hdt * lambda1;
    tmp1= 1.f + hdt * lambda1;
    veloy[ii] = (tmp * veloy[ii] + dt * mi * forcy[ii]) / tmp1;
    tmp = 1.f - hdt * lambda2;
    tmp1= 1.f + hdt * lambda2;
    veloz[ii] = (tmp * veloz[ii] + dt * mi * forcz[ii]) / tmp1;
  }
}

__global__ void
leapFrogStepV_VCouple (const IndexType numAtom,
		       const ScalorType * mass,
		       const ScalorType * massi,
		       ScalorType * velox,
		       ScalorType * veloy, 
		       ScalorType * veloz,
		       const ScalorType * forcx,
		       const ScalorType * forcy, 
		       const ScalorType * forcz,
		       const ScalorType lambda0,
		       const ScalorType lambda1,
		       const ScalorType lambda2,
		       const ScalorType dt,
		       ScalorType * statistic_buffxx,
		       ScalorType * statistic_buffyy,
		       ScalorType * statistic_buffzz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  ScalorType vx, vy, vz;
  
  if (ii < numAtom) {
    ScalorType mi = massi[ii];
    ScalorType hdt = 0.5f * dt;
    ScalorType tmp = 1.f - hdt * lambda0;
    ScalorType tmp1= 1.f + hdt * lambda0 ;
    vx = (velox[ii] = ((tmp * velox[ii] + dt * mi * forcx[ii]) / tmp1));
    tmp = 1.f - hdt * lambda1;
    tmp1= 1.f + hdt * lambda1;
    vy = (veloy[ii] = ((tmp * veloy[ii] + dt * mi * forcy[ii]) / tmp1));
    tmp = 1.f - hdt * lambda2;
    tmp1= 1.f + hdt * lambda2;
    vz = (veloz[ii] = ((tmp * veloz[ii] + dt * mi * forcz[ii]) / tmp1));
  }

  extern __shared__ volatile ScalorType buff [];

  ScalorType scalor;
  if (ii < numAtom) scalor = 0.5f * mass[ii];
  else scalor = 0.f;
  
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vx * vx;
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffxx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vy * vy;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffyy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vz * vz;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffzz[bid] = buff[0];
}


