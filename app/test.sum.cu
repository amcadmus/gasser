#include "hip/hip_runtime.h"
#include <assert.h>
#include <ctype.h>
#include <errno.h>
#include <limits.h>
#include <string.h>
#include <stdarg.h>
#include <stdlib.h>
#include <stdio.h>

#include "common.h"
#include "SumBlock.h"


__global__ void sum32 (ScalorType * a, ScalorType * result)
{
  if (blockDim.x != 32) return ;
  __shared__ ScalorType buff[32];
  buff[threadIdx.x] = a[threadIdx.x];
  SumBlock::sum32_1bsize (buff);
  if (threadIdx.x == 0) *result = buff[0];
}


#define N 32


int main(int argc, char * argv[])
{
  ScalorType hdata [N];
  for (IndexType i = 0; i < N; ++i){
    hdata[i] = i;
  }
  ScalorType * ddata;
  hipMalloc ((void**)&ddata, sizeof(ScalorType) * N);
  hipMemcpy (ddata, hdata, sizeof(ScalorType)*N, hipMemcpyHostToDevice);

  ScalorType hresule;
  ScalorType * dresult;
  hipMalloc ((void**)&dresult, sizeof(ScalorType));

  hipEvent_t start, stop;
  float tmptime = 0;
  hipEventCreate (&start);
  hipEventCreate (&stop );
  hipEventRecord(start, 0);
  for (unsigned i = 0; i < 1; ++i){
    sum32 <<<1, N>>> (ddata, dresult);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize (stop);
  hipEventElapsedTime (&tmptime, start, stop);

  hipMemcpy (&hresule, dresult, sizeof(ScalorType), hipMemcpyDeviceToHost);
  printf ("result: %f, time %.3e\n", hresule, tmptime);

    
  hipFree(dresult);
  hipEventDestroy (start);
  hipEventDestroy (stop);

  return 0;
}

