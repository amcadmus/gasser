#include <stdio.h>
#include "MDSystem_interface.h"
#include "common.h"
#include "BoxGeometry.h"
#include "MDSystem.h"
#include "RandomGenerator.h"
#include "Auxiliary.h"
#include "NeighborList_interface.h"
#include"Statistic.h"
#include "Integrator_interface.h"
#include "InteractionEngine_interface.h"
#include "tmp.h"
#include "Reshuffle_interface.h"
#include "Displacement_interface.h"

#include "Topology.h"
#include "SystemBondedInteraction.h"

#include "BondInteraction.h"
#include "NonBondedInteraction.h"


// #define NThreadsPerBlockCell	32
// #define NThreadsPerBlockAtom	4

#define NThreadsPerBlockCell	160
#define NThreadsPerBlockAtom	96

int main(int argc, char * argv[])
{
  IndexType nstep = 100000;
  IndexType confFeq = 100000;
  IndexType thermoFeq = 100;
  ScalorType rcut = 4.0;
  ScalorType nlistExten = 0.3;
  ScalorType refT = 1.50;
  ScalorType tauT = 1.;
  ScalorType refP = 0.2;
  ScalorType tauP = 1.;
  char * filename;
  
  if (argc != 4){
    printf ("Usage:\n%s conf.gro nstep device\n", argv[0]);
    return 1;
  }
  if (argc != 1){
    nstep = atoi(argv[2]);
    filename = argv[1];
  }
  printf ("# setting device to %d\n", atoi(argv[3]));
  hipSetDevice (atoi(argv[3]));
  checkCUDAError ("set device");

  MDSystem sys;
  sys.initConfig(filename);

  Topology::System sysTop;
  Topology::Molecule mol;
  mol.pushAtom (Topology::Atom (1.0, 0.0, 0));
  LennardJones6_12Parameter ljparam;
  ljparam.reinit (1.f, 1.f, 0.f, rcut);
  sysTop.addNonBondedInteraction (Topology::NonBondedInteraction(0, 0, ljparam));
  sysTop.addMolecules (mol, sys.hdata.numAtom);

  sys.initTopology (sysTop);
  sys.initDeviceData ();
  
  SystemNonBondedInteraction sysNbInter;
  sysNbInter.reinit (sysTop);
  ScalorType energyCorr = sysNbInter.energyCorrection ();
  ScalorType pressureCorr = sysNbInter.pressureCorrection ();
  
  ScalorType maxrcut = sysNbInter.maxRcut();
  ScalorType rlist = maxrcut + nlistExten;
  CellList clist (sys, rlist, NThreadsPerBlockCell, NThreadsPerBlockAtom);
  NeighborList nlist (sysNbInter, sys, rlist, NThreadsPerBlockAtom, 10.f);
  sys.normalizeDeviceData ();
  clist.rebuild (sys, NULL);
  nlist.rebuild (sys, clist, NULL);
  Displacement_max disp (sys, NThreadsPerBlockAtom);
  disp.recordCoord (sys);
  
  MDStatistic st(sys);
  MDStatistic old_st(sys), tmp_st(sys);
  old_st.setEnergyCorr (energyCorr);
  old_st.setPressureCorr (pressureCorr);
  tmp_st.setEnergyCorr (energyCorr);
  tmp_st.setPressureCorr (pressureCorr);
  TranslationalFreedomRemover tfremover (sys, NThreadsPerBlockAtom);
  InteractionEngine inter (sys, NThreadsPerBlockAtom);
  inter.registNonBondedInteraction (sysNbInter);
  
  MDTimer timer;
  unsigned i;
  ScalorType dt = 0.002;
  ScalorType seed = 1;
  RandomGenerator_MT19937::init_genrand (seed);

  VelocityVerlet inte_vv (sys, NThreadsPerBlockAtom);
  VelocityRescale inte_vr (sys, NThreadsPerBlockAtom, refT, 0.1);
  NoseHoover_Chains2_Isobaric nhcp;
  nhcp.reinit (sys, NThreadsPerBlockAtom, refT, tauT, refP, tauP, pressureCorr);

  Reshuffle resh (sys);
  
  timer.tic(mdTimeTotal);
  if (resh.calIndexTable (clist, &timer)){
    sys.reshuffle   (resh.indexTable, sys.hdata.numAtom, &timer);
    clist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
    nlist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
    disp.reshuffle  (resh.indexTable, sys.hdata.numAtom, &timer);  
  }
  
  printf ("# prepare ok, start to run\n");
  sys.recoverDeviceData (&timer);
  sys.updateHostFromRecovered (&timer);
  sys.writeHostDataGro ("confstart.gro", 0, 0.f, &timer);
  printf ("# prepare ok, start to run\n");
  printf ("#*     1     2           3         4            5       6         7    8    9     10  11     12   13   14    15   16    17\n");
  printf ("#* nstep  time  nonBondedE  kineticE  temperature  totalE  pressure  box  volume  h0  h1  NHC_H  vep  xi1  vxi1  xi2  vxi2\n");

  try{
    sys.initWriteXtc ("traj.xtc");
    sys.recoverDeviceData (&timer);
    sys.updateHostFromRecovered (&timer);
    sys.writeHostDataXtc (0, 0*dt, &timer);
    for (i = 0; i < nstep; ++i){
      if (i%10 == 0){
	tfremover.remove (sys, &timer);
      }
      old_st.deviceCopy (st);
      old_st.updateHost ();
      nhcp.operator_L_CP (0.5 * dt, sys, old_st, &timer);
      
      nhcp.operator_L_v  (0.5 * dt, sys, &timer);

      nhcp.operator_L_r  (dt, sys, &timer);
      nhcp.operator_L_box (dt, sys.box);
      
      st.clearDevice();
      inter.clearInteraction (sys);
      ScalorType maxdr = disp.calMaxDisplacemant (sys, &timer);
      if (maxdr > nlistExten * 0.5){
	// printf ("# Rebuild at step %09i ... ", i+1);
	// fflush(stdout);
	// rebuild
	sys.normalizeDeviceData (&timer);
	disp.recordCoord (sys);
	clist.rebuild (sys, &timer);
	inter.applyNonBondedInteraction (sys, clist, rcut, st, &timer);
	nlist.rebuild (sys, clist, &timer);
	// printf ("done\n");
	// fflush(stdout);
      }
      else{
	inter.applyNonBondedInteraction (sys, nlist, st, NULL, &timer);
      }
      
      tmp_st.clearDevice();
      nhcp.operator_L_v  (0.5 * dt, sys, tmp_st, &timer);
      mdStatisticItem_t tmp_array[3];
      tmp_array[0] = mdStatisticVirialXX;
      tmp_array[1] = mdStatisticVirialYY;
      tmp_array[2] = mdStatisticVirialZZ;
      tmp_st.add (st, 3, tmp_array);
      
      nhcp.operator_L_CP (0.5 * dt, sys, tmp_st, st, &timer);

      if ((i+1) % thermoFeq == 0){
	st.updateHost ();
	ScalorType e = st.nonBondedEnergy () + st.kineticEnergy();	
	ScalorType v = sys.box.size.x * sys.box.size.y * sys.box.size.z;
	ScalorType p0 = st.pressure(sys.box);
	ScalorType h0 = v * p0 + e;
	ScalorType h1 = v * refP + e;

	printf ("%09d %07e %.7e %.7e %.7e %.7e %.7e %.7f %.7e %.7e %.7e %.5e %.5e %.5e %.5e\n",
		(i+1),  
		(i+1) * dt, 
		st.nonBondedEnergy (),
		st.kineticEnergy(),
		st.kineticEnergy() * 2. / (3. * (double (sys.hdata.numAtom) - 1.)),
		e,
		p0,
		sys.box.size.x,
		v,
		h0,
		h1,
		e + nhcp.HamiltonianContribution (sys.box),
		nhcp.vep,
		nhcp.xi1,
		nhcp.vxi1);
	fflush(stdout);
      }

      if ((i+1) % confFeq == 0){
      	sys.recoverDeviceData (&timer);
      	sys.updateHostFromRecovered (&timer);
      	sys.writeHostDataXtc (i+1, (i+1)*dt, &timer);
      }

      if ((i+1) % 100 == 0){
	if (resh.calIndexTable (clist, &timer)){
	  sys.reshuffle   (resh.indexTable, sys.hdata.numAtom, &timer);
	  clist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
	  nlist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
	  disp.reshuffle  (resh.indexTable, sys.hdata.numAtom, &timer);  
	}
      }
    }
    sys.endWriteXtc();
    sys.recoverDeviceData (&timer);
    sys.updateHostFromRecovered (&timer);
    sys.writeHostDataGro ("confout.gro", nstep, nstep*dt, &timer);
    timer.toc(mdTimeTotal);
    timer.printRecord (stderr);
  }
  catch (MDExcptCuda & e){
    // resh.recoverMDDataToHost (sys, &timer);
    // sys.writeHostDataXtc (i+1, (i+1)*dt, &timer);
    timer.toc(mdTimeTotal);
    timer.printRecord (stderr);
    return 1;
  }
  catch (MDException &e){
    fprintf (stderr, "%s\n", e.what());
    return 1;
  }
  
  
  return 0;
}

  
