#include "hip/hip_runtime.h"
/**
 * @file   lipid.cu
 * @author Han Wang <han_wang@math.pku.edu.cn>
 * @date   Thu Nov 19 12:53:01 2009
 * 
 * @brief  the main program to test the membrane simulation.
 * 
 * 
 */
#include <stdio.h>
#include "common.h"
#include "BoxGeometry.h"
#include "MDSystem.h"
#include "RandomGenerator.h"
#include "Auxiliary.h"
#include "NeighborList_interface.h"
#include"Statistic.h"
#include "Integrator_interface.h"
#include "InteractionEngine_interface.h"
#include "tmp.h"
#include "Reshuffle_interface.h"
#include "BondList_interface.h"

#include "MDSystem_interface.h"


#define NThreadsPerBlockCell	192
#define NThreadsPerBlockAtom	128

int main(int argc, char * argv[])
{
  IndexType nstep = 20;
  char * filename;
  
  if (argc != 4){
    printf ("Usage:\n%s conf.gro nstep device\n", argv[0]);
    return 1;
  }
  if (argc != 1){
    nstep = atoi(argv[2]);
    filename = argv[1];
  }
  printf ("# setting device to %d\n", atoi(argv[3]));
  hipSetDevice (atoi(argv[3]));
  checkCUDAError ("set device");
  
  MDSystem sys;
  sys.initConfig(filename, "lipid.map");
  sys.initNBForce(2);
  ScalorType cosparam[mdForceNParamCosTail];  
  CosTail::initParameter (cosparam, 1.f, 0.95f, 0.f);
  sys.addNBForce (0, 0, mdForceCosTail, cosparam);
  CosTail::initParameter (cosparam, 1.f, 0.975f, 0.f);
  sys.addNBForce (0, 1, mdForceCosTail, cosparam);
  CosTail::initParameter (cosparam, 1.f, 1.0f, 1.6f);
  sys.addNBForce (1, 1, mdForceCosTail, cosparam);

  sys.initBond ();
  ScalorType hsparam[mdForceNParamHarmonicSpring] ;
  HarmonicSpring::initParameter (hsparam, 10.f, 4.f);
  ScalorType feneparam[mdForceNParamFENE];
  FENE::initParameter (feneparam, 30.f, 1.5f);
  
  for (unsigned i = 0; i < sys.hdata.numAtom; i+=3){
    sys.addBond (i, i+1, mdForceFENE, feneparam);
    sys.addBond (i+2, i+1, mdForceFENE, feneparam);
    sys.addBond (i, i+2, mdForceHarmonicSpring, hsparam);
  }   
  sys.buildBond();

  ScalorType tmpsum = 0.;
  for (IndexType i = 0; i < sys.hdata.numAtom; ++i){
    tmpsum += 0.5 * (sys.hdata.velox[i] * sys.hdata.velox[i] +
		     sys.hdata.veloy[i] * sys.hdata.veloy[i] +
		     sys.hdata.veloz[i] * sys.hdata.veloz[i] );
  }
  printf ("# tmpsum is %f\n", tmpsum);

  ScalorType maxrcut = sys.calMaxNBRcut ();
  printf ("# max rcut is %f\n", maxrcut);
  ScalorType nlistExten = 0.5f;
  ScalorType rlist = maxrcut + nlistExten;
  // NeighborList nlist(sys, rlist, NThreadsPerBlockCell, 20,
  // 		     RectangularBoxGeometry::mdRectBoxDirectionX |
  // 		     RectangularBoxGeometry::mdRectBoxDirectionY |
  // 		     RectangularBoxGeometry::mdRectBoxDirectionZ);;
  NeighborList nlist(sys, rlist, NThreadsPerBlockCell, 40,
  		     RectangularBoxGeometry::mdRectBoxDirectionX |
  		     RectangularBoxGeometry::mdRectBoxDirectionY);
  
  nlist.build (sys);
  
  Reshuffle resh (sys, nlist, NThreadsPerBlockCell);
  resh.shuffleSystem ( sys, nlist);

  MDStatistic st(sys);

  VelocityVerlet inte (sys, NThreadsPerBlockAtom);;

  ScalorType refT = 0.9977411970749;
  VelocityRescale inte_vr (sys, NThreadsPerBlockAtom, refT, 0.1);
// // printf ("%f %f\n", ddata.velox[0], ddata.velox[1]);
  // inte.removeTranslationalFreedom (ddata);
  // // printf ("%f %f\n", ddata.velox[0], ddata.velox[1]);

  TranslationalFreedomRemover tfremover (sys, NThreadsPerBlockAtom);

  InteractionEngine_interface interaction(sys, NThreadsPerBlockAtom);;

  MDTimer timer;
  unsigned i;
  ScalorType dt = 0.005;
  ScalorType seed = 1;
  RandomGenerator_MT19937::init_genrand (seed);
  
  printf ("# prepare ok, start to run\n");

  try{
    timer.tic(mdTimeTotal);
    sys.initWriteXtc ("traj.xtc");
    sys.writeHostDataXtc (0, 0*dt, &timer);
    for (i = 0; i < nstep; ++i){ 
      if (i%10 == 0){
	tfremover.remove (sys, &timer);
      }
      // if (i%1 == 0){
      //   if (i == 0) nlist.build (sys);
      //   else nlist.reBuild (sys);
      //   // resh.shuffleSystem (sys, nlist);
      // }   
	  
      if ((i+1) % 10 == 0){
	st.clearDevice();
	inte_vr.step1 (sys, dt, &timer);
	interaction.applyInteraction (sys, nlist, st, &timer);
	inte_vr.step2 (sys, dt, st, &timer);
	st.updateHost();
	printf ("%09d %07e %.7e %.7e %.7e %.7e %.7e %.7e %.7e %.7e %.7e %.7e %.7e\n",
		(i+1),  
		(i+1) * dt, 
		st.getStatistic(mdStatisticNonBondedPotential),
		st.getStatistic(mdStatisticBondedPotential),
		st.kineticEnergy(),
		st.getStatistic(mdStatisticNonBondedPotential) +
		st.getStatistic(mdStatisticBondedPotential) +
		st.kineticEnergy(),
		st.getStatistic(mdStatisticVirialXX)*0.5,
		st.getStatistic(mdStatisticVirialYY)*0.5,
		st.getStatistic(mdStatisticVirialZZ)*0.5,
		st.pressureXX(),
		st.pressureYY(),
		st.pressureZZ(),
		st.pressure());
	fflush(stdout);
      }
      else {
	inte_vr.step1 (sys, dt, &timer);
	interaction.applyInteraction (sys, nlist, &timer);
	inte_vr.step2 (sys, dt, &timer);
      }
      if (nlist.judgeRebuild(sys, 0.5 * nlistExten, &timer)){
	printf ("# Rebuild at step %09i ... ", i+1);
	fflush(stdout);
	nlist.reBuild(sys, &timer);
	printf ("done\n");
	fflush(stdout);
      }
      if ((i+1) % 1000 == 0){
	// sys.updateHost(&timer);
	resh.recoverMDDataToHost (sys, &timer);
	sys.writeHostDataXtc (i+1, (i+1)*dt, &timer);
      }
      if ((i+1) % 200 == 0){
	resh.shuffleSystem (sys, nlist, &timer);
      }
    }
    sys.endWriteXtc();
    timer.toc(mdTimeTotal);
    timer.printRecord (stderr);
  }
  catch (MDExcptCuda & e){
    resh.recoverMDDataToHost (sys, &timer);
    sys.writeHostDataXtc (i+1, (i+1)*dt, &timer);
    timer.toc(mdTimeTotal);
    timer.printRecord (stderr);
    return 1;
  }
  catch (MDException &e){
    fprintf (stderr, "%s\n", e.what());
    return 1;
  }

  return 0;
}
