#include <stdio.h>
#include "MDSystem_interface.h"
#include "common.h"
#include "BoxGeometry.h"
#include "MDSystem.h"
#include "RandomGenerator.h"
#include "Auxiliary.h"
#include "NeighborList_interface.h"
#include"Statistic.h"
#include "Integrator_interface.h"
#include "InteractionEngine_interface.h"
#include "tmp.h"
#include "Reshuffle_interface.h"
#include "Displacement_interface.h"

#include "Topology.h"
#include "SystemBondedInteraction.h"

#include "BondInteraction.h"
#include "NonBondedInteraction.h"


#define NThreadsPerBlockCell	32
#define NThreadsPerBlockAtom	4

int main(int argc, char * argv[])
{
  IndexType nstep = 20;
  char * filename;
  
  if (argc != 4){
    printf ("Usage:\n%s conf.gro nstep device\n", argv[0]);
    return 1;
  }
  if (argc != 1){
    nstep = atoi(argv[2]);
    filename = argv[1];
  }
  printf ("# setting device to %d\n", atoi(argv[3]));
  hipSetDevice (atoi(argv[3]));
  checkCUDAError ("set device");

  MDSystem sys;
  sys.initConfig(filename);

  Topology::System sysTop;
  Topology::Molecule mol;
  mol.pushAtom (Topology::Atom (1.0, 0.0, 0));
  LennardJones6_12Parameter ljparam;
  ScalorType rcut = 6.0;
  ljparam.reinit (1.f, 1.f, 0.f, rcut);
  sysTop.addNonBondedInteraction (Topology::NonBondedInteraction(0, 0, ljparam));
  sysTop.addMolecules (mol, sys.hdata.numAtom);

  sys.initTopology (sysTop);
  sys.initDeviceData ();
  
  SystemNonBondedInteraction sysNbInter;
  sysNbInter.reinit (sysTop);
  
  ScalorType maxrcut = sysNbInter.maxRcut();
  ScalorType nlistExten = 0.3;
  ScalorType rlist = maxrcut + nlistExten;
  CellList clist (sys, rlist, NThreadsPerBlockCell, NThreadsPerBlockAtom);
  NeighborList nlist (sysNbInter, sys, rlist, NThreadsPerBlockAtom, 10.f);
  sys.normalizeDeviceData ();
  clist.rebuild (sys);
  nlist.rebuild (sys, clist);
  Displacement_max disp (sys, NThreadsPerBlockAtom);
  disp.recordCoord (sys);
  
  MDStatistic st (sys);
  MDStatistic last_st (sys);
  TranslationalFreedomRemover tfremover (sys, NThreadsPerBlockAtom);
  InteractionEngine inter (sys, NThreadsPerBlockAtom);
  inter.registNonBondedInteraction (sysNbInter);
  inter.clearInteraction (sys);
  inter.applyNonBondedInteraction (sys, nlist, st);

  MDTimer timer;
  unsigned i;
  ScalorType dt = 0.001;
  ScalorType seed = 1;
  RandomGenerator_MT19937::init_genrand (seed);
  ScalorType refT = 1.34;
  ScalorType tauT = 1.;
  ScalorType refP = 0.1420;
  ScalorType tauP = 1.;
  ScalorType betaP = 1.;
  Thermostat_NoseHoover thermostat;
  thermostat.reinit (refT, dt, tauT, sys.ddata.numAtom * 3 - 3);
  Barostat_ParrinelloRahman barostat;
  barostat.reinit (dt, tauP, sys.box);
  barostat.assignGroup (mdRectBoxDirectionX |
  			mdRectBoxDirectionY |
  			mdRectBoxDirectionZ,
  			refP, betaP);
  LeapFrog_TPCouple_VCouple blpf (sys, NThreadsPerBlockAtom);
  blpf.addThermostat (thermostat);
  blpf.addBarostat   (barostat);  

  Reshuffle resh (sys);
  
  timer.tic(mdTimeTotal);
  if (resh.calIndexTable (clist, &timer)){
    sys.reshuffle   (resh.indexTable, sys.hdata.numAtom, &timer);
    clist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
    nlist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
    disp.reshuffle  (resh.indexTable, sys.hdata.numAtom, &timer);  
  }
  
  printf ("# prepare ok, start to run\n");
  // sys.recoverDeviceData (&timer);
  // sys.updateHostFromRecovered (&timer);
  // sys.writeHostDataGro ("confstart.gro", 0, 0.f, &timer);
  printf ("# prepare ok, start to run\n");
  printf ("#*     1     2           3         4            5       6         7     8       9  10  11\n");
  printf ("#* nstep  time  nonBondedE  kineticE  temperature  totalE  pressure  boxl  volume  h0  h1\n");
  try{
    // sys.initWriteXtc ("traj.xtc");
    // sys.recoverDeviceData (&timer);
    // sys.updateHostFromRecovered (&timer);
    // sys.writeHostDataXtc (0, 0*dt, &timer);
    for (i = 0; i < nstep; ++i){
      last_st.deviceCopy (st);
      if (i%10 == 0){
	tfremover.remove (sys, &timer);
      }
      st.clearDevice();
      blpf.oneStep (sys, dt, last_st, st, &timer);
      ScalorType maxdr = disp.calMaxDisplacemant (sys, &timer);
      if (maxdr > nlistExten * 0.5){
	// printf ("# Rebuild at step %09i ... ", i+1);
	// fflush(stdout);
	// rebuild
	sys.normalizeDeviceData (&timer);
	disp.recordCoord (sys);
	clist.rebuild (sys, &timer);
	nlist.rebuild (sys, clist, &timer);
	// printf ("done\n");
	// fflush(stdout);
      }
      inter.clearInteraction (sys);
      inter.applyNonBondedInteraction (sys, nlist, st, &timer);
      if ((i+1) % 1 == 0){
	st.updateHost();
	ScalorType ep = st.nonBondedEnergy ();
	ScalorType ek = st.kineticEnergy();
	ScalorType e = ep + ek;
	ScalorType v = sys.box.size.x * sys.box.size.y * sys.box.size.z;
	ScalorType p0 = st.pressure(sys.box);
	ScalorType h0 = v * p0 + e;
	ScalorType h1 = v * refP + e;
	printf ("%09d %07e %.7e %.7e %.7e %.7e %.7e %.7f %.7e %.7e %.7e\n",
		(i+1),  
		(i+1) * dt, 
		ep,
		ek, 
		ek * 2. / (3. * (double (sys.hdata.numAtom) - 1.)),
		e,
		p0,
		sys.box.size.x,
		v,
		h0,
		h1
	    );
	fflush(stdout);
      }
      if ((i+1) % 100 == 0){
	if (resh.calIndexTable (clist, &timer)){
	  sys.reshuffle   (resh.indexTable, sys.hdata.numAtom, &timer);
	  clist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
	  nlist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
	  disp.reshuffle  (resh.indexTable, sys.hdata.numAtom, &timer);  
	}
      }
    }
    // sys.endWriteXtc();
    sys.recoverDeviceData (&timer);
    sys.updateHostFromRecovered (&timer);
    sys.writeHostDataGro ("confout.gro", nstep, nstep*dt, &timer);
    timer.toc(mdTimeTotal);
    timer.printRecord (stderr);
  }
  catch (MDExcptCuda & e){
    // resh.recoverMDDataToHost (sys, &timer);
    // sys.writeHostDataXtc (i+1, (i+1)*dt, &timer);
    timer.toc(mdTimeTotal);
    timer.printRecord (stderr);
    return 1;
  }
  catch (MDException &e){
    fprintf (stderr, "%s\n", e.what());
    return 1;
  }
  
  
  return 0;
}

  
