#include "SumVector.h"

// #define N 640000
#define N 640

int main(int argc, char * argv[])
{
  
  ScalorType hdata [N];
  for (IndexType i = 0; i < N; ++i){
    hdata[i] = i;
  }

  IndexType Nt = 32;
  if (argc != 1){
    Nt = atoi(argv[1]);
    printf("Nthread  is %d\n", Nt);
  }
  
  SumVector<ScalorType> sv;
  sv.init (N, Nt);
  hipMemcpy (sv.getBuff(), hdata, sizeof(ScalorType) * N, hipMemcpyHostToDevice);
  
  ScalorType * dresult;
  hipMalloc ((void**)&dresult, sizeof(ScalorType));
  ScalorType hresule;


  hipEvent_t start, stop;
  float tmptime;
  hipEventCreate (&start);
  hipEventCreate (&stop );
  hipEventRecord(start, 0);
  // for (unsigned i = 0; i < 10000; ++i){
  for (unsigned i = 0; i < 1; ++i){
    sv.sumBuff (dresult, 0);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize (stop);
  hipEventElapsedTime (&tmptime, start, stop);
  
  hipMemcpy (&hresule, dresult, sizeof(ScalorType), hipMemcpyDeviceToHost);
  printf ("result: %f, time %.3e\n", hresule, tmptime);

  
  hipFree(dresult);
  hipEventDestroy (start);
  hipEventDestroy (stop);
  
  return 0;
}
