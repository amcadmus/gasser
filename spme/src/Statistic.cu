#include "hip/hip_runtime.h"
#include "Statistic.h"


// __global__ void clearStatisticData	(StatisticData sdata)
// {
//   IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
//   if (threadIdx.x + bid == 0)
//     for (IndexType i = 0; i < Size_StatisticData; ++i){
//       sdata.data[i] = 0;
//     }
// }


// __host__ void initHostStatistic (HostStatistic * hst)
// {
//   hst->bondedP = 0;
//   hst->nonBondedP = 0;
//   hst->electrostaticP = 0;
//   hst->totalEnergy = 0;
//   hst->kinetic = 0;
//   hst->virial = 0;
//   hst->temperature = 0;
//   hst->pressure = 0;
// }

// __host__ void initDeviceStatistic (DeviceStatistic *dst)
// {
//   size_t size = sizeof(ScalorType);
//   hipMalloc ((void**)&(dst->bondedP), size);
//   hipMalloc ((void**)&(dst->nonBondedP), size);
//   hipMalloc ((void**)&(dst->electrostaticP), size);
//   hipMalloc ((void**)&(dst->totalEnergy), size);
//   hipMalloc ((void**)&(dst->kinetic), size);
//   hipMalloc ((void**)&(dst->virial), size);
//   hipMalloc ((void**)&(dst->temperature), size);
//   hipMalloc ((void**)&(dst->pressure), size);
// }

// __host__ void cpyDeviceStatisticToHost (const DeviceStatistic * dst,
// 					HostStatistic *hst)
// {
//   size_t size = sizeof(ScalorType);
//   hipMemcpy (dst->bondedP, &(hst->bondedP), size, hipMemcpyHostToDevice);
//   hipMemcpy (dst->nonBondedP, &(hst->nonBondedP), size, hipMemcpyHostToDevice);
//   hipMemcpy (dst->electrostaticP, &(hst->electrostaticP), size, hipMemcpyHostToDevice);
//   hipMemcpy (dst->totalEnergy, &(hst->totalEnergy), size, hipMemcpyHostToDevice);
//   hipMemcpy (dst->kinetic, &(hst->kinetic), size, hipMemcpyHostToDevice);
//   hipMemcpy (dst->virial, &(hst->virial), size, hipMemcpyHostToDevice);
//   hipMemcpy (dst->temperature, &(hst->temperature), size, hipMemcpyHostToDevice);
//   hipMemcpy (dst->pressure, &(hst->pressure), size, hipMemcpyHostToDevice);
// }

// __global__ void clearDeviceStatistic (DeviceStatistic * dst)
// {
//   *(dst->bondedP) = 0;
//   *(dst->nonBondedP) = 0;
//   *(dst->electrostaticP) = 0;
//   *(dst->totalEnergy) = 0;
//   *(dst->kinetic) = 0;
//   *(dst->virial) = 0;
//   *(dst->temperature) = 0;
//   *(dst->pressure) = 0;
// }

